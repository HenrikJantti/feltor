#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <map>
#include <sstream>
#include <cmath>

#include "dg/file/nc_utilities.h"
#include "feltor.cuh"

int main( int argc, char* argv[])
{
    ////////////////////////Parameter initialisation//////////////////////////
    Json::Value js, gs;
    Json::CharReaderBuilder parser;
    parser["collectComments"] = false;
    std::string errs;
    if( argc != 4)
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] [outputfile]\n";
        return -1;
    }
    else
    {
        std::ifstream is(argv[1]);
        std::ifstream ks(argv[2]);
        parseFromStream( parser, is, &js, &errs); //read input without comments
        parseFromStream( parser, ks, &gs, &errs); //read input without comments
    }
    const feltor::Parameters p( js);
    const dg::geo::solovev::Parameters gp(gs);
    p.display( std::cout);
    gp.display( std::cout);
    std::string input = js.toStyledString(), geom = gs.toStyledString();
    ////////////////////////////////set up computations///////////////////////////
    double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscaleRp*gp.a;
    double Zmax=p.boxscaleZp*gp.a*gp.elongation;
    //Make grids
    dg::CylindricalGrid3d grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI,
        p.n, p.Nx, p.Ny, p.Nz, p.bcxN, p.bcyN, dg::PER);
    dg::CylindricalGrid3d grid_out( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI,
        p.n_out, p.Nx_out, p.Ny_out, p.Nz_out, p.bcxN, p.bcyN, dg::PER);

    //create RHS
    std::cout << "Constructing Explicit...\n";
    feltor::Explicit<dg::CylindricalGrid3d, dg::IDMatrix, dg::DMatrix, dg::DVec>
        feltor( grid, p, gp); //initialize before im!
    std::cout << "Constructing Implicit...\n";
    feltor::Implicit< dg::CylindricalGrid3d, dg::IDMatrix, dg::DMatrix, dg::DVec >
        im( grid, p, gp);
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
    dg::DVec helper(dg::evaluate(dg::zero,grid));
    //perturbation
    dg::GaussianZ gaussianZ( 0., p.sigma_z*M_PI, 1); //modulation along fieldline
    if( p.initne == "blob" || p.initne == "straight blob")
    {
        dg::Gaussian init0( gp.R_0+p.posX*gp.a, p.posY*gp.a, p.sigma, p.sigma, p.amp);
        if( p.initne == "blob")
            helper = feltor.ds().fieldaligned().evaluate( init0, gaussianZ,
                (unsigned)p.Nz/2, 3); //rounds =3 ->2*3-1
        if( p.initne == "straight blob")
            helper = feltor.ds().fieldaligned().evaluate( init0, gaussianZ,
                (unsigned)p.Nz/2, 1); //rounds =1 ->2*1-1
    }
    else if( p.initne == "turbulence")
    {
        dg::BathRZ init0(16,16,Rmin,Zmin, 30.,5.,p.amp);
        helper = feltor.ds().fieldaligned().evaluate( init0, gaussianZ,
            (unsigned)p.Nz/2, 1);
    }
    else if( p.initne == "zonal")
    {
        dg::geo::ZonalFlow init0(p.amp, p.k_psi, gp, dg::geo::solovev::Psip(gp));
    }
    else
        std::cerr <<"WARNING: Unknown initial condition for Ni!\n";
    dg::geo::Nprofile(c.psip(), p.nprofileamp/c.psip()(c.R0(),0.), p.bgprofamp )},
    std::array<std::array<dg::DVec,2>,2> y0;
    y0[0][0] = y0[0][1] = y0[1][0] = y0[1][1] = dg::evaluate( prof, grid);
    dg::blas1::axpby( 1., helper, 1., y0[0][1]); //sum up background and perturbation
    dg::blas1::plus(y0[0][1], -1); //initialize ni-1
    if( p.initne == "turbulence" || p.initne == "zonal") //Cut initialization outside separatrix
    {
        dg::DVec damping = dg::evaluate( dg::geo::GaussianProfXDamping(
            mag.psip(), gp), grid);
        dg::blas1::pointwiseDot(damping, y0[0][1], y0[0][1]);
    }
    std::cout << "initialize ne" << std::endl;
    if( p.initphi == "zero")
        feltor.initializene( y0[0][1], y0[0][0]);
    else if( p.initphi == "balance")
        dg::blas1::copy( y0[0][1], y0[0][0]); //set n_e = N_i
    else
        std::cerr <<"WARNING: Unknown initial condition for phi!\n";

    dg::blas1::copy( 0., y0[1][0]); //set Ue = 0
    dg::blas1::copy( 0., y0[1][1]); //set Ui = 0
    ////////////map quantities to output/////////////////
    //since we map pointers we don't need to update those later
    std::map<std::string, const dg::DVec* > v4d;
    v4d["electrons"] = &y0[0][0], v4d["ions"] = &y0[0][1];
    v4d["Ue"] = &y0[1][0],        v4d["Ui"] = &y0[1][1];
    v4d["potential"] = &feltor.potential()[0];
    const feltor::Quantities& q = feltor.quantities();
    double dEdt = 0, accuracy = 0, dMdt = 0, accuracyM  = 0;
    std::map<std::string, const double*> v0d{
        {"energy", &q.energy}, {"ediff", &q.ediff},
        {"mass", &q.mass}, {"diff", &q.diff},
        {"Se", &q.S[0]}, {"Si", &q.S[1]}, {"Uperp", &q.Tperp},
        {"Upare", &q.Tpar[0]}, {"Upari", &q.Tpar[1]},
        {"dEdt", &dEdt}, {"accuracy", &accuracy},
        {"aligned", &q.aligned}
    };
    /////////////////////////////set up netcdf/////////////////////////////////////
    file::NC_Error_Handle err;
    int ncid;
    err = nc_create( argv[3],NC_NETCDF4|NC_CLOBBER, &ncid);
    err = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    err = nc_put_att_text( ncid, NC_GLOBAL, "geomfile", geom.size(), geom.data());
    int dim_ids[4], tvarID;
    {
        err = file::define_dimensions( ncid, dim_ids, &tvarID, grid_out);
        dg::geo::BFieldR fieldR(mag);
        dg::geo::BFieldZ fieldZ(mag);
        dg::geo::BFieldP fieldP(mag);

        dg::HVec vecR = dg::evaluate( fieldR, grid_out);
        dg::HVec vecZ = dg::evaluate( fieldZ, grid_out);
        dg::HVec vecP = dg::evaluate( fieldP, grid_out);
        int vecID[3];
        err = nc_def_var( ncid, "BR", NC_DOUBLE, 3, &dim_ids[1], &vecID[0]);
        err = nc_def_var( ncid, "BZ", NC_DOUBLE, 3, &dim_ids[1], &vecID[1]);
        err = nc_def_var( ncid, "BP", NC_DOUBLE, 3, &dim_ids[1], &vecID[2]);
        err = nc_enddef( ncid);
        err = nc_put_var_double( ncid, vecID[0], vecR.data());
        err = nc_put_var_double( ncid, vecID[1], vecZ.data());
        err = nc_put_var_double( ncid, vecID[2], vecP.data());
        err = nc_redef(ncid);
    }

    //field IDs
    int EtimeID, EtimevarID;
    err = file::define_time( ncid, "energy_time", &EtimeID, &EtimevarID);
    std::map<std::string, int> id0d, id4d;
    for( auto pair : v0d)
        err = nc_def_var( ncid, pair.first.data(), NC_DOUBLE, 1, &EtimeID, &id0d[pair.first]);
    for( auto pair : v4d)
        err = nc_def_var( ncid, pair.first.data(), NC_DOUBLE, 4, dim_ids, &id4d[pair.first]);
    err = nc_enddef(ncid);
    ///////////////////////////////////first output/////////////////////////
    double time = 0, dt_new = p.dt, dt = 0;
    std::cout << "First output ... \n";
    //first, update quantities in feltor
    {
        std::array<std::array<dg::DVec,2>,2> y1(y0);
        feltor( time, y0, y1);
    }
    q.display(std::cout);
    double energy0 = q.energy, mass0 = q.mass, E0 = energy0, M0 = mass0;
    size_t start[4] = {0, 0, 0, 0};
    size_t count[4] = {1, grid_out.Nz(), grid_out.n()*grid_out.Ny(),
        grid_out.n()*grid_out.Nx()};
    dg::DVec transferD( dg::evaluate(dg::zero, grid_out));
    dg::HVec transferH( dg::evaluate(dg::zero, grid_out));
    dg::IDMatrix project = dg::create::projection( grid_out, grid);
    for( auto pair : v4d)
    {
        dg::blas2::symv( project, pair.second, transferD);
        dg::assign( transferD, transferH);
        err = nc_put_vara_double( ncid, id4d[pair.first], start, count, transferH.data() );
    }
    err = nc_put_vara_double( ncid, tvarID, start, count, &time);
    err = nc_put_vara_double( ncid, EtimevarID, start, count, &time);
    size_t Estart[] = {0};
    size_t Ecount[] = {1};
    for( auto pair : v0d)
        err = nc_put_vara_double( ncid, id0d[pair.first], Estart, Ecount, pair.second);
    err = nc_close(ncid);
    std::cout << "First write successful!\n";
    ///////////////////////////////////////Timeloop/////////////////////////////////
    dg::Adaptive< dg::ARKStep<std::array<std::array<dg::DVec,2>,2>> > adaptive(
        "ARK-4-2-3", y0, y0[0][0].size(), p.eps_time);
    dg::Timer t;
    t.tic();
    unsigned step = 0;
    q.display(std::cout);
    for( unsigned i=1; i<=p.maxout; i++)
    {

        dg::Timer ti;
        ti.tic();
        for( unsigned j=0; j<p.itstp; j++)
        {
            try{
                do
                {
                    dt = dt_new;
                    adaptive.step( feltor, im, time, y0, time, y0, dt_new,
                        dg::pid_control, dg::l2norm, p.rtol, 1e-10);
                    if( adaptive.failed())
                        std::cout << "FAILED STEP! REPEAT!\n";
                }while ( adaptive.failed());
            }
            catch( dg::Fail& fail) {
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                err = nc_close(ncid);
                return -1;
            }
            step++;

            dEdt = (*v0d["energy"] - E0)/dt, dMdt = (*v0d["mass"] - M0)/dt;
            E0 = *v0d["energy"], M0 = *v0d["mass"];
            accuracy  = 2.*fabs( (dEdt - *v0d["ediff"])/( dEdt + *v0d["ediff"]));
            accuracyM = 2.*fabs( (dMdt - *v0d["diff"])/( dMdt + *v0d["diff"]));
            err = nc_open(argv[3], NC_WRITE, &ncid);
            Estart[0] = step;
            err = nc_put_vara_double( ncid, EtimevarID, Estart, Ecount, &time);
            for( auto pair : v0d)
                err = nc_put_vara_double( ncid, id0d[pair.first], Estart, Ecount, pair.second);

            q.display(std::cout);
            std::cout << "(m_tot-m_0)/m_0: "<< (*v0d["mass"]-mass0)/mass0<<"\t";
            std::cout << "(E_tot-E_0)/E_0: "<< (*v0d["energy"]-energy0)/energy0<<"\t";
            std::cout <<" d E/dt = " << dEdt
                      <<" Lambda = " << *v0d["ediff"]
                      <<" -> Accuracy: " << accuracy << "\n";
            std::cout <<" d M/dt = " << dMdt
                      <<" Lambda = " << *v0d["diff"]
                      <<" -> Accuracy: " << accuracyM << "\n";
            err = nc_close(ncid);

        }
        ti.toc();
        std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout
                  << " at time "<<time;
        std::cout << "\n\t Average time for one step: "
                  << ti.diff()/(double)p.itstp<<"s";
        ti.tic();
        //////////////////////////write fields////////////////////////
        start[0] = i;
        err = nc_open(argv[3], NC_WRITE, &ncid);
        err = nc_put_vara_double( ncid, tvarID, start, count, &time);
        for( auto pair : v4d)
        {
            dg::blas2::symv( project, pair.second, transferD);
            dg::assign( transferD, transferH);
            err = nc_put_vara_double( ncid, id4d[pair.first], start, count, transferH.data() );
        }
        err = nc_close(ncid);
        ti.toc();
        std::cout << "\n\t Time for output: "<<ti.diff()<<"s\n\n"<<std::flush;
    }
    t.toc();
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";

    return 0;

}
