#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <map>
#include <sstream>
#include <cmath>

#include "dg/file/nc_utilities.h"
#include "feltor.cuh"

int main( int argc, char* argv[])
{
    ////////////////////////Parameter initialisation//////////////////////////
    Json::Value js, gs;
    Json::CharReaderBuilder parser;
    parser["collectComments"] = false;
    std::string errs;
    if( argc != 4)
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] [outputfile]\n";
        return -1;
    }
    else
    {
        std::ifstream is(argv[1]);
        std::ifstream ks(argv[2]);
        parseFromStream( parser, is, &js, &errs); //read input without comments
        parseFromStream( parser, ks, &gs, &errs); //read input without comments
    }
    const feltor::Parameters p( js);
    const dg::geo::solovev::Parameters gp(gs);
    p.display( std::cout);
    gp.display( std::cout);
    std::string input = js.toStyledString(), geom = gs.toStyledString();
    ////////////////////////////////set up computations///////////////////////////
    double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscaleRp*gp.a;
    double Zmax=p.boxscaleZp*gp.a*gp.elongation;
    //Make grids
    dg::CylindricalGrid3d grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI,
        p.n, p.Nx, p.Ny, p.Nz, p.bcxN, p.bcyN, dg::PER);
    dg::CylindricalGrid3d grid_out( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI,
        p.n_out, p.Nx_out, p.Ny_out, p.Nz_out, p.bcxN, p.bcyN, dg::PER);
    dg::geo::TokamakMagneticField mag = dg::geo::createSolovevField(gp);

    //create RHS
    std::cout << "Constructing Explicit...\n";
    feltor::Explicit<dg::CylindricalGrid3d, dg::IDMatrix, dg::DMatrix, dg::DVec>
        feltor( grid, p, mag); //initialize before im!
    std::cout << "Constructing Implicit...\n";
    feltor::Implicit< dg::CylindricalGrid3d, dg::IDMatrix, dg::DMatrix, dg::DVec >
        im( grid, p, mag);
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
    //First the profile and the source (on the host since we want to output those)
    dg::HVec profile = dg::pullback( dg::geo::Compose<dg::LinearX>( mag.psip(), p.nprofamp/mag.psip()(mag.R0(), 0.), 0.), grid);
    dg::HVec xpoint_damping = dg::evaluate( dg::one, grid);
    if( gp.hasXpoint() )
        xpoint_damping = dg::pullback(
            dg::geo::ZCutter(-1.1*gp.elongation*gp.a), grid);
    dg::HVec source_damping = dg::pullback( dg::geo::TanhDamping(
        mag.psip(), -3.*p.alpha, p.alpha, -1.), grid);
    dg::blas1::pointwiseDot( xpoint_damping, source_damping, source_damping);
    if( p.omega_source != 0)
    {
        feltor.set_source( p.omega_source, profile, source_damping);
    }
    dg::HVec profile_damping = dg::pullback(dg::geo::TanhDamping(
        //first change coordinate from psi to (psi_0 - psip)/psi_0
        dg::geo::Compose<dg::LinearX>( mag.psip(), -1./mag.psip()(mag.R0(), 0.),1.),
        //then shift tanh
        p.rho_source-3.*p.alpha, p.alpha, -1.), grid);

    dg::blas1::pointwiseDot( xpoint_damping, profile_damping, profile_damping);
    dg::blas1::pointwiseDot( profile_damping, profile, profile);

    //Now perturbation
    dg::DVec ntilde = dg::evaluate(dg::zero,grid);
    if( p.initne == "blob" || p.initne == "straight blob")
    {
        dg::GaussianZ gaussianZ( 0., p.sigma_z*M_PI, 1);
        dg::Gaussian init0( gp.R_0+p.posX*gp.a, p.posY*gp.a, p.sigma, p.sigma, p.amp);
        if( p.initne == "blob")
            ntilde = feltor.ds().fieldaligned().evaluate( init0, gaussianZ,
                (unsigned)p.Nz/2, 3); //rounds =3 ->2*3-1
        if( p.initne == "straight blob")
            ntilde = feltor.ds().fieldaligned().evaluate( init0, gaussianZ,
                (unsigned)p.Nz/2, 1); //rounds =1 ->2*1-1
    }
    else if( p.initne == "turbulence")
    {
        dg::GaussianZ gaussianZ( 0., p.sigma_z*M_PI, 1);
        dg::BathRZ init0(16,16,Rmin,Zmin, 30.,5.,p.amp);
        ntilde = feltor.ds().fieldaligned().evaluate( init0, gaussianZ,
            (unsigned)p.Nz/2, 1);
    }
    else if( p.initne == "zonal")
    {
        dg::geo::ZonalFlow init0(mag.psip(), p.amp, 0., p.k_psi);
        ntilde = dg::pullback( init0, grid);
    }
    else
        std::cerr <<"WARNING: Unknown initial condition!\n";
    std::array<std::array<dg::DVec,2>,2> y0;
    y0[0][0] = y0[0][1] = y0[1][0] = y0[1][1] = dg::construct<dg::DVec>(profile);
    dg::blas1::axpby( 1., ntilde, 1., y0[0][0]); //sum up background and perturbation
    std::cout << "initialize ni" << std::endl;
    if( p.initphi == "zero")
        feltor.initializeni( y0[0][0], y0[0][1]);
    else if( p.initphi == "balance")
        dg::blas1::copy( y0[0][0], y0[0][1]); //set N_i = n_e
    else
        std::cerr <<"WARNING: Unknown initial condition for phi!\n";

    dg::blas1::copy( 0., y0[1][0]); //set Ue = 0
    dg::blas1::copy( 0., y0[1][1]); //set Ui = 0
    ////////////map quantities to output/////////////////
    //since we map pointers we don't need to update those later
    std::map<std::string, const dg::DVec* > v4d;
    v4d["electrons"] = &feltor.fields()[0][0], v4d["ions"] = &feltor.fields()[0][1];
    v4d["Ue"] = &feltor.fields()[1][0],        v4d["Ui"] = &feltor.fields()[1][1];
    v4d["potential"] = &feltor.potential()[0];
    v4d["induction"] = &feltor.induction();
    const feltor::Quantities& q = feltor.quantities();
    double dEdt = 0, accuracy = 0, dMdt = 0, accuracyM  = 0;
    std::map<std::string, const double*> v0d{
        {"energy", &q.energy}, {"ediff", &q.ediff},
        {"mass", &q.mass}, {"diff", &q.diff}, {"Apar", &q.Apar},
        {"Se", &q.S[0]}, {"Si", &q.S[1]}, {"Uperp", &q.Tperp},
        {"Upare", &q.Tpar[0]}, {"Upari", &q.Tpar[1]},
        {"dEdt", &dEdt}, {"accuracy", &accuracy},
        {"aligned", &q.aligned}
    };
    /////////////////////////////set up netcdf/////////////////////////////////////
    file::NC_Error_Handle err;
    int ncid;
    err = nc_create( argv[3],NC_NETCDF4|NC_CLOBBER, &ncid);
    err = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    err = nc_put_att_text( ncid, NC_GLOBAL, "geomfile", geom.size(), geom.data());
    int dim_ids[4], tvarID;
    {   //output 3d variables into file
        dg::geo::BFieldR fieldR(mag);
        dg::geo::BFieldZ fieldZ(mag);
        dg::geo::BFieldP fieldP(mag);

        dg::HVec vecR = dg::pullback( fieldR, grid_out);
        dg::HVec vecZ = dg::pullback( fieldZ, grid_out);
        dg::HVec vecP = dg::pullback( fieldP, grid_out);
        dg::HVec psip = dg::pullback( mag.psip(), grid_out);
        std::map<std::string, const dg::HVec*> v3d{
            {"BR", &vecR}, {"BZ", &vecZ}, {"BP", &vecP},
            {"Psip", &psip}, {"Nprof", &profile }, {"Source", &source_damping }
        };
        err = file::define_dimensions( ncid, dim_ids, &tvarID, grid_out);
        for( auto pair : v3d)
        {
            int vecID;
            err = nc_def_var( ncid, pair.first.data(), NC_DOUBLE, 3, &dim_ids[1], &vecID);
            err = nc_enddef( ncid);
            err = nc_put_var_double( ncid, vecID, pair.second->data());
            err = nc_redef(ncid);

        }
    }

    //field IDs
    int EtimeID, EtimevarID;
    err = file::define_time( ncid, "energy_time", &EtimeID, &EtimevarID);
    std::map<std::string, int> id0d, id4d;
    for( auto pair : v0d)
        err = nc_def_var( ncid, pair.first.data(), NC_DOUBLE, 1, &EtimeID, &id0d[pair.first]);
    for( auto pair : v4d)
        err = nc_def_var( ncid, pair.first.data(), NC_DOUBLE, 4, dim_ids, &id4d[pair.first]);
    err = nc_enddef(ncid);
    ///////////////////////////////////first output/////////////////////////
    double time = 0, dt_new = p.dt, dt = 0;
    std::cout << "First output ... \n";
    //first, update quantities in feltor
    {
        std::array<std::array<dg::DVec,2>,2> y1(y0);
        feltor( time, y0, y1);
        feltor.update_quantities();
    }
    q.display(std::cout);
    double energy0 = q.energy, mass0 = q.mass, E0 = energy0, M0 = mass0;
    size_t start[4] = {0, 0, 0, 0};
    size_t count[4] = {1, grid_out.Nz(), grid_out.n()*grid_out.Ny(),
        grid_out.n()*grid_out.Nx()};
    dg::DVec transferD( dg::evaluate(dg::zero, grid_out));
    dg::HVec transferH( dg::evaluate(dg::zero, grid_out));
    dg::IDMatrix project = dg::create::projection( grid_out, grid);
    for( auto pair : v4d)
    {
        dg::blas2::symv( project, *pair.second, transferD);
        dg::assign( transferD, transferH);
        err = nc_put_vara_double( ncid, id4d[pair.first], start, count, transferH.data() );
    }
    err = nc_put_vara_double( ncid, tvarID, start, count, &time);
    err = nc_put_vara_double( ncid, EtimevarID, start, count, &time);
    size_t Estart[] = {0};
    size_t Ecount[] = {1};
    for( auto pair : v0d)
        err = nc_put_vara_double( ncid, id0d[pair.first], Estart, Ecount, pair.second);
    err = nc_close(ncid);
    std::cout << "First write successful!\n";
    ///////////////////////////////////////Timeloop/////////////////////////////////
    dg::Adaptive< dg::ARKStep<std::array<std::array<dg::DVec,2>,2>> > adaptive(
        "ARK-4-2-3", y0, y0[0][0].size(), p.eps_time);
    dg::Timer t;
    t.tic();
    unsigned step = 0;
    q.display(std::cout);
    for( unsigned i=1; i<=p.maxout; i++)
    {

        dg::Timer ti;
        ti.tic();
        for( unsigned j=0; j<p.itstp; j++)
        {
            try{
                do
                {
                    dt = dt_new;
                    adaptive.step( feltor, im, time, y0, time, y0, dt_new,
                        dg::pid_control, dg::l2norm, p.rtol, 1e-10);
                    if( adaptive.failed())
                        std::cout << "FAILED STEP! REPEAT!\n";
                }while ( adaptive.failed());
            }
            catch( dg::Fail& fail) {
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                err = nc_close(ncid);
                return -1;
            }
            step++;

            feltor.update_quantities();
            dEdt = (*v0d["energy"] - E0)/dt, dMdt = (*v0d["mass"] - M0)/dt;
            E0 = *v0d["energy"], M0 = *v0d["mass"];
            accuracy  = 2.*fabs( (dEdt - *v0d["ediff"])/( dEdt + *v0d["ediff"]));
            accuracyM = 2.*fabs( (dMdt - *v0d["diff"])/( dMdt + *v0d["diff"]));
            err = nc_open(argv[3], NC_WRITE, &ncid);
            Estart[0] = step;
            err = nc_put_vara_double( ncid, EtimevarID, Estart, Ecount, &time);
            for( auto pair : v0d)
                err = nc_put_vara_double( ncid, id0d[pair.first], Estart, Ecount, pair.second);

            q.display(std::cout);
            std::cout << "(m_tot-m_0)/m_0: "<< (*v0d["mass"]-mass0)/mass0<<"\t";
            std::cout << "(E_tot-E_0)/E_0: "<< (*v0d["energy"]-energy0)/energy0<<"\t";
            std::cout <<" d E/dt = " << dEdt
                      <<" Lambda = " << *v0d["ediff"]
                      <<" -> Accuracy: " << accuracy << "\n";
            std::cout <<" d M/dt = " << dMdt
                      <<" Lambda = " << *v0d["diff"]
                      <<" -> Accuracy: " << accuracyM << "\n";
            err = nc_close(ncid);

        }
        ti.toc();
        std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout
                  << " at time "<<time;
        std::cout << "\n\t Average time for one step: "
                  << ti.diff()/(double)p.itstp<<"s";
        ti.tic();
        //////////////////////////write fields////////////////////////
        start[0] = i;
        err = nc_open(argv[3], NC_WRITE, &ncid);
        err = nc_put_vara_double( ncid, tvarID, start, count, &time);
        for( auto pair : v4d)
        {
            dg::blas2::symv( project, *pair.second, transferD);
            dg::assign( transferD, transferH);
            err = nc_put_vara_double( ncid, id4d[pair.first], start, count, transferH.data() );
        }
        err = nc_close(ncid);
        ti.toc();
        std::cout << "\n\t Time for output: "<<ti.diff()<<"s\n\n"<<std::flush;
    }
    t.toc();
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";

    return 0;

}
