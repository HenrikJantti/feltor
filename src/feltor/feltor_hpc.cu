#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <map>
#include <sstream>
#include <cmath>
#include <csignal>

#ifdef FELTOR_MPI
#include <mpi.h>
#endif //FELTOR_MPI

#include "dg/file/nc_utilities.h"
#include "feltor.h"
#include "implicit.h"

#ifdef FELTOR_MPI
using HVec = dg::MHVec;
using DVec = dg::MDVec;
using HMatrix = dg::MHMatrix;
using DMatrix = dg::MDMatrix;
using IDMatrix = dg::MIDMatrix;
using IHMatrix = dg::MIHMatrix;
using Geometry = dg::CylindricalMPIGrid3d;
#define MPI_OUT if(rank==0)
#else //FELTOR_MPI
using HVec = dg::HVec;
using DVec = dg::DVec;
using HMatrix = dg::HMatrix;
using DMatrix = dg::DMatrix;
using IDMatrix = dg::IDMatrix;
using IHMatrix = dg::IHMatrix;
using Geometry = dg::CylindricalGrid3d;
#define MPI_OUT
#endif //FELTOR_MPI

#include "init.h"
#include "feltordiag.h"

#ifdef FELTOR_MPI
//ATTENTION: in slurm should be used with --signal=SIGINT@30 (<signal>@<time in seconds>)
void sigterm_handler(int signal)
{
    int rank;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    std::cout << " pid "<<rank<<" sigterm_handler, got signal " << signal << std::endl;
    MPI_Finalize();
    exit(signal);
}
#endif //FELTOR_MPI

int main( int argc, char* argv[])
{
#ifdef FELTOR_MPI
    ////////////////////////////////setup MPI///////////////////////////////
#ifdef _OPENMP
    int provided;
    MPI_Init_thread(&argc, &argv, MPI_THREAD_FUNNELED, &provided);
    assert( provided >= MPI_THREAD_FUNNELED && "Threaded MPI lib required!\n");
#else
    MPI_Init(&argc, &argv);
#endif
    int periods[3] = {false, false, true}; //non-, non-, periodic
    int rank, size;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    MPI_Comm_size( MPI_COMM_WORLD, &size);
#if THRUST_DEVICE_SYSTEM==THRUST_DEVICE_SYSTEM_CUDA
    int num_devices=0;
    hipGetDeviceCount(&num_devices);
    if(num_devices==0){
        std::cerr << "No CUDA capable devices found"<<std::endl;
        return -1;
    }
    int device = rank % num_devices; //assume # of gpus/node is fixed
    std::cout << "# Rank "<<rank<<" computes with device "<<device<<" !"<<std::endl;
    hipSetDevice( device);
#endif//THRUST_DEVICE_SYSTEM==THRUST_DEVICE_SYSTEM_CUDA
    int np[3];
    if(rank==0)
    {
        int num_threads = 1;
#ifdef _OPENMP
        num_threads = omp_get_max_threads( );
#endif //omp
        std::cin>> np[0] >> np[1] >>np[2];
        std::cout << "# Computing with "
                  << np[0]<<" x "<<np[1]<<" x "<<np[2] << " processes x "
                  << num_threads<<" threads = "
                  <<size*num_threads<<" total"<<std::endl;
;
        assert( size == np[0]*np[1]*np[2] &&
        "Partition needs to match total number of processes!");
    }
    MPI_Bcast( np, 3, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Comm comm;
    MPI_Cart_create( MPI_COMM_WORLD, 3, np, periods, true, &comm);
    ////////////////////////////// Install signal handler ///////////////////
    std::signal(SIGINT, sigterm_handler);
    std::signal(SIGTERM, sigterm_handler);
#endif //FELTOR_MPI
    ////////////////////////Parameter initialisation//////////////////////////
    Json::Value js, gs;
    Json::CharReaderBuilder parser;
    parser["collectComments"] = false;
    std::string errs;
    if( argc != 4 && argc != 5)
    {
        MPI_OUT std::cerr << "ERROR: Wrong number of arguments!\nUsage: "
                << argv[0]<<" [input.json] [geometry.json] [output.nc]\n OR \n"
                << argv[0]<<" [input.json] [geometry.json] [output.nc] [initial.nc] \n";
        return -1;
    }
    else
    {
        std::ifstream is(argv[1]);
        std::ifstream ks(argv[2]);
        parseFromStream( parser, is, &js, &errs); //read input without comments
        parseFromStream( parser, ks, &gs, &errs); //read input without comments
    }
    const feltor::Parameters p( js);
    const dg::geo::solovev::Parameters gp(gs);
    MPI_OUT p.display( std::cout);
    MPI_OUT gp.display( std::cout);
    std::string input = js.toStyledString(), geom = gs.toStyledString();
    ////////////////////////////////set up computations///////////////////////////
    double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscaleRp*gp.a;
    double Zmax=p.boxscaleZp*gp.a*gp.elongation;
    //Make grids
    Geometry grid( Rmin, Rmax, Zmin, Zmax, 0, 2.*M_PI,
        p.n, p.Nx, p.Ny, p.symmetric ? 1 : p.Nz, p.bcxN, p.bcyN, dg::PER
        #ifdef FELTOR_MPI
        , comm
        #endif //FELTOR_MPI
        );
    Geometry g3d_out( Rmin, Rmax, Zmin, Zmax, 0, 2.*M_PI,
        p.n_out, p.Nx_out, p.Ny_out, p.symmetric ? 1 : p.Nz_out, p.bcxN, p.bcyN, dg::PER
        #ifdef FELTOR_MPI
        , comm
        #endif //FELTOR_MPI
        );
    std::unique_ptr<typename Geometry::perpendicular_grid> g2d_out_ptr  ( dynamic_cast<typename Geometry::perpendicular_grid*>( g3d_out.perp_grid()));
#ifdef FELTOR_MPI
    unsigned local_size2d = g2d_out_ptr->local().size();
#else
    unsigned local_size2d = g2d_out_ptr->size();
#endif

    dg::geo::TokamakMagneticField mag = dg::geo::createSolovevField(gp);
    mag = dg::geo::createModifiedSolovevField(gp, (1.-p.rho_damping)*mag.psip()(mag.R0(),0.), p.alpha_mag);

    //create RHS
    MPI_OUT std::cout << "Constructing Explicit...\n";
    feltor::Explicit< Geometry, IDMatrix, DMatrix, DVec> feltor( grid, p, mag);
    MPI_OUT std::cout << "Constructing Implicit...\n";
    feltor::Implicit< Geometry, IDMatrix, DMatrix, DVec> im( grid, p, mag);
    MPI_OUT std::cout << "Done!\n";

    // helper variables for output computations
    std::map<std::string, dg::Simpsons<HVec>> time_integrals;
    dg::Average<HVec> toroidal_average( g3d_out, dg::coo3d::z);
    dg::MultiMatrix<HMatrix,HVec> projectH = dg::create::fast_projection( grid, p.cx, p.cy, dg::normed);
    dg::MultiMatrix<DMatrix,DVec> projectD = dg::create::fast_projection( grid, p.cx, p.cy, dg::normed);
    HVec transferH( dg::evaluate(dg::zero, g3d_out));
    DVec transferD( dg::evaluate(dg::zero, g3d_out));
    HVec transferH2d = dg::evaluate( dg::zero, *g2d_out_ptr);
    DVec transferD2d = dg::evaluate( dg::zero, *g2d_out_ptr);
    HVec resultH = dg::evaluate( dg::zero, grid);
    DVec resultD = dg::evaluate( dg::zero, grid);

    std::array<DVec, 3> gradPsip;
    gradPsip[0] =  dg::evaluate( mag.psipR(), grid);
    gradPsip[1] =  dg::evaluate( mag.psipZ(), grid);
    gradPsip[2] =  resultD; //zero
    feltor::Variables var = {
        feltor, p, gradPsip, gradPsip
    };
    // the vector ids
    std::map<std::string, int> id3d, id4d;

    double dEdt = 0, accuracy = 0;
    double E0 = 0.;

    /// //////////////////The initial field///////////////////////////////////////////
    double time = 0.;
    std::array<std::array<DVec,2>,2> y0;
    feltor::Initialize init( p, gp, mag);
    if( argc == 4)
        y0 = init.init_from_parameters(feltor, grid);
    if( argc == 5)
        y0 = init.init_from_file(argv[4], grid, time);
    feltor.set_source( init.profile(grid), p.omega_source, init.source_damping(grid));

    /// //////////////////////////set up netcdf/////////////////////////////////////
    file::NC_Error_Handle err;
    std::string file_name = argv[3];
    int ncid=-1;
    MPI_OUT err = nc_create( file_name.data(), NC_NETCDF4|NC_CLOBBER, &ncid);
    /// Set global attributes
    std::map<std::string, std::string> att;
    att["title"] = "Output file of feltor/src/feltor_hpc.cu";
    att["Conventions"] = "CF-1.7";
    ///Get local time and begin file history
    auto ttt = std::time(nullptr);
    auto tm = *std::localtime(&ttt);

    std::ostringstream oss;
    ///time string  + program-name + args
    oss << std::put_time(&tm, "%Y-%m-%d %H:%M:%S");
    for( int i=0; i<argc; i++) oss << " "<<argv[i];
    att["history"] = oss.str();
    att["comment"] = "Find more info in feltor/src/feltor.tex";
    att["source"] = "FELTOR";
    att["references"] = "https://github.com/feltor-dev/feltor";
    att["inputfile"] = input;
    att["geomfile"] = geom;
    for( auto pair : att)
        MPI_OUT err = nc_put_att_text( ncid, NC_GLOBAL,
            pair.first.data(), pair.second.size(), pair.second.data());

    // Define dimensions (t,z,y,x)
    int dim_ids[4], tvarID;
#ifdef FELTOR_MPI
    MPI_OUT err = file::define_dimensions( ncid, dim_ids, &tvarID, g3d_out.global());
#else //FELTOR_MPI
    err = file::define_dimensions( ncid, dim_ids, &tvarID, g3d_out);
#endif //FELTOR_MPI
    int dim_ids3d[3] = {dim_ids[0], dim_ids[2], dim_ids[3]};

    //create & output static 3d variables into file
    for ( auto& record : feltor::diagnostics3d_static_list)
    {
        int vecID;
        MPI_OUT err = nc_def_var( ncid, record.name.data(), NC_DOUBLE, 3,
            &dim_ids[1], &vecID);
        MPI_OUT err = nc_put_att_text( ncid, vecID,
            "long_name", record.long_name.size(), record.long_name.data());
        MPI_OUT err = nc_enddef( ncid);
        MPI_OUT std::cout << "Computing "<<record.name<<"\n";
        record.function( resultH, var, grid, gp, mag);
        dg::blas2::symv( projectH, resultH, transferH);
        file::write_static3d( ncid, vecID, transferH, g3d_out);
        MPI_OUT err = nc_redef(ncid);
    }

    //Create field IDs
    for( auto& record : feltor::diagnostics3d_list)
    {
        std::string name = record.name;
        std::string long_name = record.long_name;
        id4d[name] = 0;//creates a new id4d entry for all processes
        MPI_OUT err = nc_def_var( ncid, name.data(), NC_DOUBLE, 4, dim_ids,
            &id4d.at(name));
        MPI_OUT err = nc_put_att_text( ncid, id4d.at(name), "long_name", long_name.size(),
            long_name.data());
    }
    for( auto& record : feltor::diagnostics2d_list)
    {
        std::string name = record.name + "_ta2d";
        std::string long_name = record.long_name + " (Toroidal average)";
        id3d[name] = 0;//creates a new id3d entry for all processes
        MPI_OUT err = nc_def_var( ncid, name.data(), NC_DOUBLE, 3, dim_ids3d,
            &id3d.at(name));
        MPI_OUT err = nc_put_att_text( ncid, id3d.at(name), "long_name", long_name.size(),
            long_name.data());

        name = record.name + "_2d";
        long_name = record.long_name + " (Evaluated on phi = 0 plane)";
        id3d[name] = 0;
        MPI_OUT err = nc_def_var( ncid, name.data(), NC_DOUBLE, 3, dim_ids3d,
            &id3d.at(name));
        MPI_OUT err = nc_put_att_text( ncid, id3d.at(name), "long_name", long_name.size(),
            long_name.data());
    }
    MPI_OUT err = nc_enddef(ncid);
    ///////////////////////////////////first output/////////////////////////
    MPI_OUT std::cout << "First output ... \n";
    //first, update feltor (to get potential etc.)
    {
        std::array<std::array<DVec,2>,2> y1(y0);
        try{
            feltor( time, y0, y1);
        } catch( dg::Fail& fail) {
            MPI_OUT std::cerr << "CG failed to converge in first step to "
                              <<fail.epsilon()<<"\n";
            MPI_OUT err = nc_close(ncid);
            return -1;
        }
    }

    size_t start = 0, count = 1;
    MPI_OUT err = nc_put_vara_double( ncid, tvarID, &start, &count, &time);
    for( auto& record : feltor::diagnostics3d_list)
    {
        record.function( resultD, var);
        dg::blas2::symv( projectD, resultD, transferD);
        dg::assign( transferD, transferH);
        file::write_dynamic3d( ncid, id4d.at(record.name), start, transferH, g3d_out);
    }
    for( auto& record : feltor::diagnostics2d_list)
    {
        dg::Timer tti;
        tti.tic();
        record.function( resultD, var);
        dg::blas2::symv( projectD, resultD, transferD);

        //toroidal average
        std::string name = record.name + "_ta2d";
        dg::assign( transferD, transferH);
        toroidal_average( transferH, transferH2d, false);
        //create and init Simpsons for time integrals
        if( record.integral) time_integrals[name].init( time, transferH2d);
        tti.toc();
        MPI_OUT std::cout<< name << " Computing average took "<<tti.diff()<<"\n";
        tti.tic();
#ifdef FELTOR_MPI
        //only the globally first slice should write
        if( g3d_out.local().z0() - g3d_out.global().z0() < 1e-14)
#endif //FELTOR_MPI
            file::write_dynamic2d( ncid, id3d.at(name), start, transferH2d, *g2d_out_ptr);
        tti.toc();
        MPI_OUT std::cout<< name << " 2d output took "<<tti.diff()<<"\n";
        tti.tic();

        // and a slice
        name = record.name + "_2d";
        feltor::slice_vector3d( transferD, transferD2d, local_size2d);
        dg::assign( transferD2d, transferH2d);
        if( record.integral) time_integrals[name].init( time, transferH2d);
#ifdef FELTOR_MPI
        //only the globally first slice should write
        if( g3d_out.local().z0() - g3d_out.global().z0() < 1e-14)
#endif //FELTOR_MPI
            file::write_dynamic2d( ncid, id3d.at(name), start, transferH2d, *g2d_out_ptr);
        tti.toc();
        MPI_OUT std::cout<< name << " 2d output took "<<tti.diff()<<"\n";
    }
    MPI_OUT err = nc_close(ncid);
    MPI_OUT std::cout << "First write successful!\n";
    ///////////////////////////////////////Timeloop/////////////////////////////////
    dg::Karniadakis< std::array<std::array<DVec,2>,2 >,
        feltor::FeltorSpecialSolver<
            Geometry, IDMatrix, DMatrix, DVec>
        > karniadakis( grid, p, mag);
    karniadakis.init( feltor, im, time, y0, p.dt);
    dg::Timer t;
    t.tic();
    unsigned step = 0;
    for( unsigned i=1; i<=p.maxout; i++)
    {

        dg::Timer ti;
        ti.tic();
        for( unsigned j=0; j<p.itstp; j++)
        {
            double previous_time = time;
            for( unsigned k=0; k<p.inner_loop; k++)
            {
                try{
                    karniadakis.step( feltor, im, time, y0);
                }
                catch( dg::Fail& fail) {
                    MPI_OUT std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                    MPI_OUT std::cerr << "Does Simulation respect CFL condition?\n";
                    return -1;
                }
                step++;
            }
            dg::Timer tti;
            tti.tic();
            double deltat = time - previous_time;
            double energy = 0, ediff = 0.;
            for( auto& record : feltor::diagnostics2d_list)
            {
                if( std::find( feltor::energies.begin(), feltor::energies.end(), record.name) != feltor::energies.end())
                {
                    record.function( resultD, var);
                    energy += dg::blas1::dot( resultD, feltor.vol3d());
                }
                if( std::find( feltor::energy_diff.begin(), feltor::energy_diff.end(), record.name) != feltor::energy_diff.end())
                {
                    record.function( resultD, var);
                    ediff += dg::blas1::dot( resultD, feltor.vol3d());
                }
                if( record.integral)
                {
                    record.function( resultD, var);
                    dg::blas2::symv( projectD, resultD, transferD);
                    //toroidal average and add to time integral
                    dg::assign( transferD, transferH);
                    toroidal_average( transferH, transferH2d, false);
                    time_integrals.at(record.name+"_ta2d").add( time, transferH2d);

                    // 2d data of plane varphi = 0
                    feltor::slice_vector3d( transferD, transferD2d, local_size2d);
                    dg::assign( transferD2d, transferH2d);
                    time_integrals.at(record.name+"_2d").add( time, transferH2d);
                }

            }

            dEdt = (energy - E0)/deltat;
            E0 = energy;
            accuracy  = 2.*fabs( (dEdt - ediff)/( dEdt + ediff));

            MPI_OUT std::cout << "\tTime "<<time<<"\n";
            MPI_OUT std::cout <<"\td E/dt = " << dEdt
                      <<" Lambda = " << ediff
                      <<" -> Accuracy: " << accuracy << "\n";
            //----------------Test if induction equation holds
            if( p.beta != 0)
            {
                dg::blas1::pointwiseDot(
                    feltor.density(0), feltor.velocity(0), resultD);
                dg::blas1::pointwiseDot( p.beta,
                    feltor.density(1), feltor.velocity(1), -p.beta, resultD);
                double norm  = dg::blas2::dot( resultD, feltor.vol3d(), resultD);
                dg::blas1::axpby( -1., feltor.lapMperpA(), 1., resultD);
                double error = dg::blas2::dot( resultD, feltor.vol3d(), resultD);
                MPI_OUT std::cout << "\tRel. Error Induction "<<sqrt(error/norm) <<"\n";
            }
            tti.toc();
            MPI_OUT std::cout << " Time for internal diagnostics "<<tti.diff()<<"s\n";
        }
        ti.toc();
        MPI_OUT std::cout << "\n\t Step "<<step <<" of "
                    << p.inner_loop*p.itstp*p.maxout << " at time "<<time;
        MPI_OUT std::cout << "\n\t Average time for one step: "
                    << ti.diff()/(double)p.itstp/(double)p.inner_loop<<"s";
        ti.tic();
        //////////////////////////write fields////////////////////////
        start = i;
        MPI_OUT err = nc_open(file_name.data(), NC_WRITE, &ncid);
        MPI_OUT err = nc_put_vara_double( ncid, tvarID, &start, &count, &time);
        for( auto& record : feltor::diagnostics3d_list)
        {
            record.function( resultD, var);
            dg::blas2::symv( projectD, resultD, transferD);
            dg::assign( transferD, transferH);
            file::write_dynamic3d( ncid, id4d.at(record.name), start, transferH, g3d_out);
        }
        for( auto& record : feltor::diagnostics2d_list)
        {
            if(record.integral) // we already computed the output...
            {
                std::string name = record.name+"_ta2d";
                transferH2d = time_integrals.at(name).get_integral();
                time_integrals.at(name).flush();
#ifdef FELTOR_MPI
                //only the globally first slice should write
                if( g3d_out.local().z0() - g3d_out.global().z0() < 1e-14)
#endif //FELTOR_MPI
                    file::write_dynamic2d( ncid, id3d.at(name), start, transferH2d, *g2d_out_ptr);

                name = record.name+"_2d";
                transferH2d = time_integrals.at(name).get_integral( );
                time_integrals.at(name).flush( );
#ifdef FELTOR_MPI
                //only the globally first slice should write
                if( g3d_out.local().z0() - g3d_out.global().z0() < 1e-14)
#endif //FELTOR_MPI
                    file::write_dynamic2d( ncid, id3d.at(name), start, transferH2d, *g2d_out_ptr);
            }
            else //manage the time integrators
            {
                record.function( resultD, var);
                dg::blas2::symv( projectD, resultD, transferD);

                std::string name = record.name+"_ta2d";
                dg::assign( transferD, transferH);
                toroidal_average( transferH, transferH2d, false);
#ifdef FELTOR_MPI
                //only the globally first slice should write
                if( g3d_out.local().z0() - g3d_out.global().z0() < 1e-14)
#endif //FELTOR_MPI
                    file::write_dynamic2d( ncid, id3d.at(name), start, transferH2d, *g2d_out_ptr);

                // 2d data of plane varphi = 0
                name = record.name+"_2d";
                feltor::slice_vector3d( transferD, transferD2d, local_size2d);
                dg::assign( transferD2d, transferH2d);
#ifdef FELTOR_MPI
                //only the globally first slice should write
                if( g3d_out.local().z0() - g3d_out.global().z0() < 1e-14)
#endif //FELTOR_MPI
                    file::write_dynamic2d( ncid, id3d.at(name), start, transferH2d, *g2d_out_ptr);
            }
        }
        MPI_OUT err = nc_close(ncid);
        ti.toc();
        MPI_OUT std::cout << "\n\t Time for output: "<<ti.diff()<<"s\n\n"<<std::flush;
    }
    t.toc();
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    MPI_OUT std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    MPI_OUT std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    MPI_OUT std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout/p.inner_loop<<"s/step\n";
#ifdef FELTOR_MPI
    MPI_Finalize();
#endif //FELTOR_MPI

    return 0;

}
