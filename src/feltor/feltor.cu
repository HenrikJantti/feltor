#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>

#include "draw/host_window.h"

#include "feltor.h"
#include "implicit.h"

using HVec = dg::HVec;
using DVec = dg::DVec;
using DMatrix = dg::DMatrix;
using IDMatrix = dg::IDMatrix;
using IHMatrix = dg::IHMatrix;
using Geometry = dg::CylindricalGrid3d;
#define MPI_OUT

#include "init.h"
#include "feltordiag.h"

int main( int argc, char* argv[])
{
    ////Parameter initialisation ////////////////////////////////////////////
    Json::Value js, gs;
    if( argc == 1)
    {
        std::ifstream is("input.json");
        std::ifstream ks("geometry_params.json");
        is >> js;
        ks >> gs;
    }
    else if( argc == 3)
    {
        std::ifstream is(argv[1]);
        std::ifstream ks(argv[2]);
        is >> js;
        ks >> gs;
    }
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "
                  << argv[0]<<" [inputfile] [geomfile] \n";
        return -1;
    }
    const feltor::Parameters p( js);
    const dg::geo::solovev::Parameters gp(gs);
    p.display( std::cout);
    gp.display( std::cout);
    /////////////////////////////////////////////////////////////////////////
    double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscaleRp*gp.a;
    double Zmax=p.boxscaleZp*gp.a*gp.elongation;
    //Make grid
    dg::CylindricalGrid3d grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI,
        p.n, p.Nx, p.Ny, p.symmetric ? 1 : p.Nz, p.bcxN, p.bcyN, dg::PER);
    dg::geo::TokamakMagneticField mag = dg::geo::createSolovevField(gp);
    //Wall damping has to be constructed before modification (!)
    HVec damping_profile = dg::evaluate( dg::zero, grid);
    if( p.damping_alpha > 0.)
    {
        damping_profile = feltor::wall_damping( grid, p, gp, mag);
        double RO=mag.R0(), ZO=0.;
        dg::geo::findOpoint( mag.get_psip(), RO, ZO);
        double psipO = mag.psip()( RO, ZO);
        double damping_psi0p = (1.-p.damping_boundary*p.damping_boundary)*psipO;
        double damping_alphap = -(2.*p.damping_boundary+p.damping_alpha)*p.damping_alpha*psipO;
        mag = dg::geo::createModifiedSolovevField(gp, damping_psi0p+damping_alphap/2.,
                fabs(damping_alphap/2.), ((psipO>0)-(psipO<0)));
    }
    if( p.periodify)
        mag = dg::geo::periodify( mag, Rmin, Rmax, Zmin, Zmax, dg::NEU, dg::NEU);

    //create RHS
    //std::cout << "Constructing RHS...\n";
    //feltor::Explicit<Geometry, IDMatrix, DMatrix, DVec> feltor( grid, p, mag, true);
    std::cout << "Constructing Explicit...\n";
    feltor::Explicit<Geometry, IDMatrix, DMatrix, DVec> feltor( grid, p, mag, false);
    std::cout << "Constructing Implicit...\n";
    feltor::Implicit<Geometry, IDMatrix, DMatrix, DVec> im( grid, p, mag);
    std::cout << "Done!\n";

    DVec result = dg::evaluate( dg::zero, grid);
    /// Construct feltor::Variables object for diagnostics
    std::array<DVec, 3> gradPsip;
    gradPsip[0] =  dg::evaluate( mag.psipR(), grid);
    gradPsip[1] =  dg::evaluate( mag.psipZ(), grid);
    gradPsip[2] =  result; //zero
    feltor::Variables var = {
        feltor, p,gp,mag, gradPsip, gradPsip
    };



    /////////////////////The initial field///////////////////////////////////////////
    double time = 0.;
    std::array<std::array<DVec,2>,2> y0;
    try{
        y0 = feltor::initial_conditions.at(p.initne)( feltor, grid, p,gp,mag );
    }catch ( std::out_of_range& error){
        std::cerr << "Warning: initne parameter '"<<p.initne<<"' not recognized! Is there a spelling error? I assume you do not want to continue with the wrong initial condition so I exit! Bye Bye :)\n";
        return -1;
    }

    bool fixed_profile;
    HVec profile = dg::evaluate( dg::zero, grid);
    HVec source_profile;
    try{
        source_profile = feltor::source_profiles.at(p.source_type)(
            fixed_profile, profile, grid, p, gp, mag);
    }catch ( std::out_of_range& error){
        std::cerr << "Warning: source_type parameter '"<<p.source_type<<"' not recognized! Is there a spelling error? I assume you do not want to continue with the wrong source so I exit! Bye Bye :)\n";
        return -1;
    }

    feltor.set_source( fixed_profile, dg::construct<DVec>(profile),
        p.source_rate, dg::construct<DVec>(source_profile),
        p.damping_rate, dg::construct<DVec>(damping_profile)
    );


    ////////////////////////create timer and timestepper
    //
    dg::Timer t;
    unsigned step = 0;
    dg::Karniadakis< std::array<std::array<dg::DVec,2>,2 >,
        feltor::FeltorSpecialSolver<
            Geometry, IDMatrix, DMatrix, DVec>
        > karniadakis( grid, p, mag);
    //unsigned mMax = 3, restart = 3, max_iter = 100;
    //double damping = 1e-3;
    //dg::BDF< std::array<std::array<dg::DVec,2>,2 >,
    //    dg::AndersonSolver< std::array<std::array<dg::DVec,2>,2> >
    //    > bdf( 3, y0, mMax, p.rtol, max_iter, damping, restart);
    //dg::AdamsBashforth< std::array<std::array<dg::DVec,2>,2 >
    //    > bdf( 3, y0);

    std::cout << "Initialize Timestepper" << std::endl;
    karniadakis.init( feltor, im, time, y0, p.dt);
    //bdf.init( feltor, time, y0, p.dt);
    std::cout << "Done!" << std::endl;

    std::map<std::string, const dg::DVec* > v4d;
    v4d["ne-1 / "] = &y0[0][0],  v4d["ni-1 / "] = &y0[0][1];
    v4d["Ue / "]   = &feltor.velocity(0), v4d["Ui / "]   = &feltor.velocity(1);
    v4d["Ome / "] = &feltor.potential(0); v4d["Apar / "] = &feltor.induction();
    double dEdt = 0, accuracy = 0;
    double E0 = 0.;
    /////////////////////////set up transfer for glfw
    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual), avisual(hvisual);
    dg::IHMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);

    /////////glfw initialisation ////////////////////////////////////////////
    //
    std::stringstream title;
    std::ifstream is( "window_params.js");
    is >> js;
    is.close();
    unsigned red = js.get("reduction", 1).asUInt();
    double rows = js["rows"].asDouble(), cols = p.Nz/red+1,
           width = js["width"].asDouble(), height = js["height"].asDouble();
    if ( p.symmetric ) cols = rows, rows = 1;
    GLFWwindow* w = draw::glfwInitAndCreateWindow( cols*width, rows*height, "");
    draw::RenderHostData render(rows, cols);

    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    dg::Average<dg::HVec> toroidal_average( grid, dg::coo3d::z);
    title << std::setprecision(2) << std::scientific;
    while ( !glfwWindowShouldClose( w ))
    {
        title << std::fixed;
        title << "t = "<<time<<"   ";
        for( auto pair : v4d)
        {
            if(pair.first == "Ome / ")
            {
                dg::assign( feltor.lapMperpP(0), hvisual);
                dg::assign( *pair.second, hvisual);
            }
            else if(pair.first == "ne-1 / " || pair.first == "ni-1 / ")
            {
                dg::assign( *pair.second, hvisual);
                //dg::blas1::axpby( 1., hvisual, -1., profile, hvisual);
            }
            else
                dg::assign( *pair.second, hvisual);
            dg::blas2::gemv( equi, hvisual, visual);
            colors.scalemax() = dg::blas1::reduce(
                visual, 0., dg::AbsMax<double>() );
            colors.scalemin() = -colors.scalemax();
            title <<pair.first << colors.scalemax()<<"   ";
            if ( p.symmetric )
                render.renderQuad( hvisual, grid.n()*grid.Nx(),
                                            grid.n()*grid.Ny(), colors);
            else
            {
                for( unsigned k=0; k<p.Nz/red;k++)
                {
                    unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
                    dg::HVec part( visual.begin() +  k*red   *size,
                                   visual.begin() + (k*red+1)*size);
                    render.renderQuad( part, grid.n()*grid.Nx(),
                                             grid.n()*grid.Ny(), colors);
                }
                dg::blas1::scal(avisual,0.);
                toroidal_average(visual,avisual);
                render.renderQuad( avisual, grid.n()*grid.Nx(),
                                            grid.n()*grid.Ny(), colors);
            }
        }
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step
        t.tic();
        for( unsigned i=0; i<p.itstp; i++)
        {
            double previous_time = time;
            for( unsigned k=0; k<p.inner_loop; k++)
            {
                try{
                    karniadakis.step( feltor, im, time, y0);
                    //bdf.step( feltor, time, y0);
                }
                catch( dg::Fail& fail) {
                    std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                    std::cerr << "Does Simulation respect CFL condition?\n";
                    glfwSetWindowShouldClose( w, GL_TRUE);
                    break;
                }
                step++;
            }
            double deltat = time - previous_time;
            double energy = 0, ediff = 0.;
            for( auto& record : feltor::diagnostics2d_list)
            {
                if( std::find( feltor::energies.begin(), feltor::energies.end(), record.name) != feltor::energies.end())
                {
                    std::cout << record.name<<" : ";
                    record.function( result, var);
                    double norm = dg::blas1::dot( result, feltor.vol3d());
                    energy += norm;
                    std::cout << norm<<std::endl;

                }
                if( std::find( feltor::energy_diff.begin(), feltor::energy_diff.end(), record.name) != feltor::energy_diff.end())
                {
                    std::cout << record.name<<" : ";
                    record.function( result, var);
                    double norm = dg::blas1::dot( result, feltor.vol3d());
                    ediff += norm;
                    std::cout << norm<<std::endl;
                }

            }
            dEdt = (energy - E0)/deltat;
            E0 = energy;
            accuracy  = 2.*fabs( (dEdt - ediff)/( dEdt + ediff));

            std::cout << "\tTime "<<time<<"\n";
            std::cout <<"\td E/dt = " << dEdt
              <<" Lambda = " << ediff
              <<" -> Accuracy: " << accuracy << "\n";
            double max_ue = dg::blas1::reduce(
                feltor.velocity(0), 0., dg::AbsMax<double>() );
            MPI_OUT std::cout << "\tMaximum ue "<<max_ue<<"\n";
            //----------------Test if induction equation holds
            if( p.beta != 0)
            {
                dg::blas1::pointwiseDot(
                    feltor.density(0), feltor.velocity(0), dvisual);
                dg::blas1::pointwiseDot( p.beta,
                    feltor.density(1), feltor.velocity(1), -p.beta, dvisual);
                double norm  = dg::blas2::dot( dvisual, feltor.vol3d(), dvisual);
                dg::blas1::axpby( -1., feltor.lapMperpA(), 1., dvisual);
                double error = dg::blas2::dot( dvisual, feltor.vol3d(), dvisual);
                std::cout << "\tRel. Error Induction "<<sqrt(error/norm) <<"\n";
            }

        }
        t.toc();
        std::cout << "\n\t Step "<<step << " at time  "<<time;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp/(double)p.inner_loop<<"\n\n";
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////
    return 0;

}
