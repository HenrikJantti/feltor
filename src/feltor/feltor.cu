#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>

#include "draw/host_window.h"

#include "feltor.cuh"

int main( int argc, char* argv[])
{
    ////Parameter initialisation ////////////////////////////////////////////
    Json::Value js, gs;
    if( argc == 1)
    {
        std::ifstream is("input.json");
        std::ifstream ks("geometry_params.json");
        is >> js;
        ks >> gs;
    }
    else if( argc == 3)
    {
        std::ifstream is(argv[1]);
        std::ifstream ks(argv[2]);
        is >> js;
        ks >> gs;
    }
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "
                  << argv[0]<<" [inputfile] [geomfile] \n";
        return -1;
    }
    const feltor::Parameters p( js);
    const dg::geo::solovev::Parameters gp(gs);
    p.display( std::cout);
    gp.display( std::cout);
    /////////////////////////////////////////////////////////////////////////
    double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscaleRp*gp.a;
    double Zmax=p.boxscaleZp*gp.a*gp.elongation;
    //Make grid
    dg::CylindricalGrid3d grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI,
        p.n, p.Nx, p.Ny, p.Nz, p.bcxN, p.bcyN, dg::PER);
    dg::geo::TokamakMagneticField mag = dg::geo::createSolovevField(gp);

    //create RHS
    std::cout << "Constructing Explicit...\n";
    feltor::Explicit<dg::CylindricalGrid3d, dg::IDMatrix, dg::DMatrix, dg::DVec> feltor( grid, p, mag);
    std::cout << "Constructing Implicit...\n";
    feltor::Implicit<dg::CylindricalGrid3d, dg::IDMatrix, dg::DMatrix, dg::DVec> im( grid, p, mag);
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
    //First the profile and the source (on the host since we want to output those)
    dg::HVec profile = dg::pullback( dg::geo::Compose<dg::LinearX>( mag.psip(),
        p.nprofamp/mag.psip()(mag.R0(), 0.), 0.), grid);
    dg::HVec xpoint_damping = dg::evaluate( dg::one, grid);
    if( gp.hasXpoint() )
        xpoint_damping = dg::pullback(
            dg::geo::ZCutter(-1.1*gp.elongation*gp.a), grid);
    dg::HVec source_damping = dg::pullback(dg::geo::TanhDamping(
        //first change coordinate from psi to (psi_0 - psip)/psi_0
        dg::geo::Compose<dg::LinearX>( mag.psip(), -1./mag.psip()(mag.R0(), 0.),1.),
        //then shift tanh
        p.rho_source-3.*p.alpha, p.alpha, -1.), grid);
    dg::blas1::pointwiseDot( xpoint_damping, source_damping, source_damping);
    if( p.omega_source != 0)
        feltor.set_source( p.omega_source, profile, source_damping);

    dg::HVec profile_damping = dg::pullback( dg::geo::TanhDamping(
        mag.psip(), -3.*p.alpha, p.alpha, -1), grid);
    dg::blas1::pointwiseDot( xpoint_damping, profile_damping, profile_damping);
    dg::blas1::pointwiseDot( profile_damping, profile, profile);

    //Now perturbation
    dg::HVec ntilde = dg::evaluate(dg::zero,grid);
    if( p.initne == "blob" || p.initne == "straight blob")
    {
        dg::GaussianZ gaussianZ( 0., p.sigma_z*M_PI, 1);
        dg::Gaussian init0( gp.R_0+p.posX*gp.a, p.posY*gp.a, p.sigma, p.sigma, p.amp);
        if( p.symmetric)
            ntilde = dg::pullback( init0, grid);
        else if( p.initne == "blob")//rounds =3 ->2*3-1
            ntilde = feltor.fieldalignedn( init0, gaussianZ, (unsigned)p.Nz/2, 3);
        else if( p.initne == "straight blob")//rounds =1 ->2*1-1
            ntilde = feltor.fieldalignedn( init0, gaussianZ, (unsigned)p.Nz/2, 1);
    }
    else if( p.initne == "turbulence")
    {
        dg::GaussianZ gaussianZ( 0., p.sigma_z*M_PI, 1);
        dg::BathRZ init0(16,16,Rmin,Zmin, 30.,5.,p.amp);
        if( p.symmetric)
            ntilde = dg::pullback( init0, grid);
        else
            ntilde = feltor.fieldalignedn( init0, gaussianZ, (unsigned)p.Nz/2, 1);
        dg::blas1::pointwiseDot( profile_damping, ntilde, ntilde);
    }
    else if( p.initne == "zonal")
    {
        dg::geo::ZonalFlow init0(mag.psip(), p.amp, 0., p.k_psi);
        ntilde = dg::pullback( init0, grid);
        dg::blas1::pointwiseDot( profile_damping, ntilde, ntilde);
    }
    else
        std::cerr <<"WARNING: Unknown initial condition!\n";
    std::array<std::array<dg::DVec,2>,2> y0;
    y0[0][0] = y0[0][1] = y0[1][0] = y0[1][1] = dg::construct<dg::DVec>(profile);
    dg::blas1::axpby( 1., dg::construct<dg::DVec>(ntilde), 1., y0[0][0]);
    std::cout << "initialize ni" << std::endl;
    if( p.initphi == "zero")
        feltor.initializeni( y0[0][0], y0[0][1]);
    else if( p.initphi == "balance")
        dg::blas1::copy( y0[0][0], y0[0][1]); //set N_i = n_e
    else
        std::cerr <<"WARNING: Unknown initial condition for phi!\n";

    dg::blas1::copy( 0., y0[1][0]); //set we = 0
    dg::blas1::copy( 0., y0[1][1]); //set Wi = 0

    ////////////////////////create timer and timestepper
    //
    dg::Timer t;
    double time = 0, dt_new = p.dt, dt =0;
    unsigned step = 0;
    dg::Adaptive< dg::ARKStep<std::array<std::array<dg::DVec,2>,2>> > adaptive(
        "ARK-4-2-3", y0, grid.size(), p.eps_time);

    //since we map pointers we don't need to update those later

    std::map<std::string, const dg::DVec* > v4d;
    v4d["ne-1 / "] = &y0[0][0],               v4d["ni-1 / "] = &y0[0][1];
    v4d["Ue / "]   = &feltor.fields()[1][0],  v4d["Ui / "]   = &feltor.fields()[1][1];
    v4d["Omega / "] = &feltor.potential()[0]; v4d["Apar / "] = &feltor.induction();
    const feltor::Quantities& q = feltor.quantities();
    double dEdt = 0, accuracy = 0, dMdt = 0, accuracyM  = 0;
    std::map<std::string, const double*> v0d{
        {"energy", &q.energy}, {"ediff", &q.ediff},
        {"mass", &q.mass}, {"diff", &q.diff}, {"Apar", &q.Apar},
        {"Se", &q.S[0]}, {"Si", &q.S[1]}, {"Uperp", &q.Tperp},
        {"Upare", &q.Tpar[0]}, {"Upari", &q.Tpar[1]},
        {"dEdt", &dEdt}, {"accuracy", &accuracy},
        {"aligned", &q.aligned}
    };

    //first, update quantities in feltor

    {
        std::array<std::array<dg::DVec,2>,2> y1(y0);
        try{
            feltor( time, y0, y1);
        } catch( dg::Fail& fail) {
            std::cerr << "CG failed to converge in first step to "
                      << fail.epsilon()<<"\n";
            return -1;
        }
        feltor.update_quantities();
    }
    double energy0 = q.energy, mass0 = q.mass, E0 = energy0, M0 = mass0;
    /////////////////////////set up transfer for glfw
    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual), avisual(hvisual);
    dg::IHMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);
    //perp laplacian for computation of vorticity

    dg::Elliptic3d<dg::CylindricalGrid3d, dg::DMatrix, dg::DVec>
        laplacianM(grid, p.bcxP, p.bcyP, dg::PER, dg::normed, dg::centered);
    auto bhatF = dg::geo::createEPhi();
    if( p.curvmode == "true")
        bhatF = dg::geo::createBHat( mag);
    dg::SparseTensor<dg::DVec> hh = dg::geo::createProjectionTensor( bhatF, grid);
    laplacianM.set_chi( hh);

    /////////glfw initialisation ////////////////////////////////////////////
    //
    std::stringstream title;
    std::ifstream is( "window_params.js");
    is >> js;
    is.close();
    unsigned red = js.get("reduction", 1).asUInt();
    GLFWwindow* w = draw::glfwInitAndCreateWindow( (p.Nz/red+1)*js["width"].asDouble(), js["rows"].asDouble()*js["height"].asDouble(), "");
    draw::RenderHostData render(js["rows"].asDouble(), p.Nz/red + 1);

    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    dg::Average<dg::HVec> toroidal_average( grid, dg::coo3d::z);
    title << std::setprecision(2) << std::scientific;
    while ( !glfwWindowShouldClose( w ))
    {
        for( auto pair : v4d)
        {
            if(pair.first == "Omega / ")
            {
                dg::blas2::gemv( laplacianM, *pair.second, dvisual);
                dg::assign( dvisual, hvisual);
            }
            else
                dg::assign( *pair.second, hvisual);
            dg::blas2::gemv( equi, hvisual, visual);
            colors.scalemax() = (double)thrust::reduce(
                visual.begin(), visual.end(), 0., thrust::maximum<double>() );
            colors.scalemin() = -colors.scalemax();
            title <<pair.first << colors.scalemax()<<"\t";
            for( unsigned k=0; k<p.Nz/red;k++)
            {
                unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
                dg::HVec part( visual.begin() +  k*red   *size,
                               visual.begin() + (k*red+1)*size);
                render.renderQuad( part, grid.n()*grid.Nx(),
                                         grid.n()*grid.Ny(), colors);
            }
            dg::blas1::scal(avisual,0.);
            toroidal_average(visual,avisual);
            render.renderQuad( avisual, grid.n()*grid.Nx(),
                                        grid.n()*grid.Ny(), colors);
        }
        title << std::fixed;
        title << " &&   time = "<<time;
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step
        t.tic();
        for( unsigned i=0; i<p.itstp; i++)
        {
            try{
                do
                {
                    dt = dt_new;
                    adaptive.step( feltor, im, time, y0, time, y0, dt_new,
                        dg::pid_control, dg::l2norm, p.rtol, 1e-10);
                    if( adaptive.failed())
                        std::cout << "FAILED STEP! REPEAT!\n";
                }while ( adaptive.failed());
            }
            catch( dg::Fail& fail) {
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
            step++;
            feltor.update_quantities();
            std::cout << "Timestep "<<dt<<"\n";
            dEdt = (*v0d["energy"] - E0)/dt, dMdt = (*v0d["mass"] - M0)/dt;
            E0 = *v0d["energy"], M0 = *v0d["mass"];
            accuracy  = 2.*fabs( (dEdt - *v0d["ediff"])/( dEdt + *v0d["ediff"]));
            accuracyM = 2.*fabs( (dMdt - *v0d["diff"])/( dMdt + *v0d["diff"]));

            q.display(std::cout);
            std::cout << "(m_tot-m_0)/m_0: "<< (*v0d["mass"]-mass0)/mass0<<"\t";
            std::cout << "(E_tot-E_0)/E_0: "<< (*v0d["energy"]-energy0)/energy0<<"\t";
            std::cout <<" d E/dt = " << dEdt
              <<" Lambda = " << *v0d["ediff"]
              <<" -> Accuracy: " << accuracy << "\n";
            std::cout <<" d M/dt = " << dMdt
                      <<" Lambda = " << *v0d["diff"]
                      <<" -> Accuracy: " << accuracyM << "\n";

        }
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////
    return 0;

}
