#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <sstream>
#include <limits.h>  // UINT_MAX is needed in cusp (v0.5.1) but limits.h is not included
#include <thrust/remove.h>
#include <thrust/host_vector.h>

#include "dg/algorithm.h"
#include "dg/file/json_utilities.h"

#include "draw/host_window.h"

#include "shu.cuh"
#include "parameters.h"

double delta =0.05;
double rho =M_PI/15.;
double shearLayer(double x, double y){
    if( y<= M_PI)
        return delta*cos(x) - 1./rho/cosh( (y-M_PI/2.)/rho)/cosh( (y-M_PI/2.)/rho);
    return delta*cos(x) + 1./rho/cosh( (3.*M_PI/2.-y)/rho)/cosh( (3.*M_PI/2.-y)/rho);
}

using namespace std;
using namespace dg;

int main( int argc, char* argv[])
{
    ////Parameter initialisation ////////////////////////////////////////////
    Json::Value js;
    if( argc == 1)
        file::file2Json( "input/default.json", js, file::comments::are_discarded);
    else if( argc == 2)
        file::file2Json( argv[1], js);
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "<< argv[0]<<" [filename]\n";
        return -1;
    }
    const Parameters p( js);
    p.display( std::cout);
    /////////////////////////////////////////////////////////////////
    Grid2d grid( 0, p.lx, 0, p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);
    DVec w2d( create::weights(grid));
    /////////////////////////////////////////////////////////////////
    std::stringstream title;
    GLFWwindow* w = draw::glfwInitAndCreateWindow(600, 600, "");
    draw::RenderHostData render( 1,1);
    ////////////////////////////////////////////////////////////

    dg::Lamb lamb( p.posX*p.lx, p.posY*p.ly, p.R, p.U);
    dg::HVec omega;
    if( p.initial == "lamb")
        omega = dg::evaluate ( lamb, grid);
    else if ( p.initial == "shear")
        omega = dg::evaluate ( shearLayer, grid);

    DVec stencil = evaluate( one, grid);
    DVec y0( omega ), y1( y0);
    //subtract mean mass 
    if( p.bc_x == dg::PER && p.bc_y == dg::PER)
    {
        double meanMass = dg::blas2::dot( y0, w2d, stencil)/(double)(p.lx*p.ly);
        dg::blas1::axpby( -meanMass, stencil, 1., y0);
    }
    //make solver and stepper
    Shu<DMatrix, DVec> shu( grid, p.eps);
    Diffusion<DMatrix, DVec> diffusion( grid, p.D);
    Karniadakis< DVec > karniadakis( y0, y0.size(), p.eps_time);

    Timer t;
    t.tic();
    shu( 0., y0, y1);
    t.toc();
    cout << "Time for one rhs evaluation: "<<t.diff()<<"s\n";
    double vorticity = blas2::dot( stencil , w2d, y0);
    double enstrophy = 0.5*blas2::dot( y0, w2d, y0);
    double energy =    0.5*blas2::dot( y0, w2d, shu.potential()) ;
    
    std::cout << "Total energy:     "<<energy<<"\n";
    std::cout << "Total enstrophy:  "<<enstrophy<<"\n";
    std::cout << "Total vorticity:  "<<vorticity<<"\n";

    double time = 0;
    ////////////////////////////////glfw//////////////////////////////
    //create visualisation vectors
    DVec visual( grid.size());
    HVec hvisual( grid.size());
    //transform vector to an equidistant grid
    dg::IDMatrix equidistant = dg::create::backscatter( grid );
    draw::ColorMapRedBlueExt colors( 1.);
    karniadakis.init( shu, diffusion, time, y0, p.dt);
    //cout << "Press any key to start!\n";
    //double x; 
    //cin >> x;
    while (!glfwWindowShouldClose(w) && time < p.maxout*p.itstp*p.dt)
    {
        dg::blas2::symv( equidistant, y0, visual);
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), -1., dg::AbsMax<double>() );
        //draw and swap buffers
        dg::blas1::transfer( visual, hvisual);
        render.renderQuad( hvisual, p.n*p.Nx, p.n*p.Ny, colors);
        title << "Time "<<time<< " \ttook "<<t.diff()/(double)p.itstp<<"\t per step";
        glfwSetWindowTitle(w, title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers(w);
        //step 
        t.tic();
        for( unsigned i=0; i<p.itstp; i++)
        {
            karniadakis.step( shu, diffusion, time, y0 );
        }
        t.toc();
        //cout << "Timer for one step: "<<t.diff()/N<<"s\n";
        time += p.itstp*p.dt;

    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////
    cout << "Analytic formula enstrophy "<<lamb.enstrophy()<<endl;
    cout << "Analytic formula energy    "<<lamb.energy()<<endl;
    cout << "Total vorticity          is: "<<blas2::dot( stencil , w2d, y0) << "\n";
    cout << "Relative enstrophy error is: "<<(0.5*blas2::dot( w2d, y0) - enstrophy)/enstrophy<<"\n";
    cout << "Relative energy error    is: "<<(0.5*blas2::dot( shu.potential(), w2d, y0) - energy)/energy<<"\n";

    //blas1::axpby( 1., y0, -1, sol);
    //cout << "Distance to solution: "<<sqrt(blas2::dot( w2d, sol ))<<endl;

    //cout << "Press any key to quit!\n";
    //cin >> x;
    return 0;

}
