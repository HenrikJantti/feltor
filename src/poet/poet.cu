#include "hip/hip_runtime.h"
#define SILENT
// #define DG_DEBUG
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>

#include "draw/host_window.h"
//#include "draw/device_window.cuh"
#include "init.h"
#include "poet.cuh"
#include "dg/algorithm.h"
#include "dg/file/json_utilities.h"
#include "parameters.h"

#include "dg/polarization_init.h"
#include "dg/andersonacc.h"

/*
   - reads parameters from input.json or any other given file,
   - integrates the ToeflR - functor and
   - directly visualizes results on the screen using parameters in window_params.json
*/
using DVec = dg::DVec;
using DMatrix =  dg::DMatrix;



int main( int argc, char* argv[])
{
    ////Parameter initialisation ////////////////////////////////////////////
    std::stringstream title;
    Json::Value js;
    if( argc == 1)
        dg::file::file2Json( "input.json", js, dg::file::comments::are_forbidden);
    else if( argc == 2)
        dg::file::file2Json( argv[1], js, dg::file::comments::are_forbidden);
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "<< argv[0]<<" [filename]\n";
        return -1;
    }
    const Parameters p( js);
    p.display( std::cout);
    /////////glfw initialisation ////////////////////////////////////////////
    dg::file::file2Json( "window_params.json", js, dg::file::comments::are_discarded);
    GLFWwindow* w = draw::glfwInitAndCreateWindow( js["width"].asDouble(), js["height"].asDouble(), "");
    draw::RenderHostData render(js["rows"].asDouble(), js["cols"].asDouble());
    /////////////////////////////////////////////////////////////////////////

    dg::Grid2d grid( 0, p.lx, 0, p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);
    //create RHS
    poet::Explicit<dg::CartesianGrid2d, DMatrix, DVec> ex( grid, p);
    poet::Implicit<dg::CartesianGrid2d, DMatrix, DVec> im( grid, p.nu);
    //////////////////create initial vector///////////////////////////////////////
    std::vector<DVec> y0(2, dg::evaluate( dg::zero, grid)), y1(y0); // n_e' = gaussian

    if (p.init == "blob")
    {
        dg::Gaussian g( p.posX*p.lx, p.posY*p.ly, p.sigma, p.sigma, p.amp); 
        y0[0] = dg::evaluate(g, grid);
        ex.gamma1inv_y(y0[0],y0[1]); //no inversion -> smaller accuracy but n_e can be chosen instead of N_i!
//         y0[1] = dg::evaluate(g, grid);
//         ex.gamma1_y(y0[1], y0[0]); //invert Gamma operator for initialization with higher accuracy!
    }
    else if (p.init == "shearlayer")
    {
        ShearLayer layer(M_PI/15., 0.05, p.lx, p.ly); //shear layer
        std::vector<DVec> y0(2, dg::evaluate( layer, grid)), y1(y0);
        dg::blas1::scal(y0[0], p.amp);
        ex.invLap_y(y0[0], y1[0]); //phi 
        dg::blas1::scal(y0[0], 0.);
        ex.solve_Ni_lwl(y0[0], y1[0], y0[1]); //if df
        //Compute exact Ni with fixed point iteration
    //     dg::PolChargeN< dg::CartesianGrid2d, DMatrix, DVec > polN(grid, dg::DIR, dg::PER, dg::normed, dg::centered, 1.0, false);
    //     polN.set_phi(y1[0]);
    //     dg::AndersonAcceleration<DVec> acc( y1[0], 10000);
    // 
    //     dg::blas1::scal(y0[1], 0.0);
    //     dg::blas1::plus(y0[1], 1.0); //x solution must be positive 
    //     dg::blas1::scal(y0[0], 0.);  //ne_tilde = 0
    // 
    //     acc.solve( polN, y0[1], y0[0], im.weights(), 1e-4, 1e-4, grid.size(), 1e-13, 10000, true);    
    //     dg::blas1::plus(y0[1],-1.0);
    }
    else if (p.init == "rot_blob")
    {
//     //double rotating gaussian
//     dg::Gaussian g1( (0.5-p.posX)*p.lx, p.posY*p.ly, p.sigma, p.sigma, p.amp);
//     dg::Gaussian g2( (0.5+p.posX)*p.lx, p.posY*p.ly, p.sigma, p.sigma, p.amp);
// 
//     std::vector<DVec> y0(2, dg::evaluate( g1, grid)); // n_e' = gaussian
//     std::vector<DVec> y1(2, dg::evaluate( g2, grid)); // n_e' = gaussian
//     dg::blas1::axpby(1.0,y0[0],1.0,y1[0],y0[0]);
//     dg::blas1::axpby(10, y0[0], 0.0, y1[1]);
//     ex.invLap_y(y1[1], y1[0]); //phi 
//     ex.solve_Ni_lwl(y0[0], y1[0], y0[1]);
    }

    //////////////////////////////////////////////////////////////////////
    dg::ImExMultistep<std::vector<DVec>> stepper( "ImEx-TVB-3-3", y0, y0[0].size(), p.eps_time);
//     dg::Adaptive<dg::ARKStep<std::vector<DVec>>> stepper( "ARK-4-2-3", y0, y0[0].size(), p.eps_time);
//     dg::Adaptive<dg::ERKStep<std::vector<DVec>>> stepper( "Dormand-Prince-7-4-5", y0);

    DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual);
    dg::IHMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExt colors( 1.);
    //create timer
    dg::Timer t;
    double time = 0;
    stepper.init( ex, im, time, y0, p.dt);
//     double dt = 1e-5;
    const double mass0 = ex.mass(), mass_blob0 = mass0 - grid.lx()*grid.ly();
    double E0 = ex.energy(), energy0 = E0, E1 = 0, diff = 0;
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    unsigned step = 0;
    while ( !glfwWindowShouldClose( w ))
    {
        //transform field to an equidistant grid
        dvisual = y0[0];

        dg::assign( dvisual, hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw ions
        title << std::setprecision(2) << std::scientific;
        title <<"ne / "<<colors.scale()<<"\t";
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        //transform phi
        dvisual = ex.potential()[0];
        dg::blas2::gemv( ex.laplacianM(), dvisual, y1[1]);
        dg::assign( y1[1], hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw phi and swap buffers
        title <<"omega / "<<colors.scale()<<"\t";
        title << std::fixed;
        title << " &&   time = "<<time;
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<p.itstp; i++)
        {
            step++;
            {
                std::cout << "(m_tot-m_0)/m_0: "<< (ex.mass()-mass0)/mass_blob0<<"\t";
                E0 = E1;
                E1 = ex.energy();
                diff = (E1 - E0)/p.dt;
                double diss = ex.energy_diffusion( );
                std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
                std::cout << "Accuracy: "<< 2.*(diff-diss)/(diff+diss)<<"\n";

            }
            try{ stepper.step( ex, im, time, y0);}
//             try{
// //                 std::cout << "Time "<<time<<" dt "<<dt<<" success "<<!stepper.failed()<<"\n";
// //                 stepper.step( ex, im, time, y0, time, y0, dt, dg::pid_control, dg::l2norm, 1e-7, 1e-14);
// //                 stepper.step( ex, time, y0, time, y0, dt, dg::pid_control, dg::l2norm, 1e-7, 1e-14);
//             }
            catch( dg::Fail& fail) {
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
        }
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
