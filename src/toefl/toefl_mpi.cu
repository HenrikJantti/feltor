#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>

#include <mpi.h> //activate mpi

#include "netcdf_par.h"
#include "file/nc_utilities.h"

#include "toeflR.cuh"
#include "dg/algorithm.h"
#include "parameters.h"


int main( int argc, char* argv[])
{
    ////////////////////////////////setup MPI///////////////////////////////
    int provided;
    MPI_Init_thread( &argc, &argv, MPI_THREAD_FUNNELED, &provided);
    if( provided != MPI_THREAD_FUNNELED)
    {
        std::cerr << "wrong mpi-thread environment provided!\n";
        return -1;
    }
    int periods[2] = {false, true}; //non-, periodic
    int rank, size;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    MPI_Comm_size( MPI_COMM_WORLD, &size);
#if THRUST_DEVICE_SYSTEM==THRUST_DEVICE_SYSTEM_CUDA
    int num_devices=0;
    hipGetDeviceCount(&num_devices);
    if(num_devices==0){std::cerr << "No CUDA capable devices found"<<std::endl; return -1;}
    int device = rank % num_devices; //assume # of gpus/node is fixed
    hipSetDevice( device);
#endif//cuda
    int np[2];
    if(rank==0)
    {
        std::cin>> np[0] >> np[1];
        std::cout << "Computing with "<<np[0]<<" x "<<np[1]<<" = "<<size<<std::endl;
        assert( size == np[0]*np[1]);
    }
    MPI_Bcast( np, 2, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Comm comm;
    MPI_Cart_create( MPI_COMM_WORLD, 2, np, periods, true, &comm);
    ////////////////////////Parameter initialisation//////////////////////////
    Json::Value js;
    Json::CharReaderBuilder parser;
    parser["collectComments"] = false; //important since we want to write to netcdf
    std::string errs;
    if( argc != 3)
    {
        if(rank==0)std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [outputfile]\n";
        return -1;
    }
    else
    {
        std::ifstream is(argv[1]);
        parseFromStream( parser, is, &js, &errs);
    }
    const Parameters p( js);
    if(rank==0)p.display( std::cout);

    ////////////////////////////////set up computations///////////////////////////
    dg::MPIGrid2d grid( 0, p.lx, 0, p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y, comm);
    dg::MPIGrid2d grid_out( 0., p.lx, 0.,p.ly, p.n_out, p.Nx_out, p.Ny_out, p.bc_x, p.bc_y, comm);
    //create RHS
    toefl::Explicit< dg::CartesianMPIGrid2d, dg::MDMatrix, dg::MDVec > test( grid, p);
    toefl::Implicit< dg::CartesianMPIGrid2d, dg::MDMatrix, dg::MDVec > diffusion( grid, p.nu);
    //////////////////create initial vector///////////////////////////////////////
    dg::Gaussian g( p.posX*p.lx, p.posY*p.ly, p.sigma, p.sigma, p.amp);
    std::vector<dg::MDVec> y0(2, dg::evaluate( g, grid)), y1(y0); // n_e' = gaussian
    dg::blas2::symv( test.gamma(), y0[0], y0[1]); // n_e = \Gamma_i n_i -> n_i = ( 1+alphaDelta) n_e' + 1
    {
        dg::MDVec v2d = dg::create::inv_weights(grid);
        dg::blas2::symv( v2d, y0[1], y0[1]);
    }
    if( p.equations == "gravity_local" || p.equations == "gravity_global" || p.equations == "drift_global" ){
        y0[1] = dg::evaluate( dg::zero, grid);
    }
    //////////////////initialisation of timestepper and first step///////////////////
    double time = 0;
    dg::Karniadakis< std::vector<dg::MDVec> > stepper( y0, y0[0].size(), p.eps_time);
    stepper.init( test, diffusion, time, y0, p.dt);
    y1 = y0;
    /////////////////////////////set up netcdf/////////////////////////////////////
    file::NC_Error_Handle err;
    int ncid; MPI_Info info = MPI_INFO_NULL;
    err = nc_create_par( argv[2],NC_NETCDF4|NC_MPIIO|NC_CLOBBER,comm,info, &ncid);
    std::string input = js.toStyledString();
    err = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    int dim_ids[3], tvarID;
    err = file::define_dimensions( ncid, dim_ids, &tvarID, grid_out.global());
    //field IDs
    std::string names[4] = {"electrons", "ions", "potential", "vorticity"};
    int dataIDs[4];
    for( unsigned i=0; i<4; i++){
        err = nc_def_var( ncid, names[i].data(), NC_DOUBLE, 3, dim_ids, &dataIDs[i]);}

    //energy IDs
    int EtimeID, EtimevarID;
    err = file::define_time( ncid, "energy_time", &EtimeID, &EtimevarID);
    int energyID, massID, dissID, dEdtID;
    err = nc_def_var( ncid, "energy",      NC_DOUBLE, 1, &EtimeID, &energyID);
    err = nc_def_var( ncid, "mass",        NC_DOUBLE, 1, &EtimeID, &massID);
    err = nc_def_var( ncid, "dissipation", NC_DOUBLE, 1, &EtimeID, &dissID);
    err = nc_def_var( ncid, "dEdt",        NC_DOUBLE, 1, &EtimeID, &dEdtID);
      //mpi specific part
      for(unsigned i=0; i<4; i++)
          err = nc_var_par_access( ncid, dataIDs[i], NC_COLLECTIVE);
      err = nc_var_par_access( ncid, tvarID, NC_COLLECTIVE);
      err = nc_var_par_access( ncid, EtimevarID, NC_COLLECTIVE);
      err = nc_var_par_access( ncid, energyID, NC_COLLECTIVE);
      err = nc_var_par_access( ncid, massID, NC_COLLECTIVE);
      err = nc_var_par_access( ncid, dissID, NC_COLLECTIVE);
      err = nc_var_par_access( ncid, dEdtID, NC_COLLECTIVE);
      err = nc_enddef(ncid);
      int dims[2],  coords[2];
      MPI_Cart_get( comm, 2, dims, periods, coords);
      dg::MDVec transfer( dg::evaluate( dg::zero, grid));
    ///////////////////////////////////first output/////////////////////////
    size_t count[3] = {1, grid_out.n()*grid_out.local().Ny(), grid_out.n()*grid_out.local().Nx()};
    size_t start[3] = {0, coords[1]*count[1], coords[0]*count[2]};
    size_t Ecount[] = {1};
    size_t Estart[] = {0};
    std::vector<dg::DVec> transferD(4, dg::evaluate(dg::zero, grid_out.local()));
    dg::HVec transferH(dg::evaluate(dg::zero, grid_out.local()));
    dg::IDMatrix interpolate = dg::create::interpolation( grid_out.local(), grid.local());
    dg::blas2::symv( interpolate, y1[0].data(), transferD[0]);
    dg::blas2::symv( interpolate, y1[1].data(), transferD[1]);
    dg::blas2::symv( interpolate, test.potential()[0].data(), transferD[2]);
    dg::blas2::symv( diffusion.laplacianM(), test.potential()[0], transfer);
    dg::blas2::symv( interpolate, transfer.data(), transferD[3]);
    for( unsigned k=0; k<4; k++)
    {
        dg::blas1::transfer( transferD[k], transferH);
        err = nc_put_vara_double( ncid, dataIDs[k], start, count, transferH.data() );
    }
    err = nc_put_vara_double( ncid, tvarID, start, count, &time);
    //err = nc_close(ncid);
    ///////////////////////////////////////Timeloop/////////////////////////////////
    const double mass0 = test.mass(), mass_blob0 = mass0 - grid.global().lx()*grid.global().ly();
    double E0 = test.energy(), E1 = 0, diff = 0;
    dg::Timer t;
    t.tic();
    try
    {
#ifdef DG_BENCHMARK
    unsigned step = 0;
#endif //DG_BENCHMARK
    for( unsigned i=1; i<=p.maxout; i++)
    {

#ifdef DG_BENCHMARK
        dg::Timer ti;
        ti.tic();
#endif//DG_BENCHMARK
        for( unsigned j=0; j<p.itstp; j++)
        {
            stepper.step( test, diffusion, time, y1);
            //store accuracy details
            {
                if(rank==0)std::cout << "(m_tot-m_0)/m_0: "<< (test.mass()-mass0)/mass_blob0<<"\t";
                E0 = E1;
                E1 = test.energy();
                diff = (E1 - E0)/p.dt;
                double diss = test.energy_diffusion( );
                if(rank==0)std::cout << "diff: "<< diff<<" diss: "<<diss<<"\t";
                if(rank==0)std::cout << "Accuracy: "<< 2.*(diff-diss)/(diff+diss)<<"\n";
            }
            Estart[0] += 1;
            {
                //err = nc_open(argv[2], NC_WRITE, &ncid);
                double ener=test.energy(), mass=test.mass(), diff=test.mass_diffusion(), dEdt=test.energy_diffusion();
                err = nc_put_vara_double( ncid, EtimevarID, Estart, Ecount, &time);
                err = nc_put_vara_double( ncid, energyID,   Estart, Ecount, &ener);
                err = nc_put_vara_double( ncid, massID,     Estart, Ecount, &mass);
                err = nc_put_vara_double( ncid, dissID,     Estart, Ecount, &diff);
                err = nc_put_vara_double( ncid, dEdtID,     Estart, Ecount, &dEdt);
                //err = nc_close(ncid);
            }
        }
        //////////////////////////write fields////////////////////////
        start[0] = i;
        dg::blas2::symv( interpolate, y1[0].data(), transferD[0]);
        dg::blas2::symv( interpolate, y1[1].data(), transferD[1]);
        dg::blas2::symv( interpolate, test.potential()[0].data(), transferD[2]);
        dg::blas2::symv( diffusion.laplacianM(), test.potential()[0], transfer);
        dg::blas2::symv( interpolate, transfer.data(), transferD[3]);
        //err = nc_open(argv[2], NC_WRITE, &ncid);
        for( int k=0;k<4; k++)
        {
            dg::blas1::transfer( transferD[k], transferH);
            err = nc_put_vara_double( ncid, dataIDs[k], start, count, transferH.data() );
        }
        err = nc_put_vara_double( ncid, tvarID, start, count, &time);
        //err = nc_close(ncid);

#ifdef DG_BENCHMARK
        ti.toc();
        step+=p.itstp;
        if(rank==0)std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout <<" at time "<<time;
        if(rank==0)std::cout << "\n\t Average time for one step: "<<ti.diff()/(double)p.itstp<<"s\n\n"<<std::flush;
#endif//DG_BENCHMARK
    }
    }
    catch( dg::Fail& fail) {
        if(rank==0)std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
        if(rank==0)std::cerr << "Does Simulation respect CFL condition?\n";
    }
    t.toc();
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    if(rank==0)std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    if(rank==0)std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    if(rank==0)std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";
    nc_close(ncid);
    MPI_Finalize();

    return 0;

}

