#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG

#include "draw/host_window.h"
//#include "draw/device_window.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/timer.cuh"
#include "file/read_input.h"

#include "feltor.cuh"
#include "parameters.h"



/*
   - reads parameters from input.txt or any other given file, 
   - integrates the Feltor - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/


int main( int argc, char* argv[])
{
    ////////////////////////Parameter initialisation//////////////////////////
    std::vector<double> v,v2;
    std::stringstream title;
    if( argc == 1)
    {
        try{
            v = file::read_input("input.txt");
        }catch( toefl::Message& m){
            m.display();
            return -1;
        }
    }
    else if( argc == 2)
    {
        try{
            v = file::read_input(argv[1]);
        }catch( toefl::Message& m){
            m.display();
            return -1;
        }
    }
    else
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] \n";
        return -1;
    }
    const eule::Parameters p( v);
    p.display( std::cout);

    v2 = file::read_input( "window_params.txt");
    GLFWwindow* w = draw::glfwInitAndCreateWindow(  v2[2]*v2[3]*p.lx/p.ly, v2[1]*v2[4], "");
    draw::RenderHostData render( v2[1], v2[2]);



    //////////////////////////////////////////////////////////////////////////

    //Make grid
     dg::Grid2d<double > grid( 0., p.lx, 0.,p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);  
    //create RHS 
    std::cout << "Constructing Feltor...\n";
    eule::Feltor<dg::DMatrix, dg::DVec, dg::DVec > feltor( grid, p); //initialize before rolkar!
    std::cout << "Constructing Rolkar...\n";
    eule::Rolkar<dg::DMatrix, dg::DVec, dg::DVec > rolkar( grid, p);
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
    //initial perturbation
    //dg::Gaussian3d init0(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
    dg::Gaussian init0( p.posX*p.lx, p.posY*p.ly, p.sigma, p.sigma, p.amp);
//     dg::BathRZ init0(16,16,p.Nz,Rmin,Zmin, 30.,5.,p.amp);
//     solovev::ZonalFlow init0(p, gp);
//     dg::CONSTANT init0( 0.);
//      dg::Vortex init0(  p.posX*p.lx, p.posY*p.ly, 0, p.sigma, p.amp);   
    //background profile
//     solovev::Nprofile prof(p, gp); //initial background profile
    dg::CONSTANT prof(p.bgprofamp );
    //
//     dg::LinearX prof(-p.nprofileamp/((double)p.lx), p.bgprofamp + p.nprofileamp);
//     dg::SinProfX prof(p.nprofileamp, p.bgprofamp,M_PI/(2.*p.lx));
//     dg::ExpProfX prof(p.nprofileamp, p.bgprofamp,p.ln);
//     const dg::DVec prof =  dg::LinearX( -p.nprofileamp/((double)p.lx), p.bgprofamp + p.nprofileamp);
//     dg::TanhProfX prof(p.lx*p.solb,p.lx/10.,-1.0,p.bgprofamp,p.nprofileamp); //<n>
    
    std::vector<dg::DVec> y0(4, dg::evaluate( prof, grid)), y1(y0); //Ne,Ni,Te,Ti = prof    
   
   //initialization via N_i,T_I ->n_e, t_i=t_e
    y1[1] = dg::evaluate( init0, grid);
    dg::blas1::pointwiseDot(y1[1], y0[1],y1[1]); //<n>*ntilde    
    dg::blas1::axpby( 1., y1[1], 1., y0[1]); //initialize Ni = <n> + <n>*ntilde
    if (p.iso == 1) dg::blas1::axpby( 1.,y1[2], 0., y0[3]); //initialize Ti = prof
    if (p.iso == 0) dg::blas1::axpby( 1.,y0[1], 0., y0[3]); //initialize Ti = N_i
    dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); //= Ni - bg
    std::cout << "intiialize ne" << std::endl;
//     feltor.initializene( y0[1],y0[3], y0[0]);    //ne -bg
    dg::blas1::axpby( 1., y0[1], 0., y0[0], y0[0]); // for Omega*=0
    std::cout << "Done!\n";    
    
    std::cout << "intialize ti=te" << std::endl;
    if (p.iso == 1) {
        dg::blas1::transform(y0[3], y0[3], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); // =Ti - bg
        dg::blas1::axpby( 1.,y0[3], 0., y0[2]); //initialize Ti = N_i
    }
    if (p.iso == 0) {
        dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(+(p.bgprofamp + p.nprofileamp))); //Ni
        dg::blas1::pointwiseDot(y0[1],y0[3],y1[3]); // = Ni Ti
        dg::blas1::transform(y1[3], y1[3], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp)*(p.bgprofamp + p.nprofileamp))); //Pi = Pi - bg^2
//         feltor.initializepi(y1[3],y0[3], y0[2]); // = pi-bg^2    
        //compute ti-bg = ((pi-bg^2) +bg^2)/ne -bg
        dg::blas1::transform(y0[2], y0[2], dg::PLUS<>(+(p.bgprofamp + p.nprofileamp)*(p.bgprofamp + p.nprofileamp)));
        dg::blas1::transform(y0[0], y0[0], dg::PLUS<>(+(p.bgprofamp + p.nprofileamp))); //=ne    
        dg::blas1::pointwiseDivide(y0[2],y0[0],y0[2]);
        
        dg::blas1::axpby( 1., y0[3], 0., y0[2], y0[2]); //for Omega*=0
//         dg::blas1::axpby( 1., y1[2], 0., y0[2], y0[2]); //for Omega*=0 and te=1
//         dg::blas1::axpby( 1., y1[2], 0., y0[3], y0[3]); //for Omega*=0 and ti=1

        dg::blas1::transform(y0[2], y0[2], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp)));
        dg::blas1::transform(y0[0], y0[0], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); // =ne-bg
        dg::blas1::transform(y0[3], y0[3], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); // =Ti - bg
        dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); // =Ni - bg 
    }
    std::cout << "Done!\n";

    
    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    std::cout << "intiialize karniadakis" << std::endl;
    karniadakis.init( feltor, rolkar, y0, p.dt);
    std::cout << "Done!\n";

    dg::DVec dvisual( grid.size(), 0.);
    dg::DVec dvisual2( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual),avisual(hvisual);
    dg::IHMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);
    //create timer
    dg::Timer t;
    double time = 0;
    unsigned step = 0;
    
    const double mass0 = feltor.mass(), mass_blob0 = mass0 - grid.lx()*grid.ly();
    double E0 = feltor.energy(), energy0 = E0, E1 = 0., diff = 0.;
    
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);

    while ( !glfwWindowShouldClose( w ))
    {
        //draw Ne-1
        hvisual = y0[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), (double)-1e14, thrust::maximum<double>() );
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title << std::setprecision(2) << std::scientific;
        title <<"ne-1 / " << colors.scalemax() << " " << colors.scalemin()<<"\t";
//          colors.scalemin() =  -colors.scalemax();
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        //draw Ni-1
        hvisual = y0[1];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(),  (double)-1e14, thrust::maximum<double>() );
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title << std::setprecision(2) << std::scientific;
        title <<"ni-1 / " << colors.scalemax() << " " << colors.scalemin()<<"\t";
//          colors.scalemin() =  -colors.scalemax();
        render.renderQuad(visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        
        //draw potential
        hvisual = feltor.potential()[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(),  (double)-1e14, thrust::maximum<double>() );
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
        title <<"Pot / "<< colors.scalemax() << " " << colors.scalemin()<<"\t";
        colors.scalemin() =  -colors.scalemax();
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        
        //draw Te-1
        hvisual = y0[2];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), (double)-1e14, thrust::maximum<double>() );
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title << std::setprecision(2) << std::scientific;
        title <<"Te-1 / " << colors.scalemax() << " " << colors.scalemin()<<"\t";
//          colors.scalemin() =  -colors.scalemax();
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        //draw Ti-1
        hvisual = y0[3];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(),  (double)-1e14, thrust::maximum<double>() );
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title << std::setprecision(2) << std::scientific;
        title <<"Ti-1 / " << colors.scalemax() << " " << colors.scalemin()<<"\t";
//          colors.scalemin() =  -colors.scalemax();
        render.renderQuad(visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        
        //draw vor
        //transform to Vor
        dvisual=feltor.potential()[0];
        dg::blas2::gemv( rolkar.laplacianM(), dvisual, y1[1]);
        hvisual = y1[1];
         //hvisual = feltor.potential()[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(),  (double)-1e14, thrust::maximum<double>() );
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title <<"Omega / "<< colors.scalemax()<< " "<< colors.scalemin()<<"\t";
        colors.scalemin() =  -colors.scalemax();
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);     
           
        title << std::fixed; 
        title << " &&   time = "<<time;
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<p.itstp; i++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
            step++;
            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass_blob0<<"\t";
            E1 = feltor.energy();
            diff = (E1 - E0)/p.dt; //
            double diss = feltor.energy_diffusion( );
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout << " Accuracy: "<< 2.*fabs((diff-diss)/(diff+diss))<<
                         " d E/dt = " << diff <<
                         " Lambda =" << diss <<  std::endl;
 
            
            E0 = E1;

        }
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
