#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "dg/algorithm.h"
#include "dg/file/file.h"
#include "solovev.h"
#include "fluxfunctions.h"

int main( int argc, char* argv[])
{
    Json::Value geom_js;
    if( argc == 3)
    {
        std::cout << argv[0]<< " "<<argv[1]<<" -> " <<argv[2]<<std::endl;
        file::file2Json( argv[1], geom_js, file::comments::are_discarded);
    }
    else
    {
        std::cerr << "This program reads solovev parameters from an input json file and modifies c[0] such that the resulting Psi_p is zero on the X-point. The resulting parameters are written into an output file, which may overwrite the input file. The program aborts if it is unable to find an X-point\n";
        std::cerr << " Usage: "<< argv[0]<<" [input.json] [normalized.json]\n";
        return -1;
    }
    dg::geo::solovev::Parameters gp(geom_js);
    std::cout << "Input file: \n"<< geom_js.toStyledString();
    dg::geo::TokamakMagneticField mag = dg::geo::createSolovevField(gp);
    double RX = gp.R_0-1.1*gp.triangularity*gp.a;
    double ZX = -1.1*gp.elongation*gp.a;
    try{
        dg::geo::findXpoint( mag.get_psip(), RX, ZX);
    }catch ( std::exception& e)
    {
        std::cerr << e.what() << std::endl;
        return -1.;
    }
    const double psipX = mag.psip()( RX, ZX);
    std::cout << "X-point found at "<<RX<<" "<<ZX<<" with Psip = "<<psipX<<std::endl;
    gp.c[0] = gp.c[0] - psipX/gp.pp/gp.R_0;
    Json::Value output = gp.dump();
    std::cout << "Output file "<<argv[2]<<": \n"<< output.toStyledString();
    std::fstream file( argv[2], std::fstream::out | std::fstream::trunc);
    file << output.toStyledString();
    file << std::endl;
    file.close();
    return 0;
}

