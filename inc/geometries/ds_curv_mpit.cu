#include "hip/hip_runtime.h"
#include <iostream>

#include "mpi.h"

#include "dg/backend/timer.cuh"
#include "dg/backend/mpi_init.h"
#include "dg/backend/functions.h"
#include "dg/blas.h"
#include "dg/functors.h"
#include "dg/geometry/geometry.h"
#include "testfunctors.h"
#include "ds.h"
#include "solovev.h"
#include "flux.h"
#include "toroidal.h"
#include "mpi_curvilinear.h"


int main(int argc, char * argv[])
{
    MPI_Init( &argc, &argv);
    int rank;
    unsigned n, Nx, Ny, Nz; 
    MPI_Comm comm;
    dg::mpi_init3d( dg::DIR, dg::PER, dg::PER, n, Nx, Ny, Nz, comm);
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    Json::Reader reader;
    Json::Value js;
    if( argc==1)
    {
        std::ifstream is("geometry_params_Xpoint.js");
        reader.parse(is,js,false);
    }
    else
    {
        std::ifstream is(argv[1]);
        reader.parse(is,js,false);
    }
    dg::geo::solovev::Parameters gp(js);
    if(rank==0)std::cout << "Start DS test on flux grid!"<<std::endl;
    dg::geo::TokamakMagneticField mag = dg::geo::createSolovevField( gp);
    dg::Timer t;
    t.tic();
    unsigned mx=1, my=10;
    double psi_0 = -20, psi_1 = -4;
    dg::geo::FluxGenerator flux( mag.get_psip(), mag.get_ipol(), psi_0, psi_1, gp.R_0, 0., 1);
    if(rank==0)std::cout << "Constructing Grid...\n";
    dg::geo::CurvilinearProductMPIGrid3d g3d(flux, n, Nx, Ny,Nz, dg::DIR, dg::PER, dg::PER, comm);
    if(rank==0)std::cout << "Constructing Fieldlines...\n";
    dg::geo::DS<dg::aProductMPIGeometry3d, dg::MIHMatrix, dg::MHMatrix, dg::MHVec> ds( mag, g3d, dg::NEU, dg::NEU, dg::geo::FullLimiter(), dg::normed, dg::centered, 1e-8, mx, my, false, true);
    
    t.toc();
    if(rank==0)std::cout << "Construction took "<<t.diff()<<"s\n";
    dg::MHVec B = dg::pullback( dg::geo::InvB(mag), g3d), divB(B);
    dg::MHVec lnB = dg::pullback( dg::geo::LnB(mag), g3d), gradB(B);
    dg::MHVec gradLnB = dg::pullback( dg::geo::GradLnB(mag), g3d);
    dg::MHVec ones3d = dg::evaluate( dg::one, g3d);
    dg::MHVec vol3d = dg::create::volume( g3d);
    dg::blas1::pointwiseDivide( ones3d, B, B);
    dg::MHVec function = dg::pullback( dg::geo::FuncNeu(mag), g3d), derivative(function);
    ds( function, derivative);

    ds.centeredAdj( 1., B, 0., divB);
    double norm =  sqrt( dg::blas2::dot(divB, vol3d, divB));
    if(rank==0)std::cout << "Divergence of B is "<<norm<<"\n";

    ds.centered( 1., lnB, 0., gradB);
    norm = sqrt( dg::blas2::dot( gradLnB, vol3d, gradLnB) );
    dg::blas1::axpby( 1., gradB, -1., gradLnB, gradLnB);
    double norm2 = sqrt(dg::blas2::dot(gradLnB, vol3d, gradLnB));
    if(rank==0)std::cout << "rel. error of lnB is    "<<norm2/norm<<"\n";
    MPI_Finalize();
    return 0;
}
