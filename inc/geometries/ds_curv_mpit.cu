#include "hip/hip_runtime.h"
#include <iostream>
#include "json/json.h"

#include "mpi.h"

#include "dg/backend/timer.h"
#include "dg/backend/mpi_init.h"
#include "dg/geometry/functions.h"
#include "dg/blas.h"
#include "dg/functors.h"
#include "dg/geometry/geometry.h"
#include "testfunctors.h"
#include "ds.h"
#include "solovev.h"
#include "flux.h"
#include "toroidal.h"
#include "mpi_curvilinear.h"


int main(int argc, char * argv[])
{
    MPI_Init( &argc, &argv);
    int rank;
    unsigned n, Nx, Ny, Nz, mx[2];
    MPI_Comm comm;
    dg::mpi_init3d( dg::DIR, dg::PER, dg::PER, n, Nx, Ny, Nz, comm);
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    if( rank == 0)
    {
        std::cout << "Type mx (10) and my (10)\n";
        std::cin >> mx[0] >> mx[1];
        std::cout << "You typed "<<mx[0]<<" "<<mx[1]<<std::endl;
    }
    MPI_Bcast( mx, 2, MPI_INT, 0, MPI_COMM_WORLD);
    Json::Value js;
    if( argc==1)
    {
        std::ifstream is("geometry_params_Xpoint.js");
        is >> js;
    }
    else
    {
        std::ifstream is(argv[1]);
        is >> js;
    }
    dg::geo::solovev::Parameters gp(js);
    if(rank==0)std::cout << "Start DS test on flux grid!"<<std::endl;
    dg::geo::TokamakMagneticField mag = dg::geo::createSolovevField( gp);
    dg::Timer t;
    t.tic();
    double psi_0 = -20, psi_1 = -4;
    dg::geo::FluxGenerator flux( mag.get_psip(), mag.get_ipol(), psi_0, psi_1, gp.R_0, 0., 1);
    if(rank==0)std::cout << "Constructing Grid...\n";
    dg::geo::CurvilinearProductMPIGrid3d g3d(flux, n, Nx, Ny,Nz, dg::DIR, dg::PER, dg::PER, comm);
    if(rank==0)std::cout << "Constructing Fieldlines...\n";
    dg::geo::DS<dg::aProductMPIGeometry3d, dg::MIHMatrix, dg::MHMatrix, dg::MHVec> ds( mag, g3d, dg::NEU, dg::PER, dg::geo::FullLimiter(), dg::normed, dg::centered, 1e-8, mx[0], mx[1], false, true);

    t.toc();
    if(rank==0)std::cout << "Construction took "<<t.diff()<<"s\n";
    dg::MHVec B = dg::pullback( dg::geo::InvB(mag), g3d), divB(B);
    dg::MHVec lnB = dg::pullback( dg::geo::LnB(mag), g3d), gradB(B);
    const dg::MHVec gradLnB = dg::pullback( dg::geo::GradLnB(mag), g3d);
    dg::MHVec ones3d = dg::evaluate( dg::one, g3d);
    dg::MHVec vol3d = dg::create::volume( g3d);
    dg::blas1::pointwiseDivide( ones3d, B, B);

    const double sol_norm = sqrt( dg::blas2::dot( gradLnB, vol3d, gradLnB) );
    ds.centeredDiv( 1., ones3d, 0., divB);
    dg::blas1::axpby( 1., gradLnB, 1, divB);
    double norm =  sqrt( dg::blas2::dot(divB, vol3d, divB));
    if(rank==0)std::cout << "TEST ADJOINT derivatives!\n";
    if(rank==0)std::cout << "Error Centered Divergence of b is "<<norm/sol_norm<<"\n";
    ds.forwardDiv( 1., ones3d, 0., divB);
    dg::blas1::axpby( 1., gradLnB, 1, divB);
    norm =  sqrt( dg::blas2::dot(divB, vol3d, divB));
    if(rank==0)std::cout << "Error Forward  Divergence of b is "<<norm/sol_norm<<"\n";
    ds.backwardDiv( 1., ones3d, 0., divB);
    dg::blas1::axpby( 1., gradLnB, 1, divB);
    norm =  sqrt( dg::blas2::dot(divB, vol3d, divB));
    if(rank==0)std::cout << "Error Backward Divergence of b is "<<norm/sol_norm<<"\n";

    ds.centered( 1., lnB, 0., gradB);
    dg::blas1::axpby( 1., gradLnB, -1., gradB);
    norm = sqrt(dg::blas2::dot(gradB, vol3d, gradB));
    if(rank==0)std::cout << "TEST DIRECT derivatives!\n";
    if(rank==0)std::cout << "rel. error of lnB is    "<<norm/sol_norm<<"\n";
    MPI_Finalize();
    return 0;
}
