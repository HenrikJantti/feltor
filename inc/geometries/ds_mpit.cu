#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include <mpi.h>
#define DG_BENCHMARK
#undef DG_DEBUG
#include "dg/algorithm.h"
#include "magnetic_field.h"
#include "testfunctors.h"
#include "ds.h"
#include "toroidal.h"

const double R_0 = 10;
const double I_0 = 20; //q factor at r=1 is I_0/R_0
const double a  = 1; //small radius

int main(int argc, char* argv[])
{
    MPI_Init( &argc, &argv);
    int rank;
    unsigned n, Nx, Ny, Nz, mx[2], max_iter = 1e4;
    MPI_Comm comm;
    dg::mpi_init3d( dg::NEU, dg::NEU, dg::PER, n, Nx, Ny, Nz, comm);
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    if( rank == 0)
    {
        std::cout <<"# You typed\n"
                  <<"n:  "<<n<<"\n"
                  <<"Nx: "<<Nx<<"\n"
                  <<"Ny: "<<Ny<<"\n"
                  <<"Nz: "<<Nz<<std::endl;
        std::cout <<"# Type mx (10) and my (10)\n";
        std::cin >> mx[0] >> mx[1];
        std::cout << "# You typed\n"
                  <<"mx: "<<mx[0]<<"\n"
                  <<"my: "<<mx[1]<<std::endl;
        std::cout << "# Create parallel Derivative!\n";
    }
    MPI_Bcast( mx, 2, MPI_INT, 0, MPI_COMM_WORLD);

    const dg::CylindricalMPIGrid3d g3d( R_0-a, R_0+a, -a, a, 0, 2.*M_PI, n, Nx, Ny, Nz, dg::NEU, dg::NEU, dg::PER, comm);
    //create magnetic field
    const dg::geo::TokamakMagneticField mag = dg::geo::createCircularField( R_0, I_0);
    const dg::geo::BinaryVectorLvl0 bhat( (dg::geo::BHatR)(mag), (dg::geo::BHatZ)(mag), (dg::geo::BHatP)(mag));
    //create Fieldaligned object and construct DS from it
    dg::geo::Fieldaligned<dg::aProductMPIGeometry3d,dg::MIDMatrix,dg::MDVec>  dsFA( bhat, g3d, dg::NEU, dg::NEU, dg::geo::NoLimiter(), 1e-8, mx[0], mx[1]);
    dg::geo::DS<dg::aProductMPIGeometry3d, dg::MIDMatrix, dg::MDMatrix, dg::MDVec> ds( dsFA, dg::centered);
    ///##########################################################///
    //apply to function
    const dg::MDVec functionNEU = dg::evaluate( dg::geo::TestFunctionSin(mag), g3d);
    const dg::MDVec functionDIR = dg::evaluate( dg::geo::TestFunctionCos(mag), g3d);
    dg::MDVec derivative(functionNEU);
    dg::MDVec sol0 = dg::evaluate( dg::geo::DsFunction<dg::geo::TestFunctionSin>(mag), g3d);
    dg::MDVec sol1 = dg::evaluate( dg::geo::DssFunction<dg::geo::TestFunctionSin>(mag), g3d);
    dg::MDVec sol2 = dg::evaluate( dg::geo::DsDivFunction<dg::geo::TestFunctionSin>(mag), g3d);
    dg::MDVec sol3 = dg::evaluate( dg::geo::DsDivDsFunction<dg::geo::TestFunctionSin>(mag), g3d);
    std::vector<std::pair<std::string, const dg::MDVec&>> names{
         {"forward",sol0}, {"backward",sol0},
         {"centered",sol0}, {"dss",sol1},
         {"forwardDiv",sol2}, {"backwardDiv",sol2}, {"centeredDiv",sol2},
         {"forwardLap",sol3}, {"backwardLap",sol3}, {"centeredLap",sol3}
    };
    std::vector<std::pair<std::string, dg::direction>> namesLap{
         {"invForwardLap",dg::forward}, {"invBackwardLap",dg::backward}, {"invCenteredLap",dg::centered}
    };
    if(rank==0)std::cout << "# TEST NEU Boundary conditions!\n";
    if(rank==0)std::cout << "# TEST ADJOINT derivatives do unfortunately not fulfill Neumann BC!\n";
    ///##########################################################///
    if(rank==0)std::cout <<"Neumann:\n";
    dg::MDVec vol3d = dg::create::volume( g3d);
    for( const auto& name :  names)
    {
        callDS( ds, name.first, functionNEU, derivative);
        double sol = dg::blas2::dot( vol3d, name.second);
        dg::blas1::axpby( 1., name.second, -1., derivative);
        double norm = dg::blas2::dot( derivative, vol3d, derivative);
        if(rank==0)std::cout <<"    "<<name.first<<":"
                  <<std::setw(16-name.first.size())
                  <<" "<<sqrt(norm/sol)<<"\n";
    }
    ///##########################################################///
    dg::MDVec solution =dg::evaluate( dg::geo::OMDsDivDsFunction<dg::geo::TestFunctionSin>(mag), g3d);
    dg::Invert<dg::MDVec> invert( solution, max_iter, 1e-6, 1);
    dg::geo::TestInvertDS< dg::geo::DS<dg::aProductMPIGeometry3d, dg::MIDMatrix, dg::MDMatrix, dg::MDVec>, dg::MDVec>
        rhs(ds);
    for( auto name : namesLap)
    {
        ds.set_direction( name.second);
        dg::blas1::scal( derivative, 0);
        invert( rhs, derivative, solution);
        dg::blas1::axpby( 1., functionNEU, -1., derivative);
        double sol = dg::blas2::dot( vol3d, functionNEU);
        double norm = dg::blas2::dot( derivative, vol3d, derivative);
        if(rank==0)std::cout <<"    "<<name.first<<":"
                  <<std::setw(16-name.first.size())
                  <<" "<<sqrt(norm/sol)<<"\n";
    }
    ///##########################################################///
    if(rank==0)std::cout << "# Reconstruct parallel derivative!\n";
    dsFA.construct( bhat, g3d, dg::DIR, dg::DIR, dg::geo::NoLimiter(), 1e-8, mx[0], mx[1]);
    ds.construct( dsFA, dg::centered);
    sol0 = dg::evaluate( dg::geo::DsFunction<dg::geo::TestFunctionCos>(mag), g3d);
    sol1 = dg::evaluate( dg::geo::DssFunction<dg::geo::TestFunctionCos>(mag), g3d);
    sol2 = dg::evaluate( dg::geo::DsDivFunction<dg::geo::TestFunctionCos>(mag), g3d);
    sol3 = dg::evaluate( dg::geo::DsDivDsFunction<dg::geo::TestFunctionCos>(mag), g3d);
    if(rank==0)std::cout << "# TEST DIR Boundary conditions!\n";
    ///##########################################################///
    if(rank==0)std::cout << "Dirichlet: \n";
    for( const auto& name :  names)
    {
        callDS( ds, name.first, functionDIR, derivative);
        double sol = dg::blas2::dot( vol3d, name.second);
        dg::blas1::axpby( 1., name.second, -1., derivative);
        double norm = dg::blas2::dot( derivative, vol3d, derivative);
        if(rank==0)std::cout <<"    "<<name.first<<":"
                  <<std::setw(16-name.first.size())
                  <<" "<<sqrt(norm/sol)<<"\n";
    }
    ///##########################################################///
    solution =dg::evaluate( dg::geo::OMDsDivDsFunction<dg::geo::TestFunctionCos>(mag), g3d);
    for( auto name : namesLap)
    {
        ds.set_direction( name.second);
        invert(rhs, derivative, solution);
        dg::blas1::axpby( 1., functionDIR, -1., derivative);
        double sol = dg::blas2::dot( vol3d, functionDIR);
        double norm = dg::blas2::dot( derivative, vol3d, derivative);
        if(rank==0)std::cout <<"    "<<name.first<<":"
                  <<std::setw(16-name.first.size())
                  <<" "<<sqrt(norm/sol)<<"\n";
    }

    ///##########################################################///

    if(rank==0)std::cout << "TEST FIELDALIGNED EVALUATION of a Gaussian\n";
    dg::Gaussian init0(R_0+0.5, 0, 0.2, 0.2, 1);
    dg::GaussianZ modulate(0., M_PI/3., 1);
    dg::MDVec aligned = ds.fieldaligned().evaluate( init0, modulate, Nz/2, 2);
    ds( aligned, derivative);
    double norm = dg::blas2::dot(vol3d, derivative);
    if(rank==0)std::cout << "Norm Centered Derivative "<<sqrt( norm)<<" (compare with that of ds_t)\n";
    MPI_Finalize();
    return 0;
}
