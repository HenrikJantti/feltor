#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>
#include <sstream>
#include <cmath>
#include "json/json.h"

#include "dg/algorithm.h"

#include "curvilinear.h"
//#include "guenther.h"
#include "solovev.h"
#include "ribeiro.h"
//#include "ds.h"

#include "dg/file/nc_utilities.h"

thrust::host_vector<double> periodify( const thrust::host_vector<double>& in, const dg::Grid2d& g)
{
    thrust::host_vector<double> out(g.size());
    for( unsigned i=0; i<g.Ny()-1; i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[((i*g.n() + k)*g.Nx() + j)*g.n()+l] =
            in[((i*g.n() + k)*g.Nx() + j)*g.n()+l];
    for( unsigned i=g.Ny()-1; i<g.Ny(); i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[((i*g.n() + k)*g.Nx() + j)*g.n()+l] =
            in[((0*g.n() + k)*g.Nx() + j)*g.n()+l];
    return out;
}

double sineX( double x, double y) {return sin(x)*sin(y);}
double cosineX( double x, double y) {return cos(x)*sin(y);}
double sineY( double x, double y) {return sin(x)*sin(y);}
double cosineY( double x, double y) {return sin(x)*cos(y);}

int main( int argc, char* argv[])
{
    std::cout << "Type n, Nx, Ny, Nz\n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;
    Json::Value js;
    if( argc==1)
    {
        std::ifstream is("geometry_params_Xpoint.json");
        is >> js;
    }
    else
    {
        std::ifstream is(argv[1]);
        is >> js;
    }
    //write parameters from file into variables
    dg::geo::solovev::Parameters gp(js);
    dg::geo::CylindricalFunctorsLvl2 psip = dg::geo::solovev::createPsip( gp);
    std::cout << "Psi min "<<psip.f()(gp.R_0, 0)<<"\n";
    std::cout << "Type psi_0 and psi_1\n";
    double psi_0, psi_1;
    std::cin >> psi_0>> psi_1;
    gp.display( std::cout);
    dg::Timer t;
    //solovev::detail::Fpsi fpsi( gp, -10);
    std::cout << "Constructing ribeiro grid ... \n";
    t.tic();
    dg::geo::Ribeiro ribeiro( psip, psi_0, psi_1, gp.R_0, 0., 1);
    dg::geo::CurvilinearProductGrid3d g3d(ribeiro, n, Nx, Ny,Nz, dg::DIR);
    std::unique_ptr<dg::aGeometry2d> g2d( g3d.perp_grid());
    dg::Grid2d g2d_periodic(g2d->x0(), g2d->x1(), g2d->y0(), g2d->y1(), g2d->n(), g2d->Nx(), g2d->Ny()+1);
    t.toc();
    std::cout << "Construction took "<<t.diff()<<"s"<<std::endl;
    int ncid;
    dg::file::NC_Error_Handle err;
    err = nc_create( "ribeiro.nc", NC_NETCDF4|NC_CLOBBER, &ncid);
    int dim3d[2];
    err = dg::file::define_dimensions(  ncid, dim3d, g2d_periodic);
    int coordsID[2], onesID, defID, confID, volID,divBID;
    err = nc_def_var( ncid, "xc", NC_DOUBLE, 2, dim3d, &coordsID[0]);
    err = nc_def_var( ncid, "yc", NC_DOUBLE, 2, dim3d, &coordsID[1]);
    //err = nc_def_var( ncid, "zc", NC_DOUBLE, 3, dim3d, &coordsID[2]);
    err = nc_def_var( ncid, "psi", NC_DOUBLE, 2, dim3d, &onesID);
    err = nc_def_var( ncid, "deformation", NC_DOUBLE, 2, dim3d, &defID);
    err = nc_def_var( ncid, "conformal", NC_DOUBLE, 2, dim3d, &confID);
    err = nc_def_var( ncid, "volume", NC_DOUBLE, 2, dim3d, &volID);
    err = nc_def_var( ncid, "divB", NC_DOUBLE, 2, dim3d, &divBID);

    thrust::host_vector<double> psi_p = dg::pullback( psip.f(), *g2d);
    //g.display();
    err = nc_put_var_double( ncid, onesID, periodify(psi_p, g2d_periodic).data());
    dg::HVec X( g2d->map()[0]), Y(g2d->map()[1]);
    err = nc_put_var_double( ncid, coordsID[0], periodify(X, g2d_periodic).data());
    err = nc_put_var_double( ncid, coordsID[1], periodify(Y, g2d_periodic).data());

    dg::HVec temp0( g2d->size()), temp1(temp0);
    dg::HVec w2d = dg::create::weights( *g2d);

    dg::SparseTensor<dg::HVec> metric = g2d->metric();
    dg::HVec g_xx = metric.value(0,0), g_xy = metric.value(0,1), g_yy=metric.value(1,1);
    dg::HVec vol = dg::tensor::volume(metric);
    //err = nc_put_var_double( ncid, coordsID[2], g.z().data());
    //compute and write deformation into netcdf
    dg::blas1::pointwiseDivide( g_xy, g_xx, temp0);
    const dg::HVec ones = dg::evaluate( dg::one, *g2d);
    X=g_yy;
    err = nc_put_var_double( ncid, defID, periodify(X, g2d_periodic).data());
    //compute and write ribeiroratio into netcdf
    dg::blas1::pointwiseDivide( g_yy, g_xx, temp0);
    X=temp0;

    err = nc_put_var_double( ncid, confID, periodify(X, g2d_periodic).data());
    std::cout << "Construction successful!\n";

    //compute error in volume element (in ribeiro grid g^xx is the volume element)
    dg::blas1::pointwiseDot( g_xx, g_yy, temp0);
    dg::blas1::pointwiseDot( g_xy, g_xy, temp1);
    dg::blas1::axpby( 1., temp0, -1., temp1, temp0);
    dg::blas1::transfer( g_xx,  temp1);
    dg::blas1::pointwiseDot( temp1, temp1, temp1);
    dg::blas1::axpby( 1., temp1, -1., temp0, temp0);
    double error = sqrt( dg::blas2::dot( temp0, w2d, temp0)/dg::blas2::dot( temp1, w2d, temp1));
    std::cout<< "Rel Error in Determinant is "<<error<<"\n";

    //compute error in determinant vs volume form
    dg::blas1::pointwiseDot( g_xx, g_yy, temp0);
    dg::blas1::pointwiseDot( g_xy, g_xy, temp1);
    dg::blas1::axpby( 1., temp0, -1., temp1, temp0);
    dg::blas1::transform( temp0, temp0, dg::SQRT<double>());
    dg::blas1::pointwiseDivide( ones, temp0, temp0);
    dg::blas1::transfer( temp0, X);
    err = nc_put_var_double( ncid, volID, periodify(X, g2d_periodic).data());
    dg::blas1::axpby( 1., temp0, -1., vol, temp0);
    error = sqrt(dg::blas2::dot( temp0, w2d, temp0)/dg::blas2::dot( vol, w2d, vol));
    std::cout << "Rel Consistency  of volume is "<<error<<"\n";

    //compare g^xx to volume form
    dg::blas1::transfer( g_xx, temp0);
    dg::blas1::pointwiseDivide( ones, temp0, temp0);
    dg::blas1::axpby( 1., temp0, -1., vol, temp0);
    error=sqrt(dg::blas2::dot( temp0, w2d, temp0))/sqrt( dg::blas2::dot(vol, w2d, vol));
    std::cout << "Rel Error of volume form is "<<error<<"\n";

    vol = dg::create::volume( g3d);
    dg::HVec ones3d = dg::evaluate( dg::one, g3d);
    double volume = dg::blas1::dot( vol, ones3d);

    std::cout << "TEST VOLUME IS:\n";
    double psipmin, psipmax;
    if( psi_0 < psi_1) psipmax = psi_1, psipmin = psi_0;
    else               psipmax = psi_0, psipmin = psi_1;
    auto iris = dg::compose( dg::Iris(psipmin, psipmax), psip.f());
    //dg::CylindricalGrid3d<dg::HVec> g3d( gp.R_0 -2.*gp.a, gp.R_0 + 2*gp.a, -2*gp.a, 2*gp.a, 0, 2*M_PI, 3, 2200, 2200, 1, dg::PER, dg::PER, dg::PER);
//     dg::CartesianGrid2d g2dC( gp.R_0 -1.2*gp.a, gp.R_0 + 1.2*gp.a, -1.2*gp.a, 1.2*gp.a, 1, 1e3, 1e3, dg::PER, dg::PER);
    dg::CartesianGrid2d g2dC( gp.R_0 -2.0*gp.a, gp.R_0 + 2.0*gp.a, -2.0*gp.a, 2.0*gp.a, 1, 2e3, 2e3, dg::PER, dg::PER);
    dg::HVec vec  = dg::evaluate( iris, g2dC);
    dg::HVec R  = dg::evaluate( dg::cooX2d, g2dC);
    dg::HVec g2d_weights = dg::create::volume( g2dC);
    double volumeRZP = 2.*M_PI*dg::blas2::dot( vec, g2d_weights, R);
    std::cout << "volumeXYP is "<< volume<<std::endl;
    std::cout << "volumeRZP is "<< volumeRZP<<std::endl;
    std::cout << "relative difference in volume is "<<fabs(volumeRZP - volume)/volume<<std::endl;
    std::cout << "Note that the error might also come from the volume in RZP!\n"; //since integration of jacobian is fairly good probably

    return 0;
}
