#include "hip/hip_runtime.h"
#include <iostream>

#include <cusp/print.h>
#define DG_BENCHMARK
#include "dg/algorithm.h"
#include "magnetic_field.h"
#include "ds.h"
#include "toroidal.h"


const double R_0 = 10;
const double I_0 = 20; //q factor at r=1 is I_0/R_0
const double a  = 1; //small radius
double funcNEU(double R, double Z, double phi)
{
    return sin(M_PI*(R-R_0)/2.)*sin(M_PI*Z/2.)*sin(phi);
}
double deriNEU(double R, double Z, double phi)
{
    double r2 = (R-R_0)*(R-R_0)+Z*Z; //(grad psi)^2
    return ( Z     *M_PI/2.*cos(M_PI*(R-R_0)/2.)*sin(M_PI*Z/2.)*sin(phi)
           -(R-R_0)*M_PI/2.*sin(M_PI*(R-R_0)/2.)*cos(M_PI*Z/2.)*sin(phi)
           + I_0/R*sin(M_PI*(R-R_0)/2.)*sin(M_PI*Z/2.)*cos(phi)
           )/sqrt(I_0*I_0+r2);
}
double deriAdjNEU(double R, double Z, double phi)
{
    double r2 = (R-R_0)*(R-R_0)+Z*Z; //(grad psi)^2
    return Z/R/(I_0*I_0+r2)*funcNEU(R,Z,phi) + deriNEU(R,Z,phi);
}
double funcDIR(double R, double Z, double phi)
{
    return cos(M_PI*(R-R_0)/2.)*cos(M_PI*Z/2.)*sin(phi);
}
double deriDIR(double R, double Z, double phi)
{
    double r2 = (R-R_0)*(R-R_0)+Z*Z; //(grad psi)^2
    return (-Z      *M_PI/2.*sin(M_PI*(R-R_0)/2.)*cos(M_PI*Z/2.)*sin(phi)
            +(R-R_0)*M_PI/2.*cos(M_PI*(R-R_0)/2.)*sin(M_PI*Z/2.)*sin(phi)
            +I_0/R*cos(M_PI*(R-R_0)/2.)*cos(M_PI*Z/2.)*cos(phi)
           )/sqrt(I_0*I_0+r2);
}
double deriAdjDIR(double R, double Z, double phi)
{
    double r2 = (R-R_0)*(R-R_0)+Z*Z; //(grad psi)^2
    return Z/R/(I_0*I_0+r2)*funcDIR(R,Z,phi) + deriDIR(R,Z,phi);
}

int main(int argc, char * argv[])
{
    std::cout << "First test the cylindrical version\n";
    std::cout << "Type n (3), Nx(20), Ny(20), Nz(20)\n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;
    std::cout << "You typed "<<n<<" "<<Nx<<" "<<Ny<<" "<<Nz<<std::endl;
    std::cout << "Type mx (10) and my (10)\n";
    unsigned mx, my;
    std::cin >> mx>> my;
    std::cout << "You typed "<<mx<<" "<<my<<std::endl;
    std::cout << "Create parallel Derivative!\n";

    //![doxygen]
    const dg::CylindricalGrid3d g3d( R_0 - a, R_0+a, -a, a, 0, 2.*M_PI, n, Nx, Ny, Nz, dg::DIR, dg::DIR);
    //create magnetic field
    const dg::geo::TokamakMagneticField mag = dg::geo::createCircularField( R_0, I_0);
    const dg::geo::BinaryVectorLvl0 bhat( (dg::geo::BHatR)(mag), (dg::geo::BHatZ)(mag), (dg::geo::BHatP)(mag));
    //create Fieldaligned object and construct DS from it
    dg::geo::Fieldaligned<dg::aProductGeometry3d,dg::IDMatrix,dg::DVec>  dsFA( bhat, g3d, dg::NEU, dg::NEU, dg::geo::NoLimiter(), 1e-8, mx, my, true,true,true);
    dg::geo::DS<dg::aProductGeometry3d, dg::IDMatrix, dg::DMatrix, dg::DVec> ds( dsFA, dg::not_normed, dg::centered);
    ///##########################################################///
    //apply to function
    dg::DVec function = dg::evaluate( funcNEU, g3d), derivative(function);
    ds.centered( function, derivative);
    //![doxygen]
    std::cout << "TEST NEU Boundary conditions!\n";
    dg::DVec solution = dg::evaluate( deriNEU, g3d);
    const dg::DVec vol3d = dg::create::volume( g3d);
    double sol = dg::blas2::dot( vol3d, solution);
    dg::blas1::axpby( 1., solution, -1., derivative);
    double norm = dg::blas2::dot( derivative, vol3d, derivative);
    std::cout << "Error centered derivative \t"<< sqrt( norm/sol )<<"\n";
    ds.forward( 1., function, 0., derivative);
    dg::blas1::axpby( 1., solution, -1., derivative);
    norm = dg::blas2::dot(vol3d, derivative);
    std::cout << "Error Forward  Derivative \t"<<sqrt( norm/sol)<<"\n";
    ds.backward( 1., function, 0., derivative);
    dg::blas1::axpby( 1., solution, -1., derivative);
    norm = dg::blas2::dot(vol3d, derivative);
    std::cout << "Error Backward Derivative \t"<<sqrt( norm/sol)<<"\n";
    ///We unfortunately cannot test convergence of adjoint because
    ///b and therefore bf does not fulfill Neumann boundary conditions
    ///##########################################################///
    std::cout << "TEST DIR Boundary conditions!\n";
    dsFA.construct( bhat, g3d, dg::DIR, dg::DIR, dg::geo::NoLimiter(), 1e-8, mx, my, true,true,true);
    ds.construct( dsFA, dg::not_normed, dg::centered);
    //apply to function
    dg::DVec functionDIR = dg::evaluate( funcDIR, g3d);
    solution = dg::evaluate( deriDIR, g3d);
    sol = dg::blas2::dot( vol3d, solution);

    ds.centered( functionDIR, derivative);
    dg::blas1::axpby( 1., solution, -1., derivative);
    norm = dg::blas2::dot( derivative, vol3d, derivative);
    std::cout << "Error centered derivative \t"<< sqrt( norm/sol )<<"\n";
    ds.forward( 1., functionDIR, 0., derivative);
    dg::blas1::axpby( 1., solution, -1., derivative);
    norm = dg::blas2::dot(vol3d, derivative);
    std::cout << "Error Forward  Derivative \t"<<sqrt( norm/sol)<<"\n";
    ds.backward( 1., functionDIR, 0., derivative);
    dg::blas1::axpby( 1., solution, -1., derivative);
    norm = dg::blas2::dot(vol3d, derivative);
    std::cout << "Error Backward Derivative \t"<<sqrt( norm/sol)<<"\n";

    ///##########################################################///
    std::cout << "TEST ADJOINT derivatives!\n";
    solution = dg::evaluate( deriAdjDIR, g3d);
    sol = dg::blas2::dot( vol3d, solution);

    ds.centeredDiv( functionDIR, derivative);
    dg::blas1::axpby( 1., solution, -1., derivative);
    norm = dg::blas2::dot( derivative, vol3d, derivative);
    std::cout << "Error centered divergence \t"<< sqrt( norm/sol )<<"\n";
    ds.forwardDiv( 1., functionDIR, 0., derivative);
    dg::blas1::axpby( 1., solution, -1., derivative);
    norm = dg::blas2::dot(vol3d, derivative);
    std::cout << "Error Forward  divergence \t"<<sqrt( norm/sol)<<"\n";
    ds.backwardDiv( 1., functionDIR, 0., derivative);
    dg::blas1::axpby( 1., solution, -1., derivative);
    norm = dg::blas2::dot(vol3d, derivative);
    std::cout << "Error Backward divergence \t"<<sqrt( norm/sol)<<"\n";

    ///##########################################################///
    std::cout << "TEST FIELDALIGNED EVALUATION of a Gaussian\n";
    dg::Gaussian init0(R_0+0.5, 0, 0.2, 0.2, 1);
    dg::GaussianZ modulate(0., M_PI/3., 1);
    function = dsFA.evaluate( init0, modulate, Nz/2, 2);
    ds( function, derivative);
    norm = dg::blas2::dot(vol3d, derivative);
    std::cout << "Norm Centered Derivative "<<sqrt( norm)<<" (compare with that of ds_mpit)\n";

    return 0;
}
