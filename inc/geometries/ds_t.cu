#include "hip/hip_runtime.h"
#include <iostream>

#include <cusp/print.h>

#include "dg/backend/functions.h"
#include "dg/backend/timer.cuh"
#include "dg/blas.h"
#include "dg/functors.h"
#include "dg/geometry/geometry.h"
#include "magnetic_field.h"
#include "ds.h"
#include "toroidal.h"


const double R_0 = 10;
const double I_0 = 20; //q factor at r=1 is I_0/R_0
const double a  = 1; //small radius  
dg::geo::TokamakMagneticField mag = dg::geo::createCircularField( R_0, I_0);
dg::geo::GradLnB gradLnB(mag);
double func(double R, double Z, double phi)
{
    double r2 = (R-R_0)*(R-R_0)+Z*Z;
    return r2*sin(phi);
}
double deri(double R, double Z, double phi)
{
    double r2 = (R-R_0)*(R-R_0)+Z*Z; //(grad psi)^2
    return I_0/R/sqrt(I_0*I_0 + r2)* r2*cos(phi);
}
double adjoint(double R, double Z, double phi)
{
    return -gradLnB(R,Z)*func(R,Z,phi) + deri(R,Z,phi);
}

int main(int argc, char * argv[])
{
    std::cout << "First test the cylindrical version\n";
    std::cout << "Type n, Nx, Ny, Nz\n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;
    std::cout << "You typed "<<n<<" "<<Nx<<" "<<Ny<<" "<<Nz<<std::endl;
    std::cout << "Type mx and my\n";
    unsigned mx, my;
    std::cin >> mx>> my;
    std::cout << "You typed "<<mx<<" "<<my<<std::endl;
    dg::CylindricalGrid3d g3d( R_0 - 1, R_0+1, -1, 1, 0, 2.*M_PI, n, Nx, Ny, Nz, dg::NEU, dg::NEU, dg::PER);
    const dg::DVec vol3d = dg::create::volume( g3d);
    std::cout << "Create parallel Derivative!\n";
    dg::geo::BinaryVectorLvl0 bhat( (dg::geo::BHatR)(mag), (dg::geo::BHatZ)(mag), (dg::geo::BHatP)(mag));
    dg::geo::Fieldaligned<dg::aProductGeometry3d,dg::IDMatrix,dg::DVec>  dsFA( bhat, g3d, mx, my, true,true, 1e-6, dg::NEU, dg::NEU, dg::geo::NoLimiter());
    dg::geo::DS<dg::aProductGeometry3d, dg::IDMatrix, dg::DMatrix, dg::DVec> ds( dsFA, dg::not_normed, dg::centered);

    ///##########################################################///
    dg::DVec function = dg::evaluate( func, g3d), derivative(function);
    const dg::DVec solution = dg::evaluate( deri, g3d);
    ds( function, derivative);
    dg::blas1::axpby( 1., solution, -1., derivative);
    double norm = dg::blas2::dot( derivative, vol3d, derivative);
    const double sol = dg::blas2::dot( vol3d, solution);
    std::cout << "Error centered derivative "<< sqrt( norm/sol )<<"\n";
    ds.forward( 1., function, 0., derivative);
    dg::blas1::axpby( 1., solution, -1., derivative);
    norm = dg::blas2::dot(vol3d, derivative);
    std::cout << "Error Forward  Derivative "<<sqrt( norm/sol)<<"\n";
    ds.backward( 1., function, 0., derivative);
    dg::blas1::axpby( 1., solution, -1., derivative);
    norm = dg::blas2::dot(vol3d, derivative);
    std::cout << "Error Backward Derivative "<<sqrt( norm/sol)<<"\n";
    std::cout << "(Since the function is a parabola, the error is from the parallel derivative only if n>2/ no interpolation error)\n"; 
    ///##########################################################///
    std::cout << "TEST FIELDALIGNED EVALUATION of a Gaussian\n";
    dg::Gaussian init0(R_0+0.5, 0, 0.2, 0.2, 1);
    dg::GaussianZ modulate(0., M_PI/3., 1);
    function = ds.fieldaligned().evaluate( init0, modulate, Nz/2, 2);
    ds( function, derivative);
    norm = dg::blas2::dot(vol3d, derivative);
    std::cout << "Norm Centered Derivative "<<sqrt( norm)<<" (compare with that of ds_mpib)\n";
    ///##########################################################///
    std::cout << "TEST ADJOINT DERIVATIVE! \n";
    function = dg::evaluate( func, g3d);
    const dg::DVec adjoint_solution = dg::evaluate( adjoint, g3d);
    const double adj = dg::blas2::dot( vol3d, adjoint_solution);

    ds.centeredAdj( -1., function, 0., derivative);
    dg::blas1::axpby( 1., adjoint_solution, -1., derivative);
    norm = dg::blas2::dot( vol3d, derivative);
    std::cout << "Error centered derivative "<< sqrt( norm/adj )<<"\n";
    ds.forwardAdj( -1., function, 0., derivative);
    dg::blas1::axpby( 1., adjoint_solution, -1., derivative);
    norm = dg::blas2::dot(vol3d, derivative);
    std::cout << "Error Forward  Derivative "<<sqrt( norm/adj)<<"\n";
    ds.backwardAdj( -1., function, 0., derivative);
    dg::blas1::axpby( 1., adjoint_solution, -1., derivative);
    norm = dg::blas2::dot(vol3d, derivative);
    std::cout << "Error Backward Derivative "<<sqrt( norm/adj)<<"\n";

    return 0;
}
