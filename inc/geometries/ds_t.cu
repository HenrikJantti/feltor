#include "hip/hip_runtime.h"
#include <iostream>

#include <cusp/print.h>
#include "dg/algorithm.h"
#include "magnetic_field.h"
#include "testfunctors.h"
#define DG_BENCHMARK
#include "ds.h"
#include "toroidal.h"


const double R_0 = 10;
const double I_0 = 20; //q factor at r=1 is I_0/R_0
const double a  = 1; //small radius

int main(int argc, char * argv[])
{
    std::cout << "# This program tests the parallel derivative DS in cylindrical coordinates for circular flux surfaces with DIR and NEU boundary conditions.\n";
    std::cout << "# Type n (3), Nx(20), Ny(20), Nz(20)\n";
    unsigned n, Nx, Ny, Nz, mx, my, max_iter;
    std::cin >> n>> Nx>>Ny>>Nz;
    std::cout <<"# You typed\n"
              <<"n: "<<n<<"\n"
              <<"Nx: "<<Nx<<"\n"
              <<"Ny: "<<Ny<<"\n"
              <<"Nz: "<<Nz<<std::endl;
    std::cout << "# Type mx (10) and my (10)\n";
    std::cin >> mx>> my;
    std::cout << "# You typed\n"
              <<"mx: "<<mx<<"\n"
              <<"my: "<<my<<std::endl;
    std::cout << "# Type max iterations (1000) \n";
    std::cin >> max_iter;
    std::cout << "# You typed\n"
              <<"max_iter: "<<max_iter<<"\n";
    std::cout << "# Create parallel Derivative!\n";

    //![doxygen]
    const dg::CylindricalGrid3d g3d( R_0 - a, R_0+a, -a, a, 0, 2.*M_PI, n, Nx, Ny, Nz, dg::NEU, dg::NEU, dg::PER);
    //create magnetic field
    const dg::geo::TokamakMagneticField mag = dg::geo::createCircularField( R_0, I_0);
    const dg::geo::BinaryVectorLvl0 bhat( (dg::geo::BHatR)(mag), (dg::geo::BHatZ)(mag), (dg::geo::BHatP)(mag));
    //create Fieldaligned object and construct DS from it
    dg::geo::Fieldaligned<dg::aProductGeometry3d,dg::IDMatrix,dg::DVec>  dsFA( bhat, g3d, dg::NEU, dg::NEU, dg::geo::NoLimiter(), 1e-8, mx, my);
    dg::geo::DS<dg::aProductGeometry3d, dg::IDMatrix, dg::DMatrix, dg::DVec> ds( dsFA, dg::normed, dg::centered);
    //![doxygen]
    ///##########################################################///
    //apply to function
    const dg::DVec function = dg::evaluate( dg::geo::FunctionSinNEU(mag), g3d);
    dg::DVec derivative(function);
    dg::DVec sol0 = dg::evaluate( dg::geo::DsFunction<dg::geo::FunctionSinNEU>(mag), g3d);
    dg::DVec sol1 = dg::evaluate( dg::geo::DssFunction<dg::geo::FunctionSinNEU>(mag), g3d);
    dg::DVec sol2 = dg::evaluate( dg::geo::DsDivFunction<dg::geo::FunctionSinNEU>(mag), g3d);
    dg::DVec sol3 = dg::evaluate( dg::geo::DsDivDsFunction<dg::geo::FunctionSinNEU>(mag), g3d);
    std::vector<std::pair<std::string, const dg::DVec&>> names{
         {"forward",sol0}, {"backward",sol0},
         {"centered",sol0}, {"dss",sol1},
         {"forwardDiv",sol2}, {"backwardDiv",sol2}, {"centeredDiv",sol2},
         {"forwardLap",sol3}, {"backwardLap",sol3}, {"centeredLap",sol3}
    };
    std::cout << "# TEST NEU Boundary conditions!\n";
    std::cout << "# TEST ADJOINT derivatives do unfortunately not fulfill Neumann BC!\n";
    ///##########################################################///
    std::cout <<"Neumann:\n";
    const dg::DVec vol3d = dg::create::volume( g3d);
    for( const auto& name :  names)
    {
        callDS( ds, name.first, function, derivative);
        double sol = dg::blas2::dot( vol3d, name.second);
        dg::blas1::axpby( 1., name.second, -1., derivative);
        double norm = dg::blas2::dot( derivative, vol3d, derivative);
        std::cout <<"    "<<name.first<<":"
                  <<std::setw(16-name.first.size())
                  <<" "<<sqrt(norm/sol)<<"\n";
    }
    ///##########################################################///
    dg::DVec solution = dg::evaluate( dg::geo::DsDivDsFunction<dg::geo::FunctionSinNEU>(mag), g3d);
    ds.set_direction( dg::forward);
    ds.set_norm( dg::not_normed);
    dg::geo::DSS< dg::geo::DS<dg::aProductGeometry3d, dg::IDMatrix, dg::DMatrix, dg::DVec>, dg::DVec> dss( ds);
    dg::Invert<dg::DVec> invert( solution, max_iter, 1e-5);
    invert( dss, derivative, solution);

    double sol = dg::blas2::dot( vol3d, function);
    dg::blas1::axpby( 1., function, 1., derivative);
    double norm = dg::blas2::dot( derivative, vol3d, derivative);
    std::cout << "    invForwardLap:   "<< sqrt( norm/sol )<<"\n";

    ///##########################################################///
    std::cout << "# Reconstruct parallel derivative!\n";
    dsFA.construct( bhat, g3d, dg::DIR, dg::DIR, dg::geo::NoLimiter(), 1e-8, mx, my);
    ds.construct( dsFA, dg::normed, dg::centered);
    const dg::DVec functionDIR = dg::evaluate( dg::geo::FunctionSinDIR(mag), g3d);
    sol0 = dg::evaluate( dg::geo::DsFunction<dg::geo::FunctionSinDIR>(mag), g3d);
    sol1 = dg::evaluate( dg::geo::DssFunction<dg::geo::FunctionSinDIR>(mag), g3d);
    sol2 = dg::evaluate( dg::geo::DsDivFunction<dg::geo::FunctionSinDIR>(mag), g3d);
    sol3 = dg::evaluate( dg::geo::DsDivDsFunction<dg::geo::FunctionSinDIR>(mag), g3d);
    std::cout << "# TEST DIR Boundary conditions!\n";
    ///##########################################################///
    std::cout << "Dirichlet: \n";
    for( const auto& name :  names)
    {
        callDS( ds, name.first, functionDIR, derivative);
        double sol = dg::blas2::dot( vol3d, name.second);
        dg::blas1::axpby( 1., name.second, -1., derivative);
        double norm = dg::blas2::dot( derivative, vol3d, derivative);
        std::cout <<"    "<<name.first<<":"
                  <<std::setw(16-name.first.size())
                  <<" "<<sqrt(norm/sol)<<"\n";
    }
    ///##########################################################///
    solution = dg::evaluate( dg::geo::DsDivDsFunction<dg::geo::FunctionSinDIR>(mag), g3d);
    ds.set_direction( dg::forward);
    ds.set_norm( dg::not_normed);
    dg::Invert<dg::DVec> invertDIR( solution, max_iter, 1e-5);
    invertDIR( dss, derivative, solution);
    sol = dg::blas2::dot( vol3d, functionDIR);

    ////ds.symv( functionDIR, derivative);
    dg::blas1::axpby( 1., functionDIR, 1., derivative);
    norm = dg::blas2::dot( derivative, vol3d, derivative);
    std::cout << "    invForwardLap:   "<< sqrt( norm/sol )<<"\n";

    ///##########################################################///
    std::cout << "# TEST FIELDALIGNED EVALUATION of a Gaussian\n";
    dg::Gaussian init0(R_0+0.5, 0, 0.2, 0.2, 1);
    dg::GaussianZ modulate(0., M_PI/3., 1);
    dg::DVec aligned = dsFA.evaluate( init0, modulate, Nz/2, 2);
    ds( aligned, derivative);
    norm = dg::blas2::dot(vol3d, derivative);
    std::cout << "# Norm Centered Derivative "<<sqrt( norm)<<" (compare with that of ds_mpit)\n";

    return 0;
}
