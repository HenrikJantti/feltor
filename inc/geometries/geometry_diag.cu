#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>
#include <functional>
#include <sstream>
#include <ctime>
#include <cmath>

#include "dg/algorithm.h"
#include "dg/file/file.h"

#include "solovev.h"
//#include "taylor.h"
#include "magnetic_field.h"
#include "testfunctors.h"
#include "curvilinearX.h"
#include "separatrix_orthogonal.h"
#include "average.h"


// - write magnetic functions into file
// - compute Flux - surface averages and write into file
struct Parameters
{
    unsigned n, Nx, Ny, Nz, Npsi;
    double boxscaleRm, boxscaleRp;
    double boxscaleZm, boxscaleZp;
    double amp, k_psi, nprofileamp;
    double sigma, posX, posY;
    double damping_boundary, source_alpha, damping_alpha, source_boundary;
    double profile_alpha;
    Parameters( const Json::Value& js){
        n = js.get("n",3).asUInt();
        Nx = js.get("Nx",100).asUInt()/js["compression"].get(0u,1).asUInt();
        Ny = js.get("Ny",100).asUInt()/js["compression"].get(1u,1).asUInt();
        Nz = js.get("Nz", 1).asUInt();
        Npsi = js.get("Npsi", 32).asUInt();
        boxscaleRm = js["box"]["scaleR"].get(0u, 1.1).asDouble();
        boxscaleRp = js["box"]["scaleR"].get(1u, 1.1).asDouble();
        boxscaleZm = js["box"]["scaleZ"].get(0u, 1.2).asDouble();
        boxscaleZp = js["box"]["scaleZ"].get(1u, 1.1).asDouble();
        amp = js.get("amplitude", 1.).asDouble();
        k_psi = js.get("k_psi", 1.).asDouble();
        nprofileamp = js["profile"].get("amp", 1.).asDouble();
        profile_alpha = js["profile"].get("alpha", 0.1).asDouble();
        sigma = js.get("sigma", 10).asDouble();
        posX = js.get("posX", 0.5).asDouble();
        posY = js.get("posY", 0.5).asDouble();
        damping_boundary = js["damping"].get("boundary", 1.2).asDouble();
        damping_alpha = js["damping"].get("alpha", 0.1).asDouble();
        source_alpha = js["source"].get("alpha", 0.5).asDouble();
        source_boundary = js["source"].get("boundary", 0.5).asDouble();
    }
    void display( std::ostream& os = std::cout ) const
    {
        os << "Input parameters are: \n";
        os  <<" n             = "<<n<<"\n"
            <<" Nx            = "<<Nx<<"\n"
            <<" Ny            = "<<Ny<<"\n"
            <<" Nz            = "<<Nz<<"\n"
            <<" Npsi          = "<<Npsi<<"\n"
            <<" boxscaleRm    = "<<boxscaleRm<<"\n"
            <<" boxscaleRp    = "<<boxscaleRp<<"\n"
            <<" boxscaleZm    = "<<boxscaleZm<<"\n"
            <<" boxscaleZp    = "<<boxscaleZp<<"\n"
            <<" source bound  = "<<source_boundary<<"\n"
            <<" source alpha  = "<<source_alpha<<"\n"
            <<" damping bound = "<<damping_boundary<<"\n"
            <<" damping alpha = "<<damping_alpha<<"\n"
            <<" amp           = "<<amp<<"\n"
            <<" k_psi         = "<<k_psi<<"\n"
            <<" nprofileamp   = "<<nprofileamp<<"\n"
            <<" sigma         = "<<sigma<<"\n"
            <<" posX          = "<<posX<<"\n"
            <<" posY          = "<<posY<<"\n";
        os << std::flush;
    }
};


int main( int argc, char* argv[])
{
    std::string newfilename;
    Json::Value input_js, geom_js;
    if( argc == 4)
    {
        newfilename = argv[3];
        std::cout << argv[0]<< " "<<argv[1]<<" & "<<argv[2]<<" -> " <<argv[3]<<std::endl;
        file::file2Json( argv[1], input_js, file::comments::are_discarded);
        file::file2Json( argv[2], geom_js, file::comments::are_discarded);
    }
    else if( argc == 3)
    {
        newfilename = argv[2];
        std::cout << argv[0]<< " "<<argv[1]<<" -> " <<argv[2]<<std::endl;
        file::NC_Error_Handle err;
        int ncid_in;
        err = nc_open( argv[1], NC_NOWRITE, &ncid_in); //open 3d file
        size_t length;
        err = nc_inq_attlen( ncid_in, NC_GLOBAL, "inputfile", &length);
        std::string inputfile(length, 'x');
        err = nc_get_att_text( ncid_in, NC_GLOBAL, "inputfile", &inputfile[0]);
        err = nc_inq_attlen( ncid_in, NC_GLOBAL, "geomfile", &length);
        std::string geomfile(length, 'x');
        err = nc_get_att_text( ncid_in, NC_GLOBAL, "geomfile", &geomfile[0]);
        err = nc_close( ncid_in);
        Json::Value js,gs;
        file::string2Json(inputfile, input_js, file::comments::are_discarded);
        file::string2Json(geomfile, geom_js, file::comments::are_discarded);
    }
    else
    {
        std::cerr << "ERROR: Wrong number of arguments!\n";
        std::cerr << " Usage: "<< argv[0]<<" [input.json] [geom.json] [output.nc]\n";
        std::cerr << " ( Minimum input json file is { \"n\" : 3, \"Nx\": 100, \"Ny\":100 })\n";
        std::cerr << "Or \n Usage: "<< argv[0]<<" [file.nc] [output.nc]\n";
        std::cerr << " ( Program searches for string variables 'inputfile' and 'geomfile' in file.nc and tries a json parser)\n";
        return -1;
    }
    std::cout << input_js<<std::endl;
    const Parameters p(input_js);
    const dg::geo::solovev::Parameters gp(geom_js);
    p.display( std::cout);
    gp.display( std::cout);
    std::string input = input_js.toStyledString();
    std::string geom = geom_js.toStyledString();
    unsigned n, Nx, Ny, Nz;
    n = p.n, Nx = p.Nx, Ny = p.Ny, Nz = p.Nz;
    double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscaleRp*gp.a;
    double Zmax=p.boxscaleZp*gp.a*gp.elongation;

    //Test coefficients
    dg::geo::TokamakMagneticField mag_origin = dg::geo::createSolovevField(gp);
    dg::geo::TokamakMagneticField mag = mag_origin;
    //Find O-point
    double RO = gp.R_0, ZO = 0.;
    int point = 1;
    if( !gp.isToroidal() )
        point = dg::geo::findOpoint( mag.get_psip(), RO, ZO);
    const double psipO = mag.psip()( RO, ZO);
    std::cout << "O-point found at "<<RO<<" "<<ZO<<" with Psip "<<psipO<<std::endl;
    if( point == 1 )
        std::cout << " (minimum)"<<std::endl;
    if( point == 2 )
        std::cout << " (maximum)"<<std::endl;
    const double psip0 = mag.psip()(gp.R_0, 0);
    std::cout << "psip( R_0, 0) = "<<psip0<<"\n";
    if( p.damping_alpha > 0.)
    {
        double damping_psi0p = (1.-p.damping_boundary*p.damping_boundary)*psipO;
        double damping_alphap = -(2.*p.damping_boundary+p.damping_alpha)*p.damping_alpha*psipO;
        std::cout<< " damping "<< damping_psi0p << " "<<damping_alphap<<"\n";
        mag = dg::geo::createModifiedSolovevField(gp, damping_psi0p+damping_alphap/2., fabs(damping_alphap/2.), ((psipO>0)-(psipO<0)));
    }


    dg::Grid2d grid2d(Rmin,Rmax,Zmin,Zmax, n,Nx,Ny);
    dg::DVec psipog2d   = dg::evaluate( mag.psip(), grid2d);
    std::vector<std::tuple<std::string, dg::HVec, std::string> > map1d;
    ///////////TEST CURVILINEAR GRID TO COMPUTE FSA QUANTITIES
    unsigned npsi = 3, Npsi = p.Npsi;//set number of psivalues (NPsi % 8 == 0)
    //Generate list of functions to evaluate
    std::vector< std::tuple<std::string, std::string, dg::geo::CylindricalFunctor >> map{
        {"Psip", "Flux function", mag.psip()},
        {"PsipR", "Flux function derivative in R", mag.psipR()},
        {"PsipZ", "Flux function derivative in Z", mag.psipZ()},
        {"PsipRR", "Flux function derivative in RR", mag.psipRR()},
        {"PsipRZ", "Flux function derivative in RZ", mag.psipRZ()},
        {"PsipZZ", "Flux function derivative in ZZ", mag.psipZZ()},
        {"Ipol", "Poloidal current", mag.ipol()},
        {"IpolR", "Poloidal current derivative in R", mag.ipolR()},
        {"IpolZ", "Poloidal current derivative in Z", mag.ipolZ()},
        {"Rho_p", "Normalized Poloidal flux label", dg::geo::RhoP(mag)},
        {"Bmodule", "Magnetic field strength", dg::geo::Bmodule(mag)},
        {"InvB", "Inverse of Bmodule", dg::geo::InvB(mag)},
        {"LnB", "Natural logarithm of Bmodule", dg::geo::LnB(mag)},
        {"GradLnB", "The parallel derivative of LnB", dg::geo::GradLnB(mag)},
        {"Divb", "The divergence of the magnetic unit vector", dg::geo::Divb(mag)},
        {"B_R", "Derivative of Bmodule in R", dg::geo::BR(mag)},
        {"B_Z", "Derivative of Bmodule in Z", dg::geo::BZ(mag)},
        {"CurvatureNablaBR",  "R-component of the (toroidal) Nabla B curvature vector", dg::geo::CurvatureNablaBR(mag,+1)},
        {"CurvatureNablaBZ",  "Z-component of the (toroidal) Nabla B curvature vector", dg::geo::CurvatureNablaBZ(mag,+1)},
        {"CurvatureKappaR",   "R-component of the (toroidal) Kappa B curvature vector", dg::geo::CurvatureKappaR(mag,+1)},
        {"CurvatureKappaZ",   "Z-component of the (toroidal) Kappa B curvature vector", dg::geo::CurvatureKappaZ(mag,+1)},
        {"DivCurvatureKappa", "Divergence of the (toroidal) Kappa B curvature vector", dg::geo::DivCurvatureKappa(mag,+1)},
        {"DivCurvatureNablaB","Divergence of the (toroidal) Nabla B curvature vector", dg::geo::DivCurvatureNablaB(mag,+1)},
        {"TrueCurvatureNablaBR", "R-component of the (true) Nabla B curvature vector", dg::geo::TrueCurvatureNablaBR(mag)},
        {"TrueCurvatureNablaBZ", "Z-component of the (true) Nabla B curvature vector", dg::geo::TrueCurvatureNablaBZ(mag)},
        {"TrueCurvatureNablaBP", "Contravariant Phi-component of the (true) Nabla B curvature vector", dg::geo::TrueCurvatureNablaBP(mag)},
        {"TrueCurvatureKappaR", "R-component of the (true) Kappa B curvature vector", dg::geo::TrueCurvatureKappaR(mag)},
        {"TrueCurvatureKappaZ", "Z-component of the (true) Kappa B curvature vector", dg::geo::TrueCurvatureKappaZ(mag)},
        {"TrueCurvatureKappaP", "Contravariant Phi-component of the (true) Kappa B curvature vector", dg::geo::TrueCurvatureKappaP(mag)},
        {"TrueDivCurvatureKappa", "Divergence of the (true) Kappa B curvature vector", dg::geo::TrueDivCurvatureKappa(mag)},
        {"TrueDivCurvatureNablaB","Divergence of the (true) Nabla B curvature vector",  dg::geo::TrueDivCurvatureNablaB(mag)},
        {"BFieldR", "R-component of the magnetic field vector", dg::geo::BFieldR(mag)},
        {"BFieldZ", "Z-component of the magnetic field vector", dg::geo::BFieldZ(mag)},
        {"BFieldP", "Contravariant Phi-component of the magnetic field vector", dg::geo::BFieldP(mag)},
        {"BHatR", "R-component of the magnetic field unit vector", dg::geo::BHatR(mag)},
        {"BHatZ", "Z-component of the magnetic field unit vector", dg::geo::BHatZ(mag)},
        {"BHatP", "Contravariant Phi-component of the magnetic field unit vector", dg::geo::BHatP(mag)},
        {"GradBHatR", "Parallel derivative of BHatR", dg::geo::BHatR(mag)},
        {"GradBHatZ", "Parallel derivative of BHatZ", dg::geo::BHatZ(mag)},
        {"GradBHatP", "Parallel derivative of BHatP", dg::geo::BHatP(mag)},
        {"NormGradPsip", "Norm of gradient of Psip", dg::geo::SquareNorm( dg::geo::createGradPsip(mag), dg::geo::createGradPsip(mag))},
        {"CurvatureNablaBGradPsip", "(Toroidal) Nabla B curvature dot the gradient of Psip", dg::geo::ScalarProduct( dg::geo::createCurvatureNablaB(mag, +1), dg::geo::createGradPsip(mag))},
        {"CurvatureKappaGradPsip", "(Toroidal) Kappa curvature dot the gradient of Psip", dg::geo::ScalarProduct( dg::geo::createCurvatureKappa(mag, +1), dg::geo::createGradPsip(mag))},
        {"TrueCurvatureNablaBGradPsip", "True Nabla B curvature dot the gradient of Psip", dg::geo::ScalarProduct( dg::geo::createTrueCurvatureNablaB(mag), dg::geo::createGradPsip(mag))},
        {"TrueCurvatureKappaGradPsip", "True Kappa curvature dot the gradient of Psip", dg::geo::ScalarProduct( dg::geo::createTrueCurvatureKappa(mag), dg::geo::createGradPsip(mag))},
        //////////////////////////////////
        {"Iris", "A flux aligned Iris", dg::compose( dg::Iris( 0.5, 0.7), dg::geo::RhoP(mag))},
        {"Pupil", "A flux aligned Pupil", dg::compose( dg::Pupil(0.7), dg::geo::RhoP(mag)) },
        {"GaussianDamping", "A flux aligned Heaviside with Gaussian damping", dg::compose( dg::GaussianDamping( 0.8, p.source_alpha), dg::geo::RhoP(mag)) },
        {"ZonalFlow",  "Flux aligned Sine function", dg::compose( dg::SinX ( p.amp, 0., 2.*M_PI*p.k_psi ), mag.psip())},
        {"PsiLimiter", "A flux aligned Heaviside", dg::compose( dg::Heaviside( 1.03), dg::geo::RhoP(mag) )},
        {"SourceProfile", "A source profile", dg::compose( dg::PolynomialHeaviside(
                    p.source_boundary-p.source_alpha/2., p.source_alpha/2., -1 ),
                dg::geo::RhoP(mag))},
        {"ProfileDamping", "Density profile damping", dg::compose(dg::PolynomialHeaviside(
            1.-p.profile_alpha/2., p.profile_alpha/2., -1), dg::geo::RhoP(mag)) },
        {"MagneticTransition", "The region where the magnetic field is modified", dg::compose(dg::DPolynomialHeaviside(
            p.damping_boundary+p.damping_alpha/2.,
            p.damping_alpha/2., +1 ), dg::geo::RhoP(mag_origin))},
        {"Nprofile", "A flux aligned profile", dg::compose( dg::LinearX( p.nprofileamp/mag.psip()(mag.R0(),0.), p.nprofileamp ), mag.psip())},
        {"Delta", "A flux aligned Gaussian peak", dg::compose( dg::GaussianX( psipO*0.2, 0.1, 1./(sqrt(2.*M_PI)*0.1)), mag.psip())},
        {"TanhDamping", "A flux aligned Heaviside with Tanh Damping", dg::compose( dg::TanhProfX( -3*p.source_alpha, p.source_alpha, -1), mag.psip())},
        ////
        {"BathRZ", "A randomized field", dg::BathRZ( 16, 16, Rmin,Zmin, 30.,2, p.amp)},
        {"Gaussian3d", "A Gaussian field", dg::Gaussian3d(gp.R_0+p.posX*gp.a, p.posY*gp.a,
            M_PI, p.sigma, p.sigma, p.sigma, p.amp)},
        { "Hoo", "The novel h02 factor", dg::geo::Hoo( mag) }
    };

    /// -------  Elements for fsa on X-point grid ----------------
    double psipmax = dg::blas1::reduce( psipog2d, 0., thrust::maximum<double>()); //DEPENDS ON GRID RESOLUTION!!
    std::unique_ptr<dg::geo::CurvilinearGridX2d> gX2d;
    if( gp.hasXpoint())
    {
        std::cout << "Generate X-point flux-aligned grid ... \n";
        double RX = gp.R_0-1.1*gp.triangularity*gp.a;
        double ZX = -1.1*gp.elongation*gp.a;
        dg::geo::findXpoint( mag.get_psip(), RX, ZX);
        double psipX = mag.psip()(RX, ZX);
        std::cout << "Found X-point at "<<RX<<" "<<ZX<<" with Psip = "<<psipX<<std::endl;
        if( fabs(psipX ) > 1e-10)
        {
            std::cerr << " Psip at X-point is not zero. Unable to construct grid\n";
            return -1;
        }
        dg::geo::CylindricalSymmTensorLvl1 monitor_chi = dg::geo::make_Xconst_monitor( mag.get_psip(), RX, ZX) ;
        dg::geo::SeparatrixOrthogonal generator(mag.get_psip(), monitor_chi, psipO, RX, ZX, mag.R0(), 0, 0, false);
        double fx_0 = 1./8.;
        psipmax = -fx_0/(1.-fx_0)*psipO;
        //std::cout << "psi 1 is          "<<psipmax<<"\n";
        gX2d = std::make_unique<dg::geo::CurvilinearGridX2d>(generator, fx_0, 0., npsi, Npsi, 640, dg::DIR, dg::NEU);
        std::cout << "DONE! \n";
        dg::Average<dg::HVec > avg_eta( gX2d->grid(), dg::coo2d::y);
        std::vector<dg::HVec> coordsX = gX2d->map();
        dg::SparseTensor<dg::HVec> metricX = gX2d->metric();
        dg::HVec volX2d = dg::tensor::volume2d( metricX);
        dg::blas1::pointwiseDot( coordsX[0], volX2d, volX2d); //R\sqrt{g}
        const double f0 = (gX2d->x1()-gX2d->x0())/ ( psipmax - psipO);
        dg::HVec dvdpsip;
        avg_eta( volX2d, dvdpsip, false);
        dg::blas1::scal( dvdpsip, 4.*M_PI*M_PI*f0);
        dg::Grid1d gX1d(psipO<psipmax ? psipO : psipmax,
            psipO<psipmax ? psipmax : psipO, npsi ,Npsi,dg::DIR_NEU); //inner value is always zero
        dg::HVec X_psi_vol = dg::integrate( dvdpsip, gX1d);
        map1d.emplace_back( "dvdpsip", dvdpsip,
            "Derivative of flux volume with respect to flux label psi");
        map1d.emplace_back( "psi_vol", X_psi_vol,
            "Flux volume on X-point grid");

        //NOTE: VOLUME is WITHIN cells while AREA is ON gridpoints
        dg::HVec gradZetaX = metricX.value(0,0), X_psi_area;
        dg::blas1::transform( gradZetaX, gradZetaX, dg::SQRT<double>());
        dg::blas1::pointwiseDot( volX2d, gradZetaX, gradZetaX); //R\sqrt{g}|\nabla\zeta|
        avg_eta( gradZetaX, X_psi_area, false);
        dg::blas1::scal( X_psi_area, 4.*M_PI*M_PI);
        map1d.emplace_back( "psi_area", X_psi_area,
            "Flux area on X-point grid");
        std::cout << "Total volume within separatrix is "<< dg::interpolate( dg::xspace, X_psi_vol, 0., gX1d)<<std::endl;

        //Compute FSA of cylindrical functions
        dg::HVec transferH, transferH1d;
        for( auto tp : map)
        {
            transferH = dg::pullback( std::get<2>(tp), *gX2d);
            dg::blas1::pointwiseDot( volX2d, transferH, transferH);
            avg_eta( transferH, transferH1d, false);
            dg::blas1::scal( transferH1d, 4*M_PI*M_PI*f0); //
            dg::blas1::pointwiseDivide( transferH1d, dvdpsip, transferH1d );
            map1d.emplace_back( std::get<0>(tp)+"_fsa", transferH1d,
                std::get<1>(tp)+" (Flux surface average)");
            dg::blas1::pointwiseDot( transferH1d, dvdpsip, transferH1d );
            transferH1d = dg::integrate( transferH1d, gX1d);
            map1d.emplace_back( std::get<0>(tp)+"_ifs", transferH1d,
                std::get<1>(tp)+" (Flux surface integral)");

        }
    }
    /// --------- More flux labels --------------------------------
    dg::Grid1d grid1d(psipO<psipmax ? psipO : psipmax,
            psipO<psipmax ? psipmax : psipO, npsi ,Npsi,dg::DIR_NEU); //inner value is always zero
    if( !gp.isToroidal())
    {
        dg::HVec rho = dg::evaluate( dg::cooX1d, grid1d);
        dg::blas1::axpby( -1./psipO, rho, +1., 1., rho); //transform psi to rho
        map1d.emplace_back("rho", rho,
            "Alternative flux label rho = -psi/psimin + 1");
        dg::blas1::transform( rho, rho, dg::SQRT<double>());
        map1d.emplace_back("rho_p", rho,
            "Alternative flux label rho_p = Sqrt[-psi/psimin + 1]");
        //if( gp.equilibrium == "solovev")
        {
            dg::geo::SafetyFactor qprof( mag);
            dg::HVec qprofile = dg::evaluate( qprof, grid1d);
            map1d.emplace_back("q-profile", qprofile,
                "q-profile (Safety factor) using direct integration");
            dg::HVec psit = dg::integrate( qprofile, grid1d);
            map1d.emplace_back("psit1d", psit,
                "Toroidal flux label psi_t integrated  on grid1d using direct q");
            //we need to avoid integrating >=0
            dg::Grid1d g1d_fine(psipO<0. ? psipO : 0.,
                    psipO<0. ? 0. : psipO, npsi, Npsi,dg::NEU);
            qprofile = dg::evaluate( qprof, g1d_fine);
            dg::HVec w1d = dg::create::weights( g1d_fine);
            double psit_tot = dg::blas1::dot( w1d, qprofile);
            //std::cout << "psit tot "<<psit_tot<<"\n";
            dg::blas1::scal ( psit, 1./psit_tot);
            dg::blas1::transform( psit, psit, dg::SQRT<double>());
            map1d.emplace_back("rho_t", psit,
                "Toroidal flux label rho_t = sqrt( psit/psit_tot) evaluated on grid1d");
        }
    }

    /////////////////////////////set up netcdf/////////////////////////////////////
    std::cout << "CREATING/OPENING FILE AND WRITING ... \n";
    file::NC_Error_Handle err;
    int ncid;
    err = nc_create( newfilename.data(), NC_NETCDF4|NC_CLOBBER, &ncid);
    /// Set global attributes
    std::map<std::string, std::string> att;
    att["title"] = "Output file of feltor/inc/geometries/geometry_diag.cu";
    att["Conventions"] = "CF-1.7";
    ///Get local time and begin file history
    auto t = std::time(nullptr);
    auto tm = *std::localtime(&t);

    std::ostringstream oss;
    ///time string  + program-name + args
    oss << std::put_time(&tm, "%Y-%m-%d %H:%M:%S");
    for( int i=0; i<argc; i++) oss << " "<<argv[i];
    att["history"] = oss.str();
    att["comment"] = "Find more info in feltor/src/feltor.tex";
    att["source"] = "FELTOR";
    att["references"] = "https://github.com/feltor-dev/feltor";
    att["inputfile"] = input;
    att["geomfile"] = geom;
    for( auto pair : att)
        err = nc_put_att_text( ncid, NC_GLOBAL,
            pair.first.data(), pair.second.size(), pair.second.data());

    int dim1d_ids[1], dim2d_ids[2], dim3d_ids[3] ;
    if( gp.hasXpoint())
    {
        int dim_idsX[2] = {0,0};
        err = file::define_dimensions( ncid, dim_idsX, gX2d->grid(), {"eta", "zeta"} );
        std::string long_name = "Flux surface label";
        err = nc_put_att_text( ncid, dim_idsX[0], "long_name",
            long_name.size(), long_name.data());
        long_name = "Flux angle";
        err = nc_put_att_text( ncid, dim_idsX[1], "long_name",
            long_name.size(), long_name.data());
        int xccID, yccID;
        err = nc_def_var( ncid, "xcc", NC_DOUBLE, 2, dim_idsX, &xccID);
        err = nc_def_var( ncid, "ycc", NC_DOUBLE, 2, dim_idsX, &yccID);
        long_name="Cartesian x-coordinate";
        err = nc_put_att_text( ncid, xccID, "long_name",
            long_name.size(), long_name.data());
        long_name="Cartesian y-coordinate";
        err = nc_put_att_text( ncid, yccID, "long_name",
            long_name.size(), long_name.data());
        err = nc_enddef( ncid);
        err = nc_put_var_double( ncid, xccID, gX2d->map()[0].data());
        err = nc_put_var_double( ncid, yccID, gX2d->map()[1].data());
        err = nc_redef(ncid);
        dim1d_ids[0] = dim_idsX[1];
    }
    else
    {
        err = file::define_dimension( ncid, &dim1d_ids[0], grid1d, "zeta");
        std::string psi_long_name = "Flux surface label";
        err = nc_put_att_text( ncid, dim1d_ids[0], "long_name",
            psi_long_name.size(), psi_long_name.data());
    }
    dg::CylindricalGrid3d grid3d(Rmin,Rmax,Zmin,Zmax, 0, 2.*M_PI, n,Nx,Ny,Nz);
    dg::RealCylindricalGrid3d<float> fgrid3d(Rmin,Rmax,Zmin,Zmax, 0, 2.*M_PI, n,Nx,Ny,Nz);

    err = file::define_dimensions( ncid, &dim3d_ids[0], fgrid3d);
    dim2d_ids[0] = dim3d_ids[1], dim2d_ids[1] = dim3d_ids[2];

    //write 1d vectors
    std::cout << "WRTING 1D FIELDS ... \n";
    for( auto tp : map1d)
    {
        int vid;
        err = nc_def_var( ncid, std::get<0>(tp).data(), NC_DOUBLE, 1,
            &dim1d_ids[0], &vid);
        err = nc_put_att_text( ncid, vid, "long_name",
            std::get<2>(tp).size(), std::get<2>(tp).data());
        err = nc_enddef( ncid);
        err = nc_put_var_double( ncid, vid, std::get<1>(tp).data());
        err = nc_redef(ncid);
    }
    //write 2d vectors
    //allocate mem for visual
    dg::HVec hvisual = dg::evaluate( dg::zero, grid2d);
    dg::HVec hvisual3d = dg::evaluate( dg::zero, grid3d);
    dg::fHVec fvisual, fvisual3d;
    std::cout << "WRTING 2D/3D CYLINDRICAL FIELDS ... \n";
    for(auto tp : map)
    {
        int vectorID, vectorID3d;
        err = nc_def_var( ncid, std::get<0>(tp).data(), NC_FLOAT, 2,
            &dim2d_ids[0], &vectorID);
        err = nc_def_var( ncid, (std::get<0>(tp)+"3d").data(), NC_FLOAT, 3,
            &dim3d_ids[0], &vectorID3d);
        err = nc_put_att_text( ncid, vectorID, "long_name",
            std::get<1>(tp).size(), std::get<1>(tp).data());
        err = nc_put_att_text( ncid, vectorID3d, "long_name",
            std::get<1>(tp).size(), std::get<1>(tp).data());
        std::string coordinates = "zc yc xc";
        err = nc_put_att_text( ncid, vectorID3d, "coordinates", coordinates.size(), coordinates.data());
        err = nc_enddef( ncid);
        hvisual = dg::evaluate( std::get<2>(tp), grid2d);
        dg::extend_line( grid2d.size(), grid3d.Nz(), hvisual, hvisual3d);
        dg::assign( hvisual, fvisual);
        dg::assign( hvisual3d, fvisual3d);
        err = nc_put_var_float( ncid, vectorID, fvisual.data());
        err = nc_put_var_float( ncid, vectorID3d, fvisual3d.data());
        err = nc_redef(ncid);
    }
    std::cout << "WRTING 3D FIELDS ... \n";
    //compute & write 3d vectors
    std::vector< std::tuple<std::string, std::string, std::function< double(double,double,double)> > > map3d{
        {"BR", "R-component of the magnetic field vector (3d version of BFieldR)",
            dg::geo::BFieldR(mag)},
        {"BZ", "Z-component of the magnetic field vector (3d version of BFieldZ)",
            dg::geo::BFieldZ(mag)},
        {"BP", "Contravariant Phi-component of the magnetic field vector (3d version of BFieldP)",
            dg::geo::BFieldP(mag)},
        {"xc", "x-coordinate in Cartesian coordinate system", dg::cooRZP2X},
        {"yc", "y-coordinate in Cartesian coordinate system", dg::cooRZP2Y},
        {"zc", "z-coordinate in Cartesian coordinate system", dg::cooRZP2Z}
    };
    for( auto tp : map3d)
    {
        int vectorID;
        err = nc_def_var( ncid, std::get<0>(tp).data(), NC_FLOAT, 3,
            &dim3d_ids[0], &vectorID);
        err = nc_put_att_text( ncid, vectorID, "long_name",
            std::get<1>(tp).size(), std::get<1>(tp).data());
        if( std::get<1>(tp) != "xc" && std::get<1>(tp) != "yc" &&std::get<1>(tp) != "zc")
        {
            std::string coordinates = "zc yc xc";
            err = nc_put_att_text( ncid, vectorID, "coordinates", coordinates.size(), coordinates.data());
        }
        err = nc_enddef( ncid);
        hvisual3d = dg::evaluate( std::get<2>(tp), grid3d);
        dg::assign( hvisual3d, fvisual3d);
        err = nc_put_var_float( ncid, vectorID, fvisual3d.data());
        err = nc_redef(ncid);
    }
    //////////////////////////////Finalize////////////////////////////////////
    err = nc_close(ncid);
    return 0;
}
