#include "hip/hip_runtime.h"
#include <iostream>
#include <functional>

#undef DG_DEBUG
#include "multistep.h"
#include "runge_kutta.h"
#include "elliptic.h"

//![function]
//method of manufactured solution
struct Solution{
    Solution(double t, double nu):t(t), nu(nu){}
DG_DEVICE
    double operator()(double x, double y) const{
        return sin(t)*exp( -2.*nu*t)*sin(x)*sin(y);
    }
    private:
    double t, nu;
};

struct Source{
    Source(double t, double nu):t(t), nu(nu){}
DG_DEVICE
    double operator()(double x, double y) const{
        return sin(x)*sin(y)*cos(t)*exp(-2*t*nu)*(1-sin(t));
    }
    private:
    double t, nu;
};

//the explicit part contains the source Tp = S(x,y,t)
template<class container>
struct Explicit
{
    Explicit( const dg::Grid2d& g, double nu):
        m_nu( nu),
        m_x ( dg::evaluate(dg::cooX2d, g)),//x-coordinate
        m_y ( dg::evaluate(dg::cooY2d, g)) //y-coordinate
    {}
    void operator()( double t, const container& T, container& Tp) {
        dg::blas1::evaluate( Tp, dg::equals(), Source(t,m_nu), m_x, m_y);
    }
    private:
    const double m_nu;
    const container m_x, m_y;

};

//the implicit part contains  Tp = nu Delta T(x,y,t) + cos(t) T(x,y,t)
template< class Matrix, class container>
struct Implicit
{
    Implicit( const dg::Grid2d& g, double nu):
        m_nu(nu),
        m_w2d( dg::create::weights(g)),
        m_v2d( dg::create::inv_weights(g)),
        m_LaplacianM( g, dg::normed)
        { }

    void operator()( double t, const container& T, container& Tp)
    {
        dg::blas2::gemv( m_LaplacianM, T, Tp);
        dg::blas1::axpby( cos(t), T, -m_nu, Tp);
    }
    //required by inversion in semi-implicit schemes
    const container& inv_weights(){return m_v2d;}
    const container& weights(){return m_w2d;}
    const container& precond(){return m_v2d;}
  private:
    double m_nu;
    const container m_w2d, m_v2d, m_x, m_y;
    dg::Elliptic<dg::CartesianGrid2d, Matrix, container> m_LaplacianM;
};

//![function]

template< class Matrix, class container>
struct Full
{
    Full( const dg::Grid2d& g, double nu):
        m_exp( g, nu), m_imp( g, nu), m_temp( dg::evaluate( dg::one, g))

    { }
    void operator()( double t, const container& y, container& yp) {
        m_exp( t, y, yp);
        m_imp( t, y, m_temp);
        dg::blas1::axpby( 1., m_temp, 1., yp);
    }
  private:
    Explicit<container> m_exp;
    Implicit<Matrix, container> m_imp;
    container m_temp;
};


const double lx = 2.*M_PI;
const double ly = 2.*M_PI;

//const unsigned NT = (unsigned)(nu*T*n*n*N*N/0.01/lx/lx);

int main()
{
    unsigned n = 3, Nx = 50 , Ny = 50;
    std::cout << "Program tests Multistep and Semi-Implicit methods on a manufactured PDE\n";
    const double T = 0.1;
    const double NT= 40, eps = 1e-8;
    const double dt = (T/NT);
    const double nu = 0.01;
    //construct the grid and the explicit and implicit parts
    dg::Grid2d grid( 0, lx, 0, ly, n, Nx, Ny, dg::PER, dg::PER);
    Explicit<dg::DVec> exp( grid, nu);
    Implicit<dg::DMatrix, dg::DVec> imp( grid, nu);

    Full<dg::DMatrix, dg::DVec> full( grid, nu);
    //evaluate the initial condition
    const dg::DVec init( dg::evaluate(Solution(0.,nu), grid));
    dg::DVec y0(init);

    const dg::DVec sol = dg::evaluate( Solution(T,nu), grid);
    const dg::DVec w2d = dg::create::weights( grid);
    const double norm_sol = dg::blas2::dot( w2d, sol);
    double time = 0.;
    dg::DVec error( sol);

    dg::AB< 1, dg::DVec > ab1( y0);
    dg::AB< 2, dg::DVec > ab2( y0);
    dg::AB< 3, dg::DVec > ab3( y0);
    dg::AB< 4, dg::DVec > ab4( y0);
    dg::AB< 5, dg::DVec > ab5( y0);
    ab1.init( full, time, y0, dt);
    ab2.init( full, time, y0, dt);
    ab3.init( full, time, y0, dt);
    ab4.init( full, time, y0, dt);
    ab5.init( full, time, y0, dt);

    exblas::udouble res;

    //main time loop
    time = 0., y0 =  init;
    for( unsigned i=0; i<NT; i++)
        ab1.step( full, time, y0);
    dg::blas1::axpby( -1., sol, 1., y0);
    res.d = sqrt(dg::blas2::dot( w2d, y0)/norm_sol);
    std::cout << "Relative error AB 1        is "<< res.d<<"\t"<<res.i<<std::endl;
    //main time loop
    time = 0., y0 =  init;
    for( unsigned i=0; i<NT; i++)
        ab2.step( full, time, y0);
    dg::blas1::axpby( -1., sol, 1., y0);
    res.d = sqrt(dg::blas2::dot( w2d, y0)/norm_sol);
    std::cout << "Relative error AB 2        is "<< res.d<<"\t"<<res.i<<std::endl;
    //main time loop
    time = 0., y0 =  init;
    for( unsigned i=0; i<NT; i++)
        ab3.step( full, time, y0);
    dg::blas1::axpby( -1., sol, 1., y0);
    res.d = sqrt(dg::blas2::dot( w2d, y0)/norm_sol);
    std::cout << "Relative error AB 3        is "<< res.d<<"\t"<<res.i<<std::endl;
    //main time loop
    time = 0., y0 =  init;
    for( unsigned i=0; i<NT; i++)
        ab4.step( full, time, y0);
    dg::blas1::axpby( -1., sol, 1., y0);
    res.d = sqrt(dg::blas2::dot( w2d, y0)/norm_sol);
    std::cout << "Relative error AB 4        is "<< res.d<<"\t"<<res.i<<std::endl;
    //main time loop
    time = 0., y0 =  init;
    for( unsigned i=0; i<NT; i++)
        ab5.step( full, time, y0);
    dg::blas1::axpby( -1., sol, 1., y0);
    res.d = sqrt(dg::blas2::dot( w2d, y0)/norm_sol);
    std::cout << "Relative error AB 5        is "<< res.d<<"\t"<<res.i<<std::endl;
    //![sirk]
    //construct time stepper (eps = 1e-8)
    dg::SIRK< dg::DVec > sirk( y0, y0.size(), eps);
    time = 0., y0 = init; //y0 and init are of type dg::DVec and contain the initial condition
    //main time loop (NT = 20, exp and imp are objects of type Explicit and Implicit defined above)
    for( unsigned i=0; i<NT; i++)
        sirk.step( exp, imp, time, y0, time, y0, dt); //inplace step
    //![sirk]
    dg::blas1::axpby( -1., sol, 1., y0);
    res.d = sqrt(dg::blas2::dot( w2d, y0)/norm_sol);
    std::cout << "Relative error SIRK        is "<< res.d<<"\t"<<res.i<<std::endl;
    //![karniadakis]
    //construct time stepper
    dg::Karniadakis< dg::DVec > karniadakis( y0, y0.size(), eps);
    time = 0., y0 = init; //y0 and init are of type dg::DVec and contain the initial condition
    //initialize the timestepper (exp and imp are objects of type Explicit and Implicit defined above)
    karniadakis.init( exp, imp, time, y0, dt);
    //main time loop (NT = 20)
    for( unsigned i=0; i<NT; i++)
        karniadakis.step( exp, imp, time, y0); //inplace step
    //![karniadakis]
    dg::blas1::axpby( -1., sol, 1., y0);
    res.d = sqrt(dg::blas2::dot( w2d, y0)/norm_sol);
    std::cout << "Relative error Karniadakis is "<< res.d<<"\t"<<res.i<<std::endl;
    //main time loop
    std::cout << "\nAdaptive SIRK Timer \n";
    time = 0., y0 =  init;
    double adapt = dt;
    while( time < T-adapt){
        sirk.adaptive_step( exp, imp, time, y0, time, y0, adapt, 1e-8, true);
    }
    adapt = T - time;
    sirk.adaptive_step( exp, imp, time, y0, time, y0, adapt, 1e-8, true);
    dg::blas1::axpby( -1., sol, 1., y0);
    res.d = sqrt(dg::blas2::dot( w2d, y0)/norm_sol);
    std::cout << "Relative error adaptive sirk: "<< res.d<<"\t"<<res.i<<std::endl;
    return 0;
}
