#include "hip/hip_runtime.h"
// #undef DG_BENCHMARK

#include <iostream>
#include <iomanip>
#include "backend/timer.h"

#include "lanczos.h"
#include "helmholtz.h"


const double lx = 2.*M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;
const double alpha = -0.5;


double lhs( double x, double y) {return sin(x)*sin(y);}
double rhs( double x, double y){ return (1.-2.*alpha)*sin(x)*sin(y);}
double lhs2( double x, double y) {return sin(x)*sin(4.*y);}
double rhs2( double x, double y){ return (1.-17.*alpha)*sin(x)*sin(4.*y);}

using dia_type = cusp::dia_matrix<int, double, cusp::device_memory>;
using coo_type = cusp::coo_matrix<int, double, cusp::device_memory>;
using Mat_type = dg::DMatrix;
using Container_type = dg::DVec;
int main()
{
    dg::Timer t;
    unsigned n, Nx, Ny;
    std::cout << "Type n, Nx and Ny\n";
    std::cin >> n >> Nx >> Ny;
    unsigned max_iter;
    std::cout << "# of max_iterations\n"; 
    std::cin >> max_iter;
    std::cout << "Type in eps\n";
    double eps = 1e-6; 
    std::cin >> eps;
    dg::CartesianGrid2d grid( 0., lx, 0, ly, n, Nx, Ny, bcx, bcy);
    
    const Container_type w2d = dg::create::weights( grid);
    const Container_type v2d = dg::create::inv_weights( grid);
        
    Container_type x = dg::evaluate( lhs, grid), b(x), zero(x), one(x),bsymv(x), error(x), bsymv2(x), helper(x);
    Container_type x2 = dg::evaluate( lhs2, grid);
    Container_type bexac = dg::evaluate( rhs, grid);
    Container_type bexac2 = dg::evaluate( rhs2, grid);
    dg::blas1::scal(zero, 0.0);
    one = dg::evaluate(dg::one, grid);
    dg::Helmholtz<dg::CartesianGrid2d, Mat_type, Container_type> A( grid, alpha, dg::centered); //not_normed
    
    //Create Lanczos class
    t.tic();
    dg::Lanczos< Container_type > lanczos(x, max_iter);
    t.toc();
    std::cout << "Creation of Lanczos  took "<< t.diff()<<"s   \n";

    dia_type T; 
    coo_type V, Vt;
    std::pair<dia_type, coo_type> TVpair; 
    
    std::cout << "Computing with Lanczos method \n";
    t.tic();
    TVpair = lanczos(A, x, b); 
    dg::blas2::symv( v2d, b, b);     //normalize
    t.toc();
    T = TVpair.first; 
    V = TVpair.second;
    cusp::transpose(V, Vt);
    
    //Compute error with method 1
    dg::blas2::symv(A, x, helper);
    dg::blas2::symv( v2d, helper, bsymv); //normalize operator
    dg::blas1::axpby(-1.0, bsymv, 1.0, b,error);
    std::cout << "# of Lanczos Iterations: "<< lanczos.get_iter() <<" | time: "<< t.diff()<<"s \n";
    std::cout << "# Relative error between b=||x||_S S^{-1}V^T T e_1 and to b=S^{-1} A x: " << sqrt(dg::blas2::dot(w2d, error)/dg::blas2::dot(w2d, bsymv)) << " \n";
    dg::blas1::axpby(-1.0, bexac, 1.0, b,error);
    std::cout << "# Relative error between b=||x||_S S^{-1}V^T T e_1 and b: " << sqrt(dg::blas2::dot(w2d, error)/dg::blas2::dot(w2d, bexac)) << " \n";   
    //Compute error with method 2
    Container_type e1( lanczos.get_iter(), 0.), temp(e1);
    e1[0]=1.;  
//     dg::blas2::symv( w2d, x,x); //normalize
    dg::blas2::symv(Vt, x, e1); //V^T x
    dg::blas2::symv(T, e1, temp); //T V^T x
    dg::blas2::symv(V, temp, x); // V T V^T x
    dg::blas2::symv( v2d, x, b);     //normalize
    dg::blas1::axpby(-1.0, bsymv, 1.0, b,error);
    std::cout << "# Relative error between b=S^{-1}V T V^T x  and to b=S^{-1} A x:" << sqrt(dg::blas2::dot(w2d, error)/dg::blas2::dot(w2d, bsymv)) << "\n";    
    dg::blas1::axpby(-1.0, bexac, 1.0, b,error);
    std::cout << "# Relative error between b=S^{-1}V T V^T  x  and b: " << sqrt(dg::blas2::dot(w2d, error)/dg::blas2::dot(w2d, bexac)) << " \n";     


    std::cout << "\nComputing with M-Lanczos method \n";
    x = dg::evaluate( lhs, grid);
    dg::blas2::symv(A, x, helper);
    dg::blas2::symv( v2d, helper, bsymv); //normalize operator
    bexac= dg::evaluate( rhs, grid);

    t.tic();
    TVpair = lanczos(A, x, b, w2d, v2d, eps); 
    t.toc();
    T = TVpair.first; 
    V = TVpair.second;
    cusp::transpose(V, Vt);
    //Compute error with Method 1
    dg::blas1::axpby(-1.0, bsymv, 1.0, b,error);
    std::cout << "# of Lanczos Iterations: "<< lanczos.get_iter() <<" | time: "<< t.diff()<<"s \n";
    std::cout << "# Relative error between b=||x||_S V^T T e_1 and to b=S^{-1} A x: " << sqrt(dg::blas2::dot(w2d, error)/dg::blas2::dot(w2d, bsymv)) << " \n";
    dg::blas1::axpby(-1.0, bexac, 1.0, b,error);
    std::cout << "# Relative error between b=||x||_S V^T T e_1 and b: " << sqrt(dg::blas2::dot(w2d, error)/dg::blas2::dot(w2d, bexac)) << " \n";
    //Compute error with method 2
    e1.resize( lanczos.get_iter(), 0.), temp.resize( lanczos.get_iter(), 0.);
    e1[0]=1.;
    dg::blas2::symv( w2d, x,helper); //normalize
    dg::blas2::symv(Vt, helper, e1); //V^T x
    dg::blas2::symv(T, e1, temp); //T V^T x
    dg::blas2::symv(V, temp, b); // V T V^T x
    dg::blas1::axpby(-1.0, bsymv, 1.0, b,error);
    std::cout << "# Relative error between b=V T V^T S x  and to b=S^{-1} A x:" << sqrt(dg::blas2::dot(w2d, error)/dg::blas2::dot(w2d, bsymv)) << "\n";    
    dg::blas1::axpby(-1.0, bexac, 1.0, b,error);
    std::cout << "# Relative error between b=V T V^T S x  and b: " << sqrt(dg::blas2::dot(w2d, error)/dg::blas2::dot(w2d, bexac)) << " \n";  
    
    return 0;
}
