#include "hip/hip_runtime.h"
/*
 * %%%%%%%%%%%%%%%%%%%%%%%Original development%%%%%%%%%%%%%%%%%%%%%%%%%
 *  Copyright (c) 2016 Inria and University Pierre and Marie Curie 
 *  All rights reserved.
 * %%%%%%%%%%%%%%%%%%%%%%%Modifications and further additions%%%%%%%%%%
 *  Matthias Wiesenberger, 2017, within FELTOR and EXBLAS licenses
 */
#include "thrust/device_vector.h"

namespace exblas{

static constexpr uint BIN_COUNT     =  39; //size of superaccumulator
static constexpr uint NBFPE         =  3;  //size of floating point expansion
////////////// parameters for superaccumulator operations //////////////////////
static constexpr int KRX            =  8;  //High-radix carry-save bits
static constexpr int DIGITS         =  64 - KRX; //must be int because appears in integer expresssion
static constexpr int F_WORDS        =  20;
//static constexpr int TSAFE          =  0;
static constexpr double DELTASCALE = double(1ull << DIGITS); // Assumes KRX>0

////////////// parameters for Kernel execution            //////////////////////
//Kernel paramters for EXDOT
static constexpr uint WARP_COUNT               = 16 ; //# of sub superaccs
static constexpr uint WARP_SIZE                = 16 ; 
static constexpr uint WORKGROUP_SIZE           = (WARP_COUNT * WARP_SIZE); //# threads per block
static constexpr uint PARTIAL_SUPERACCS_COUNT  = 128; //# of groups; each has a partial SuperAcc (somehow does not work for 128???)
//Kernel paramters for EXDOTComplete
static constexpr uint MERGE_SUPERACCS_SIZE     = 128; //# of sa each block merges
static constexpr uint MERGE_WORKGROUP_SIZE     = 64;  //we need only 39 of those


////////////////////////////////////////////////////////////////////////////////
// Auxiliary functions
////////////////////////////////////////////////////////////////////////////////
__device__ 
double TwoProductFMA(double a, double b, double *d) {
    double p = a * b;
    *d = fma(a, b, -p);
    return p;
}

__device__ 
double KnuthTwoSum(double a, double b, double *s) {
    double r = a + b;
    double z = r - a;
    *s = (a - (r - z)) + (b - z);
    return r;
}

//returns the original value at address
__device__ long long int atomicAdd( long long int* address, long long int val)
{
    unsigned long long int* address_as_ull =
        (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do
    {
        assumed = old; //*address_as_ull might change during the time the CAS is reached
        old = atomicCAS(address_as_ull, assumed,
                          (unsigned long long int)(val + (long long int)old));
    } while( old != assumed);//try as often as necessary
    //assume that bit patterns don't change when casting
    //return the original value stored at address
    return (long long int)(old);
}
// signedcarry in {-1, 0, 1}
__device__ long long int xadd( long long int *sa, long long int x, unsigned char *of) {
    // OF and SF  -> carry=1
    // OF and !SF -> carry=-1
    // !OF        -> carry=0
    //long long int y = atom_add(sa, x);
    long long int y = atomicAdd(sa, x); 
    long long int z = y + x; // since the value sa->superacc[i] can be changed by another work item

    // TODO: cover also underflow
    *of = 0;
    if(x > 0 && y > 0 && z < 0)
        *of = 1;
    if(x < 0 && y < 0 && z > 0)
        *of = 1;

    return y;
}


////////////////////////////////////////////////////////////////////////////////
// Rounding functions
////////////////////////////////////////////////////////////////////////////////
__host__ __device__
double OddRoundSumNonnegative_gpu(double th, double tl) {
    union {
        double d;
        long long int l;
    } thdb;

    thdb.d = th + tl;
    // - if the mantissa of th is odd, there is nothing to do
    // - otherwise, round up as both tl and th are positive
    // in both cases, this means setting the msb to 1 when tl>0
    thdb.l |= (tl != 0.0);
    return thdb.d;
}

__device__
int Normalize( long long int *accumulator, int *imin, int *imax) {
    long long int carry_in = accumulator[*imin] >> DIGITS;
    accumulator[*imin] -= carry_in << DIGITS;
    int i;
    // Sign-extend all the way
    for (i = *imin + 1; i < BIN_COUNT; ++i) {
        accumulator[i] += carry_in;
        long long int carry_out = accumulator[i] >> DIGITS;    // Arithmetic shift
        accumulator[i] -= (carry_out << DIGITS);
        carry_in = carry_out;
    }
    *imax = i - 1;

    // Do not cancel the last carry to avoid losing information
    accumulator[*imax] += carry_in << DIGITS;

    return carry_in < 0;
}
__device__
int NormalizeT( long long int *accumulator, int *imin, int *imax) {
    long long int carry_in = accumulator[(*imin)*WARP_COUNT] >> DIGITS;
    accumulator[(*imin)*WARP_COUNT] -= carry_in << DIGITS;
    int i;
    // Sign-extend all the way
    for (i = *imin + 1; i < BIN_COUNT; ++i) {
        accumulator[i*WARP_COUNT] += carry_in;
        long long int carry_out = accumulator[i*WARP_COUNT] >> DIGITS;    // Arithmetic shift
        accumulator[i*WARP_COUNT] -= (carry_out << DIGITS);
        carry_in = carry_out;
    }
    *imax = i - 1;

    // Do not cancel the last carry to avoid losing information
    accumulator[(*imax)*WARP_COUNT] += carry_in << DIGITS;

    return carry_in < 0;
}

__device__
double Round( long long int *accumulator) {
    int imin = 0;
    int imax = 38;
    int negative = Normalize(accumulator, &imin, &imax);

    //Find leading word
    int i;
    //Skip zeroes
    for (i = imax; accumulator[i] == 0 && i >= imin; --i) {
    }
    if (negative) {
        //Skip ones
        for(; (accumulator[i] & ((1l << DIGITS) - 1)) == ((1l << DIGITS) - 1) && i >= imin; --i) {
        }
    }
    if (i < 0)
        return 0.0;

    long long int hiword = negative ? ((1l << DIGITS) - 1) - accumulator[i] : accumulator[i];
    double rounded = (double) hiword;
    double hi = ldexp(rounded, (i - F_WORDS) * DIGITS);
    if (i == 0)
        return negative ? -hi : hi;  // Correct rounding achieved
    hiword -= (long long int) rint(rounded);
    double mid = ldexp((double) hiword, (i - F_WORDS) * DIGITS);

    //Compute sticky
    long long int sticky = 0;
    for (int j = imin; j != i - 1; ++j)
        sticky |= negative ? (1l << DIGITS) - accumulator[j] : accumulator[j];

    long long int loword = negative ? (1l << DIGITS) - accumulator[i - 1] : accumulator[i - 1];
    loword |= !!sticky;
    double lo = ldexp((double) loword, (i - 1 - F_WORDS) * DIGITS);

    //Now add3(hi, mid, lo)
    //No overlap, we have already normalized
    if (mid != 0)
        lo = OddRoundSumNonnegative_gpu(mid, lo);

    //Final rounding
    hi = hi + lo;
    return negative ? -hi : hi;
}


////////////////////////////////////////////////////////////////////////////////
// Main computation pass: compute partial superaccs
////////////////////////////////////////////////////////////////////////////////
__device__
void AccumulateWord( long long int *sa, int i, long long int x) {
    // With atomic superacc updates
    // accumulation and carry propagation can happen in any order,
    // as long long int as addition is atomic
    // only constraint is: never forget an overflow bit
    unsigned char overflow;
    long long int carry = x;
    long long int carrybit;
    long long int oldword = xadd(&sa[i * WARP_COUNT], x, &overflow);

    // To propagate over- or underflow
    while (overflow) {
        // Carry or borrow
        // oldword has sign S
        // x has sign S
        // superacc[i] has sign !S (just after update)
        // carry has sign !S
        // carrybit has sign S
        carry = (oldword + carry) >> DIGITS;    // Arithmetic shift
        bool s = oldword > 0;
        carrybit = (s ? 1l << KRX : -1l << KRX);

        // Cancel carry-save bits
        xadd(&sa[i * WARP_COUNT], (long long int) -(carry << DIGITS), &overflow);
        //if (TSAFE && (s ^ overflow))
        if (0 && (s ^ overflow)) //MW: TSAFE is always 0
            carrybit *= 2;
        carry += carrybit;

        ++i;
        if (i >= BIN_COUNT)
            return;
        oldword = xadd(&sa[i * WARP_COUNT], carry, &overflow);
    }
}

__device__
void Accumulate( long long int *sa, double x) {
    if (x == 0)
        return;

    int e;
    frexp(x, &e); //extract the exponent of x (lies in -1024;1023 ?)
    int exp_word = e / DIGITS;  // Word containing MSbit
    int iup = exp_word + F_WORDS; //can be at most 18 + 20 

    double xscaled = ldexp(x, -DIGITS * exp_word);

    int i;
    for (i = iup; xscaled != 0; --i) {
        double xrounded = rint(xscaled);
        long long int xint = (long long int) xrounded;

        AccumulateWord(sa, i, xint);

        xscaled -= xrounded;
        xscaled *= DELTASCALE;
    }
}


__global__ void ExDOT(
    long long int *d_PartialSuperaccs,
    const double *d_a,
    const double *d_b,
    const uint NbElements
) {
    __shared__ long long int l_sa[WARP_COUNT * BIN_COUNT]; //shared variables live for a thread block (39 rows, 16 columns!)
    long long int *l_workingBase = l_sa + (threadIdx.x & (WARP_COUNT - 1)); //the bitwise & with 15 is a modulo operation: threadIdx.x % 16
    //Initialize superaccs
    for (uint i = 0; i < BIN_COUNT; i++)
        l_workingBase[i * WARP_COUNT] = 0;
    __syncthreads();

    //Read data from global memory and scatter it to sub-superaccs
    double a[NBFPE] = {0.0};
    for(uint pos = blockIdx.x*blockDim.x+threadIdx.x; pos < NbElements; pos += gridDim.x*blockDim.x) {
            double x = d_a[pos]*d_b[pos];
            #pragma unroll
            for(uint i = 0; i != NBFPE; ++i) {
                double s;
                a[i] = KnuthTwoSum(a[i], x, &s);
                x = s;
            }
            if (x != 0.0) {
                Accumulate(l_workingBase, x);
                // Flush FPEs to superaccs
                #pragma unroll
                for(uint i = 0; i != NBFPE; ++i) {
                    Accumulate(l_workingBase, a[i]);
                    a[i] = 0.0;
                }
            }
        //double r = 0.0;
        //double x = TwoProductFMA(d_a[pos], d_b[pos], &r);

        //#pragma unroll
        //for(uint i = 0; i != NBFPE; ++i) {
        //    double s;
        //    a[i] = KnuthTwoSum(a[i], x, &s);
        //    x = s;
        //}
        //if (x != 0.0) {
        //    Accumulate(l_workingBase, x);
        //    // Flush FPEs to superaccs
        //    #pragma unroll
        //    for(uint i = 0; i != NBFPE; ++i) {
        //        Accumulate(l_workingBase, a[i]);
        //        a[i] = 0.0;
        //    }
        //}

        //if (r != 0.0) {//add the rest r in the same manner
        //    #pragma unroll
        //    for(uint i = 0; i != NBFPE; ++i) {
        //        double s;
        //        a[i] = KnuthTwoSum(a[i], r, &s);
        //        r = s;
        //    }
        //    if (r != 0.0) {
        //        Accumulate(l_workingBase, r);
        //        // Flush FPEs to superaccs
        //        #pragma unroll
        //        for(uint i = 0; i != NBFPE; ++i) {
        //            Accumulate(l_workingBase, a[i]);
        //            a[i] = 0.0;
        //        }
        //    }
        //}
    }
	//Flush FPEs to superaccs
    #pragma unroll
    for(uint i = 0; i != NBFPE; ++i)
        Accumulate(l_workingBase, a[i]);
    __syncthreads();

    //Merge sub-superaccs into work-group partial-accumulator ( ATTENTION: PartialSuperacc is transposed!)
    uint pos = threadIdx.x;
//if(pos < WARP_COUNT) {
//        int imin = 0;
//        int imax = 38;
//    NormalizeT( l_workingBase, &imin, &imax);
//}
//    __syncthreads();

    if (pos < BIN_COUNT) {
        long long int sum = 0;

        for(uint i = 0; i < WARP_COUNT; i++)
            sum += l_sa[pos * WARP_COUNT + i];

        d_PartialSuperaccs[blockIdx.x * BIN_COUNT + pos] = sum;
    }

    __syncthreads();
    if (pos == 0) {
        int imin = 0;
        int imax = 38;
        Normalize(&d_PartialSuperaccs[blockIdx.x * BIN_COUNT], &imin, &imax);
    }
}

__global__ void ExDOT(
    long long int *d_PartialSuperaccs,
    const double *d_a,
    const double *d_b,
    const double *d_c,
    const uint NbElements
) {
    __shared__ long long int l_sa[WARP_COUNT * BIN_COUNT]; //shared variables live for a thread block (39 rows, 16 columns!)
    long long int *l_workingBase = l_sa + (threadIdx.x & (WARP_COUNT - 1)); //the bitwise & with 15 is a modulo operation: threadIdx.x % 16
    //Initialize superaccs
    for (uint i = 0; i < BIN_COUNT; i++)
        l_workingBase[i * WARP_COUNT] = 0;
    __syncthreads();

    //Read data from global memory and scatter it to sub-superaccs
    double a[NBFPE] = {0.0};
    for(uint pos = blockIdx.x*blockDim.x+threadIdx.x; pos < NbElements; pos += gridDim.x*blockDim.x) {
            double x = d_a[pos]*d_c[pos]*d_b[pos];
            #pragma unroll
            for(uint i = 0; i != NBFPE; ++i) {
                double s;
                a[i] = KnuthTwoSum(a[i], x, &s);
                x = s;
            }
            if (x != 0.0) {
                Accumulate(l_workingBase, x);
                // Flush FPEs to superaccs
                #pragma unroll
                for(uint i = 0; i != NBFPE; ++i) {
                    Accumulate(l_workingBase, a[i]);
                    a[i] = 0.0;
                }
            }
        //double r  = 0.0, r2 = 0.0;
        //double x  = TwoProductFMA(d_a[pos], d_b[pos], &r);
        //double x2 = TwoProductFMA(x , d_c[pos], &r2);


        //if( x2 != 0.0) {//accumulate x2
        //    #pragma unroll
        //    for(uint i = 0; i != NBFPE; ++i) {
        //        double s;
        //        a[i] = KnuthTwoSum(a[i], x2, &s);
        //        x2 = s;
        //    }
        //    if (x2 != 0.0) {
        //        Accumulate(l_workingBase, x2);
        //        // Flush FPEs to superaccs
        //        #pragma unroll
        //        for(uint i = 0; i != NBFPE; ++i) {
        //            Accumulate(l_workingBase, a[i]);
        //            a[i] = 0.0;
        //        }
        //    }
        //}
        //if (r2 != 0.0) {//add the rest r2 
        //    #pragma unroll
        //    for(uint i = 0; i != NBFPE; ++i) {
        //        double s;
        //        a[i] = KnuthTwoSum(a[i], r2, &s);
        //        r2 = s; //error was here r = s
        //    }
        //    if (r2 != 0.0) { //error was here r != 0.0
        //        Accumulate(l_workingBase, r2);
        //        // Flush FPEs to superaccs
        //        #pragma unroll
        //        for(uint i = 0; i != NBFPE; ++i) {
        //            Accumulate(l_workingBase, a[i]);
        //            a[i] = 0.0;
        //        }
        //    }
        //}

        //if (r != 0.0) {//add the rest r*c in the same manner
        //    x2 = TwoProductFMA(r , d_c[pos], &r2);
        //    if( x2 != 0.0) {//accumulate x2
        //        #pragma unroll
        //        for(uint i = 0; i != NBFPE; ++i) {
        //            double s;
        //            a[i] = KnuthTwoSum(a[i], x2, &s);
        //            x2 = s;
        //        }
        //        if (x2 != 0.0) {
        //            Accumulate(l_workingBase, x2);
        //            // Flush FPEs to superaccs
        //            #pragma unroll
        //            for(uint i = 0; i != NBFPE; ++i) {
        //                Accumulate(l_workingBase, a[i]);
        //                a[i] = 0.0;
        //            }
        //        }
        //    }
        //    if (r2 != 0.0) {//add the rest r2 
        //        #pragma unroll
        //        for(uint i = 0; i != NBFPE; ++i) {
        //            double s;
        //            a[i] = KnuthTwoSum(a[i], r2, &s);
        //            r2 = s; //error was here r = s
        //        }
        //        if (r2 != 0.0) { //error was here r != 0.0
        //            Accumulate(l_workingBase, r2);
        //            // Flush FPEs to superaccs
        //            #pragma unroll
        //            for(uint i = 0; i != NBFPE; ++i) {
        //                Accumulate(l_workingBase, a[i]);
        //                a[i] = 0.0;
        //            }
        //        }
        //    }
        //}
    }
	//Flush FPEs to superaccs
    #pragma unroll
    for(uint i = 0; i != NBFPE; ++i)
        Accumulate(l_workingBase, a[i]);
    __syncthreads();

    //Merge sub-superaccs into work-group partial-accumulator ( ATTENTION: PartialSuperacc is transposed!)
    uint pos = threadIdx.x;
    if (pos < BIN_COUNT) {
        long long int sum = 0;

        for(uint i = 0; i < WARP_COUNT; i++)
            sum += l_sa[pos * WARP_COUNT + i];

        d_PartialSuperaccs[blockIdx.x * BIN_COUNT + pos] = sum;
    }

    __syncthreads();
    if (pos == 0) {
        int imin = 0;
        int imax = 38;
        Normalize(&d_PartialSuperaccs[blockIdx.x * BIN_COUNT], &imin, &imax);
    }
}

////////////////////////////////////////////////////////////////////////////////
// Merging
////////////////////////////////////////////////////////////////////////////////
__global__
void ExDOTComplete(
     double *d_Res,
     long long int *d_PartialSuperaccs
) {
    uint lid = threadIdx.x;
    uint gid = blockIdx.x;

    if (lid < BIN_COUNT) {
        long long int sum = 0;

        for(uint i = 0; i < MERGE_SUPERACCS_SIZE; i++)
            sum += d_PartialSuperaccs[(gid * MERGE_SUPERACCS_SIZE + i) * BIN_COUNT + lid];

        d_PartialSuperaccs[gid * BIN_COUNT + lid] = sum;
    }

    __syncthreads();
    if (lid == 0) {
        int imin = 0;
        int imax = 38;
        Normalize(&d_PartialSuperaccs[gid * BIN_COUNT], &imin, &imax);
    }

    __syncthreads();
    if ((lid < BIN_COUNT) && (gid == 0)) {
        long long int sum = 0;

        for(uint i = 0; i < gridDim.x; i++)
            sum += d_PartialSuperaccs[i * BIN_COUNT + lid];

        d_PartialSuperaccs[lid] = sum;

        //__syncthreads();
        //if (lid == 0)
        //    d_Res[0] = Round(d_PartialSuperaccs);
    }
}

__host__
std::vector<int64_t> exdot_gpu(unsigned size, const double* x1_ptr, const double* x2_ptr)
{
    thrust::device_vector<long long int> d_PartialSuperaccsV( PARTIAL_SUPERACCS_COUNT*BIN_COUNT); //39 columns and PSC rows
    long long int *d_PartialSuperaccs = thrust::raw_pointer_cast( d_PartialSuperaccsV.data());
    ExDOT<<<PARTIAL_SUPERACCS_COUNT, WORKGROUP_SIZE>>>( d_PartialSuperaccs, x1_ptr, x2_ptr,size);
    thrust::device_vector<double> r(1,0);
    double *r_ptr = thrust::raw_pointer_cast( r.data());
    ExDOTComplete<<<PARTIAL_SUPERACCS_COUNT/MERGE_SUPERACCS_SIZE, MERGE_WORKGROUP_SIZE>>>( r_ptr, d_PartialSuperaccs );
    std::vector<int64_t> h_Superacc(BIN_COUNT);
    hipMemcpy( &h_Superacc[0], d_PartialSuperaccs, BIN_COUNT*sizeof(long long int), hipMemcpyDeviceToHost);
    return h_Superacc;
}
__host__
std::vector<int64_t> exdot_gpu(unsigned size, const double* x1_ptr, const double* x2_ptr, const double* x3_ptr)
{
    thrust::device_vector<long long int> d_PartialSuperaccsV( PARTIAL_SUPERACCS_COUNT*BIN_COUNT); //39 columns and PSC rows
    long long int *d_PartialSuperaccs = thrust::raw_pointer_cast( d_PartialSuperaccsV.data());
    ExDOT<<<PARTIAL_SUPERACCS_COUNT, WORKGROUP_SIZE>>>( d_PartialSuperaccs, x1_ptr, x2_ptr, x3_ptr,size);
    thrust::device_vector<double> r(1,0);
    double *r_ptr = thrust::raw_pointer_cast( r.data());
    ExDOTComplete<<<PARTIAL_SUPERACCS_COUNT/MERGE_SUPERACCS_SIZE, MERGE_WORKGROUP_SIZE>>>( r_ptr, d_PartialSuperaccs );

    std::vector<int64_t> h_Superacc(BIN_COUNT, 1);
    hipMemcpy( &h_Superacc[0], d_PartialSuperaccs, BIN_COUNT*sizeof(long long int), hipMemcpyDeviceToHost);
    return h_Superacc;
}
}//namespace exblas

