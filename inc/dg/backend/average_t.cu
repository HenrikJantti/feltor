#include "hip/hip_runtime.h"
#include <iostream>
#include "average.cuh"
#include "../blas2.h"
#include "typedefs.cuh"


const double lx = 2.*M_PI;
const double ly = M_PI;
double function( double x, double y) {return cos(x)*sin(y);}
double pol_average( double x, double y) {return cos(x)*2./M_PI;}

int main()
{
    unsigned n, Nx, Ny;
    std::cout << "Type n, Nx and Ny!\n";
    std::cin >> n >> Nx >> Ny;
    //![doxygen]
    const dg::Grid2d g( 0, lx, 0, ly, n, Nx, Ny);

    dg::PoloidalAverage<dg::HVec, dg::iHVec > pol(g);

    const dg::HVec vector = dg::evaluate( function ,g);
    dg::HVec average_y( vector);
    std::cout << "Averaging ... \n";
    pol( vector, average_y);
    //![doxygen]
    const dg::HVec w2d = dg::create::weights( g);
    const dg::HVec solution = dg::evaluate( pol_average, g);
    dg::blas1::axpby( 1., solution, -1., average_y);
    std::cout << "Distance to solution is: "<<sqrt(dg::blas2::dot( average_y, w2d, average_y))<<std::endl;



    return 0;
}
