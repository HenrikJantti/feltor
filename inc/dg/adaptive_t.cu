#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <functional>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "backend/typedefs.h"
#include "geometry/evaluation.h"
#include "arakawa.h"
#include "runge_kutta.h"
#include "adaptive.h"


//![function]
void rhs(double t, const std::array<double,2>& y, std::array<double,2>& yp, double damping, double omega_0, double omega_drive){
    //damped driven harmonic oscillator
    // x -> y[0] , v -> y[1]
    yp[0] = y[1];
    yp[1] = -2.*damping*omega_0*y[1] - omega_0*omega_0*y[0] + sin(omega_drive*t);
}
//![function]

std::array<double, 2> solution( double t, double damping, double omega_0, double omega_drive)
{
    double tmp1 = (2.*omega_0*damping);
    double tmp2 = (omega_0*omega_0 - omega_drive*omega_drive)/omega_drive;
    double amp = 1./sqrt( tmp1*tmp1 + tmp2*tmp2);
    double phi = atan( 2.*omega_drive*omega_0*damping/(omega_drive*omega_drive-omega_0*omega_0));

    double x = amp*sin(omega_drive*t+phi)/omega_drive;
    double v = amp*cos(omega_drive*t+phi);
    return {x,v};
}

int main()
{
    std::cout << "Program to test correct implementation of adaptive methods in adaptive.h at the example of the damped driven harmonic oscillator. Errors should be small! \n";
    std::cout << std::scientific;
    //![doxygen]
    //... in main
    //set start and end time
    const double t_start = 0., t_end = 1.;
    //set physical parameters and initial condition
    const double damping = 0.2, omega_0 = 1.0, omega_drive = 0.9;
    std::array<double,2> u_start = solution(t_start, damping, omega_0, omega_drive), u_end(u_start);
    //construct a functor with the right interface
    using namespace std::placeholders; //for _1, _2, _3
    auto functor = std::bind( rhs, _1, _2, _3, damping, omega_0, omega_drive);
    double dt= 0;
    //integration
    int counter = dg::integrateRK45( functor, t_start, u_start, t_end, u_end, dt, 1e-6);
    //now compute error
    dg::blas1::axpby( 1., solution(t_end, damping, omega_0, omega_drive), -1., u_end);
    std::cout << "With "<<counter<<"\t Embedded RK 4-5 steps norm of error is\t "<<sqrt(dg::blas1::dot( u_end, u_end))<<"\n";
    //![doxygen]
    dt = 0;
    counter = dg::integrateHRK<4>( functor, t_start, u_start, t_end, u_end, dt, 1e-6);
    dg::blas1::axpby( 1., solution(t_end, damping, omega_0, omega_drive), -1., u_end);
    std::cout << "With "<<counter<<"\t Halfstep RK 4 steps norm of error is\t "<<sqrt(dg::blas1::dot( u_end, u_end))<<"\n";

    dt = 0;
    counter = dg::integrateHRK<6>( functor, t_start, u_start, t_end, u_end, dt, 1e-6);
    dg::blas1::axpby( 1., solution(t_end, damping, omega_0, omega_drive), -1., u_end);
    std::cout << "With "<<counter<<"\t Halfstep RK 6 steps norm of error is\t "<<sqrt(dg::blas1::dot( u_end, u_end))<<"\n";

    dt = 0.;
    counter = dg::integrateHRK<17>( functor, t_start, u_start, t_end, u_end, dt, 1e-6);
    dg::blas1::axpby( 1., solution(t_end, damping, omega_0, omega_drive), -1., u_end);
    std::cout << "With "<<counter<<"\t Halfstep RK 17 steps norm of error is\t "<<sqrt(dg::blas1::dot( u_end, u_end))<<"\n";

    return 0;
}
