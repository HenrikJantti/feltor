#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include "cg.h"
#include "elliptic.h"
#include "chebyshev.h"

const double lx = 2.*M_PI;
const double ly = 2.*M_PI;


double fct(double x, double y){ return sin(y)*sin(x);}
double laplace_fct( double x, double y) { return 2*sin(y)*sin(x);}
double initial( double x, double y) {return sin(0);}

int main()
{
    //global relative error in L2 norm is O(h^P)
    //more N means less iterations for same error
    unsigned n, Nx, Ny;
    std::cout << "Type n, Nx and Ny! \n";
    std::cin >> n >> Nx >> Ny;
    std::cout << "Computing on the Grid " <<n<<" x "<<Nx<<" x "<<Ny <<std::endl;
    dg::Grid2d grid( 0, lx, 0, ly,n, Nx, Ny, dg::PER, dg::PER);
    std::cout<<"Evaluate initial condition\n";
    dg::HVec x = dg::evaluate( initial, grid);
    unsigned max_iter = n*n*Nx*Ny;
    const dg::HVec& copyable_vector = x;

//! [doxygen]
    // create volume and inverse volume on previously defined grid
    const dg::HVec w2d = dg::create::weights( grid);
    const dg::HVec v2d = dg::create::inv_weights( grid);

    // Create unnormalized Laplacian
    dg::Elliptic<dg::CartesianGrid2d, dg::HMatrix, dg::HVec> A( grid);

    // allocate memory in conjugate gradient
    dg::CG<dg::HVec > pcg( copyable_vector, max_iter);

    // Evaluate right hand side and solution on the grid
    dg::HVec b = dg::evaluate ( laplace_fct, grid);
    const dg::HVec solution = dg::evaluate ( fct, grid);

    // normalize right hand side
    dg::blas2::symv( w2d, b, b);

    // use inverse volume as preconditioner in solution method
    const double eps = 1e-6;
    std::cout << "Number of pcg iterations "<< pcg( A, x, b, v2d, eps)<<std::endl;
//! [doxygen]
    std::cout << "For a precision of "<< eps<<std::endl;
    //compute error
    dg::HVec error( solution);
    dg::blas1::axpby( 1.,x,-1.,error);

    dg::HVec Ax(x), resi( b);
    dg::blas2::symv(  A, x, Ax);
    dg::blas1::axpby( 1.,Ax,-1.,resi);

    exblas::udouble res;
    res.d = sqrt(dg::blas2::dot( w2d, x));
    std::cout << "L2 Norm of x0 is              " << res.d<<"\t"<<res.i << std::endl;
    res.d = sqrt(dg::blas2::dot(w2d , solution));
    std::cout << "L2 Norm of Solution is        " << res.d<<"\t"<<res.i << std::endl;
    res.d = sqrt(dg::blas2::dot(w2d , error));
    std::cout << "L2 Norm of Error is           " << res.d<<"\t"<<res.i << std::endl;
    res.d = sqrt(dg::blas2::dot( w2d, resi));
    std::cout << "L2 Norm of Residuum is        " << res.d<<"\t"<<res.i << std::endl;
    //Fehler der Integration des Sinus ist vernachlässigbar (vgl. evaluation_t)
    dg::blas1::copy( 0., x);
    dg::Chebyshev<dg::HVec> cheby( copyable_vector);
    double lmin = 1+1, lmax = n*n*Nx*Nx + n*n*Ny*Ny; //Eigenvalues of Laplace
    double hxhy = lx*ly/(n*n*Nx*Ny);
    lmin *= hxhy, lmax *= hxhy; //we multiplied the matrix by w2d
    unsigned num_iter = 200;
    cheby.solve( A, x, b, lmin, lmax, num_iter);
    std::cout << "After "<<num_iter<<" Chebyshev iterations we have:\n";

    dg::blas1::copy( solution, error);
    dg::blas1::axpby( 1.,x,-1.,error);

    dg::blas1::copy( b, resi);
    dg::blas2::symv(  A, x, Ax);
    dg::blas1::axpby( 1.,Ax,-1.,resi);

    res.d = sqrt(dg::blas2::dot( w2d, x));
    std::cout << "L2 Norm of x0 is              " << res.d<<"\n";
    res.d = sqrt(dg::blas2::dot(w2d , solution));
    std::cout << "L2 Norm of Solution is        " << res.d<<"\n";
    res.d = sqrt(dg::blas2::dot(w2d , error));
    std::cout << "L2 Norm of Error is           " << res.d<<"\n";
    res.d = sqrt(dg::blas2::dot( w2d, resi));
    std::cout << "L2 Norm of Residuum is        " << res.d<<"\n";


    return 0;
}
