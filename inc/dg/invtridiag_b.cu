#include "hip/hip_runtime.h"
// #undef DG_BENCHMARK
// #define DG_DEBUG

#include <iostream>
#include <iomanip>

#include "lanczos.h"

#include "backend/timer.h"
#include <cusp/transpose.h>
#include <cusp/print.h>
#include <cusp/array2d.h>
#include <cusp/elementwise.h>
#include <cusp/blas/blas.h>
#include "cg.h"
#include "lgmres.h"

using value_type = double;
using memory_type = cusp::host_memory;
using CooMatrix =  cusp::coo_matrix<int, value_type, memory_type>;
using DiaMatrix =  cusp::dia_matrix<int, value_type, memory_type>;
using Container = dg::HVec;
int main()
{
    dg::Timer t;
    unsigned size = 50;
    std::cout << "#Specify size of vectors (50)\n";
    std::cin >> size;
    //vectors of the tridiagonal matrix
//     std::vector<value_type> a = {1.98242, 4.45423, 5.31867, 7.48144, 7.11534};
//     std::vector<value_type> b = {-0.00710891, -0.054661, -0.0554193, -0.0172191, -0.297645};
//     std::vector<value_type> c = {-1.98242, -4.44712, -5.26401, -7.42602, -7.09812}; 
    std::cout << "#Constructing and filling vectors\n";
    std::vector<value_type> a(size,1.);
    std::vector<value_type> b(size,1.);
    std::vector<value_type> c(size,1.);
    for (unsigned i=0;i<size; i++)
    {
        a[i] = (1.+0.1*i);
        b[i] = 1./(1.+0.1*i);
        c[i] = (1.+0.1*i)+1./(1.+0.1*i);
    }
    std::cout << "#Constructing and filling containers\n";
    const Container d(size,1.);
    Container x(size,0.), x_sol(x), err(x);
    std::cout << "#Constructing Matrix inversion and linear solvers\n";
    value_type eps= 1e-14;
    t.tic();
    dg::CG <Container> pcg( x,  size*size);
    t.toc();
    std::cout << "#Construction of CG took "<< t.diff()<<"s \n";
    t.tic();
    dg::LGMRES <Container> lgmres( x, 30, 3, 1000*size);
    t.toc();
    std::cout << "#Construction of LGMRES took "<< t.diff()<<"s \n";
    t.tic();
    dg::InvTridiag<Container, DiaMatrix, CooMatrix> invtridiag(a);
    t.toc();
    std::cout << "#Construction of Tridiagonal inversion routine took "<< t.diff()<<"s \n";
    
    //Create Tridiagonal and fill matrix
    DiaMatrix T, Tsym; 
    T.resize(size, size, 3*size-2, 3);
    T.diagonal_offsets[0] = -1;
    T.diagonal_offsets[1] =  0;
    T.diagonal_offsets[2] =  1;
    Tsym.resize(size, size, 3*size-2, 3);
    Tsym.diagonal_offsets[0] = -1;
    Tsym.diagonal_offsets[1] =  0;
    Tsym.diagonal_offsets[2] =  1;
    
    for( unsigned i=0; i<size-1; i++)
    {
        T.values(i,1)   =  a[i];  // 0 diagonal
        T.values(i+1,0) =  c[i];  // -1 diagonal
        T.values(i,2)   =  b[i];  // +1 diagonal //dia_rows entry works since its outside of matrix
        Tsym.values(i,1)   =  a[i];  // 0 diagonal
        Tsym.values(i+1,0) =  b[i];  // -1 diagonal
        Tsym.values(i,2)   =  b[i];  // +1 diagonal //dia_rows entry works since its outside of matrix
    }
    T.values(size-1,1) =  a[size-1];
    Tsym.values(size-1,1) =  a[size-1];
//     std::cout << "T matrix\n";
//     cusp::print(T);
//     std::cout << "Tsym matrix\n";
//     cusp::print(Tsym);
    
    //Create and fill Inverse of tridiagonal matrix (the solution)
    CooMatrix Tinv, Tsyminv;
//     cusp::array2d<value_type ,memory_type> H(size,size), error(size,size,0.);
//     H(0,0) = 0.505249;  H(1,0) = 0.000814795;  H(2,0) =  8.4358e-6;     H(3,0) = 6.26392e-8;     H(4,0) =  1.51587e-10;
//     H(0,1) = 0.227217;  H(1,1) = 0.227217;     H(2,1) =  0.00235244;    H(3,1) = 0.0000174678;   H(4,1) =  4.22721e-8;
//     H(0,2) = 0.19139;   H(1,2) = 0.19139;      H(2,2) =  0.19139;       H(3,2) = 0.00142115;     H(4,2) =  3.43918e-6;
//     H(0,3) = 0.134988;  H(1,3) = 0.134988;     H(2,3) =  0.134988;      H(3,3) = 0.134988;       H(4,3) =  0.000326671;
//     H(0,4) = 0.140882;  H(1,4) = 0.140882;     H(2,4) =  0.140882;      H(3,4) = 0.140882;       H(4,4) = 0.140882;
//     CooMatrix Tinv_sol, Tinv_error;
//     cusp::convert(H,Tinv_sol);
//     cusp::convert(error,Tinv_error);
    
    

    std::cout << "####Compute inverse of symmetric tridiagonal matrix\n";
    dg::blas1::scal(x_sol, 0.);
    t.tic();
    unsigned number = pcg( Tsym, x_sol, d, d, eps);
    if(  number == pcg.get_max())
        throw dg::Fail( eps);
    t.toc();
    std::cout <<  "#CG took: "<< t.diff()<<"s \n";
    t.tic();
    Tsyminv = invtridiag(a,b,b);
    t.toc();
    dg::blas2::gemv(Tsyminv, d, x);
    dg::blas1::axpby(1.0, x, -1.0, x_sol, err );
    std::cout <<  "#Invtridiag with vectors took: "<< t.diff()<<"s \n";
    std::cout <<  "#Relative error to CG: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_sol,x_sol)) << "\n";

    t.tic();
    Tsyminv = invtridiag(Tsym);
    t.toc();
    dg::blas2::gemv(Tsyminv, d, x);
    dg::blas1::axpby(1.0, x, -1.0, x_sol, err );
    std::cout <<  "#Invtridiag with Matrix took: "<< t.diff()<<"s \n";
    std::cout <<  "#Relative error to CG: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_sol,x_sol)) << "\n";
    

    std::cout << "####Compute inverse of non-symmetric tridiagonal matrix\n";
    t.tic();
    number = lgmres.solve( T, x_sol, d , d, d, eps, 1);    
    t.toc();
    std::cout <<  "#lGMRES took: "<< t.diff()<<"s \n";
    
    t.tic();
    Tinv = invtridiag(a,b,c);
    t.toc();
    dg::blas2::gemv(Tinv, d, x);
    dg::blas1::axpby(1.0, x, -1.0, x_sol, err );
    std::cout <<  "#Invtridiag with vectors took: "<< t.diff()<<"s \n";
    std::cout <<  "#Relative error to lGMRES: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_sol,x_sol)) << "\n";
    t.tic();
    Tinv = invtridiag(T);
    t.toc();
    dg::blas2::gemv(Tinv, d, x);
    dg::blas1::axpby(1.0, x, -1.0, x_sol, err );
    std::cout <<  "#Invtridiag with Matrix took: "<< t.diff()<<"s \n";
    std::cout <<  "#Relative error to lGMRES: " << sqrt(dg::blas1::dot(err,err)/dg::blas1::dot(x_sol,x_sol)) << "\n";    
    return 0;
}
