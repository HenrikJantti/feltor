#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <mpi.h>
#include <thrust/device_vector.h>
#include "blas1.h"
#include "backend/mpi_evaluation.h"


//test program that calls every blas1 function for every specialization
double two( double x, double y){return 2.;}
double three( double x, double y){return 3.;}

//typedef dg::MPI_Vector<thrust::device_vector<double> > MHVec;
typedef dg::MPI_Vector<cusp::array1d<double, cusp::device_memory> > MHVec;

struct EXP{ __host__ __device__ double operator()(double x){return exp(x);}};

int main( int argc, char* argv[])
{
    MPI_Init(&argc, &argv);
    int np[2];
    int periods[2] = {0,0};
    periods[0] = 1;
    periods[1] = 1;
    int rank;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    if( rank == 0)
    {
        std::cout << "Type npx and npy\n";
        std::cin >> np[0] >> np[1];
        std::cout<< "You typed "<<np[0] <<" and "<<np[1]<<std::endl;
    }
    MPI_Bcast( np, 2, MPI_INT, 0, MPI_COMM_WORLD);

    int size;
    MPI_Comm_size( MPI_COMM_WORLD, &size);
    if( rank == 0)
    {
        std::cout << "Size is "<<size<<std::endl;
        assert( size == np[0]*np[1]);
    }

    MPI_Comm comm;
    MPI_Cart_create( MPI_COMM_WORLD, 2, np, periods, true, &comm);
    dg::MPIGrid2d g( 0,1,0,1, 3,12,12, comm);
    if( rank == 0)
        g.display();
    MHVec v1 = dg::evaluate( two, g);
    MHVec v2 = dg::evaluate( three, g); 
    MHVec v3(v1);
    unsigned gsize = g.global().n()*g.global().n()*g.global().Nx()*g.global().Ny();

    double temp = dg::blas1::dot(v1,v2);
    if(rank==0)std::cout << "(2*3) = "<<temp/gsize << " (6)\n"; 
    dg::blas1::axpby( 2., v1, 3., v2, v3);
    if(rank==0)std::cout << "2*2+ 3*3 = " << v3[0] <<" (13)\n";
    dg::blas1::axpby( 0., v1, 3., v2, v3);
    if(rank==0)std::cout << "0*2+ 3*3 = " << v3[0] <<" (9)\n";
    dg::blas1::axpby( 2., v1, 0., v2, v3);
    if(rank==0)std::cout << "2*2+ 0*3 = " << v3[0] <<" (4)\n";
    dg::blas1::pointwiseDot( v1, v2, v3);
    if(rank==0)std::cout << "2*3 = "<<v3[0]<<" (6)\n";
    dg::blas1::pointwiseDot( 2., v1, v2, -4., v3);
    if(rank==0)std::cout << "2*2*3 -4*6 = "<<v3[0]<<" (-12)\n";
    dg::blas1::axpby( 2., v1, 3., v2);
    if(rank==0)std::cout << "2*2+ 3*3 = " << v2[0] <<" (13)\n";
    dg::blas1::axpby( 2.5, v1, 0., v2);
    if(rank==0)std::cout << "2.5*2+ 0 = " << v2[0] <<" (5)\n";
    dg::blas1::copy( v2, v1);
    if(rank==0)std::cout << "5 = " << v1[0] <<" (5)"<< std::endl;
    dg::blas1::scal( v1, 0.4);
    if(rank==0)std::cout << "5*0.4 = " << v1[0] <<" (2)"<< std::endl;
    dg::blas1::transform( v1, v3, EXP());
    if(rank==0)std::cout << "e^2 = " << v3[0] <<" (7.389056...)"<< std::endl;
    dg::blas1::scal( v2, 0.6);
    dg::blas1::plus( v3, -7.0);
    if(rank==0)std::cout << "e^2-7 = " << v3[0] <<" (0.389056...)"<< std::endl;

    //v1 = 2, v2 = 3

    if(rank==0)std::cout << "Test std::vector \n";
    std::vector<MHVec > w1( 2, v1), w2(2, v2), w3( w2);
    temp = dg::blas1::dot( w1, w2);
    if(rank==0)std::cout << "2*(2*3) = "<<temp/gsize << " (12)\n"; 
    dg::blas1::axpby( 2., w1, 3., w2, w3);
    if(rank==0)std::cout << "2*2+ 3*3 = " << w3[0][0] <<" (13)\n";
    dg::blas1::axpby( 0., w1, 3., w2, w3);
    if(rank==0)std::cout << "0*2+ 3*3 = " << w3[0][0] <<" (9)\n";
    dg::blas1::axpby( 2., w1, 0., w2, w3);
    if(rank==0)std::cout << "2*2+ 0*3 = " << w3[0][0] <<" (4)\n";
    dg::blas1::pointwiseDot( w1, w2, w3);
    if(rank==0)std::cout << "2*3 = "<<w3[0][0]<<" (6)\n";
    dg::blas1::pointwiseDot( 2., w1, w2, -4., w3);
    if(rank==0)std::cout << "2*2*3 -4*6 = "<<w3[0][0]<<" (-12)\n";
    dg::blas1::axpby( 2., w1, 3., w2);
    if(rank==0)std::cout << "2*2+ 3*3 = " << w2[0][0] <<" (13)\n";
    dg::blas1::axpby( 2.5, w1, 0., w2);
    if(rank==0)std::cout << "2.5*2+ 0 = " << w2[0][0] <<" (5)\n";
    dg::blas1::copy( w2, w1);
    if(rank==0)std::cout << "5 = " << w1[0][0] <<" (5)"<< std::endl;
    dg::blas1::scal( w1, 0.4);
    if(rank==0)std::cout << "5*0.5 = " << w1[0][0] <<" (2)"<< std::endl;
    dg::blas1::transform( w1, w3, EXP());
    if(rank==0)std::cout << "e^2 = " << w3[0][0] <<" (7.389056...)"<< std::endl;
    dg::blas1::scal( w2, 0.6);
    dg::blas1::plus( w3, -7.0);
    if(rank==0)std::cout << "e^2-7 = " << w3[0][0] <<" (0.389056...)"<< std::endl;
    if(rank==0)std::cout << "FINISHED\n\n";



    MPI_Finalize();
    return 0;

}
