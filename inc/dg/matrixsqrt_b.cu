#include "hip/hip_runtime.h"
#define __STDCPP_WANT_MATH_SPEC_FUNCS__ 1
#define SILENT
#include <boost/math/special_functions/jacobi_elliptic.hpp>

#include <iostream>
#include <iomanip>

#include "blas.h"
#include "backend/typedefs.h"
#include "topology/evaluation.h"
#include "adaptive.h"
#include "helmholtz.h"
#include "backend/timer.h"

#include "lanczos.h"
#include "sqrt_cauchy.h"
#include "matrixsqrt.h"
#include "eve.h"

const double lx = 2.*M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;
const double alpha = -0.5;

double lhs( double x, double y){ return sin(x)*sin(y);}
double rhsHelmholtz( double x, double y){ return (1.-2.*alpha)*sin(x)*sin(y);}
double rhsHelmholtzsqrt( double x, double y){ return sqrt(1.-2.*alpha)*sin(x)*sin(y);}

using DiaMatrix =  cusp::dia_matrix<int, double, cusp::host_memory>;
using CooMatrix =  cusp::coo_matrix<int, double, cusp::host_memory>;
int main()
{
    
    dg::Timer t;

    std::cout << "Test program for A^(1/2) x computation \n";
    
    unsigned n, Nx, Ny;
    std::cout << "Type n, Nx and Ny! \n";
    std::cin >> n >> Nx >> Ny;
    std::cout << "Computing on the Grid " <<n<<" x "<<Nx<<" x "<<Ny <<std::endl;
    dg::Grid2d grid( 0, lx, 0, ly,n, Nx, Ny, bcx, bcy);
   //start and end vectors
    dg::HVec x = dg::evaluate(lhs, grid);
    dg::HVec b = dg::evaluate(rhsHelmholtz, grid);
    dg::HVec b_exac = dg::evaluate(rhsHelmholtzsqrt, grid), error(b_exac);

    const dg::HVec w2d = dg::create::weights( grid);
    const dg::HVec v2d = dg::create::inv_weights( grid);

    dg::Helmholtz<dg::CartesianGrid2d, dg::HMatrix, dg::HVec> A( grid, alpha, dg::centered); //not_normed
    double epsCG, epsTimerel, epsTimeabs;
//     std::cout << "Type epsilon for CG (1e-5), and eps_rel (1e-5) and eps_abs (1e-10) for TimeStepper\n";
//     std::cin >> epsCG >> epsTimerel >> epsTimeabs;
    epsCG=1e-8;
    epsTimerel=1e-8;
    epsTimeabs=1e-12;
    int counter = 0;
    double erel = 0;
    
    //////////////////////////Direct Cauchy integral solve
    std::cout << "Solving  via Cauchy integral\n";
    CauchySqrtInt<dg::CartesianGrid2d, dg::HMatrix, dg::HVec> cauchysqrtint(A, grid, epsCG);
    dg::EVE<dg::HVec> eve(x, 100);
    unsigned iter;
    std::cout << "# of Cauchy terms?\n";
    std::cin >> iter;
    double lambda_min = 1; //Exact estimate missing, However as long as chi in helmholtz is 1 it is correct
    double lambda_max;
    t.tic();
    eve(A, b, b, A.precond(),lambda_max);
    std::cout << "Maximum EV is: "<< lambda_max << "\n";
    cauchysqrtint(x, b, lambda_min, lambda_max , iter);
    t.toc();
    dg::blas1::axpby(1.0, b, -1.0, b_exac, error);
    erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, b_exac));   
    std::cout << "   Time: "<<t.diff()<<"s  Relative error: "<<erel <<"\n";    //error should be much smaller after a few iterations with correct EVs (does it converge properly?)
    
    
    //////////////////////////Direct sqrt ODE solve
    std::cout << "Solving  via Direct sqrt ODE\n";
    DirectSqrtSolve<dg::CartesianGrid2d, dg::HMatrix, dg::HVec> directsqrtsolve(A, grid, epsCG, epsTimerel, epsTimeabs);
    t.tic();
    counter = directsqrtsolve(x, b); //overwrites b
    t.toc();

    dg::blas1::axpby(1.0, b, -1.0, b_exac, error);
    erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, b_exac));   
    std::cout << "Time steps: "<<std::setw(6)<<counter  << "   Time: "<<t.diff()<<"s  Relative error: "<<erel <<"\n"; 
    

    ////////////////////////Krylov solve via Lanczos method and ODE sqrt solve
    std::cout << "Solving  via Krylov method and sqrt ODE\n";
    std::cout << "# of Lanczos iterations?\n";
    std::cin >> iter;
  
    KrylovSqrtSolve<dg::CartesianGrid2d, dg::HMatrix, DiaMatrix, CooMatrix, dg::HVec> krylovsqrtsolve(A, grid, x,  epsCG, epsTimerel, epsTimeabs, iter);
    t.tic();
    counter = krylovsqrtsolve(x, b); //overwrites b
    t.toc();
    
    dg::blas1::axpby(1.0, b, -1.0, b_exac, error);
    erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, b_exac));   
    std::cout << "Time steps: "<<std::setw(6)<<counter  << " Time: "<<t.diff()<<"s  Relative error: "<<erel <<"\n";
   
    return 0;
}
