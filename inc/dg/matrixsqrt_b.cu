#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include "blas.h"
#include "backend/typedefs.h"
#include "topology/evaluation.h"
#include "adaptive.h"
#include "helmholtz.h"
#include "backend/timer.h"

#include "lanczos.h"

#include "matrixsqrt.h"

const double lx = 2.*M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;
const double alpha = -0.5;

double lhs( double x, double y){ return sin(x)*sin(y);}
// double rhsHelmholtz( double x, double y){ return (1.-2.*alpha)*sin(x)*sin(y);}
double rhsHelmholtzsqrt( double x, double y){ return sqrt(1.-2.*alpha)*sin(x)*sin(y);}

using DiaMatrix =  cusp::dia_matrix<int, double, cusp::host_memory>;
using CooMatrix =  cusp::coo_matrix<int, double, cusp::host_memory>;
int main()
{
    
    dg::Timer t;

    std::cout << "Test program for A^(1/2) x computation \n";
    
    unsigned n, Nx, Ny;
    std::cout << "Type n, Nx and Ny! \n";
    std::cin >> n >> Nx >> Ny;
    std::cout << "Computing on the Grid " <<n<<" x "<<Nx<<" x "<<Ny <<std::endl;
    dg::Grid2d grid( 0, lx, 0, ly,n, Nx, Ny, bcx, bcy);
   //start and end vectors
    dg::HVec x = dg::evaluate(lhs, grid), b(x), b_exac(x), error(b_exac);
    const dg::HVec w2d = dg::create::weights( grid);
    const dg::HVec v2d = dg::create::inv_weights( grid);
    b_exac = dg::evaluate(rhsHelmholtzsqrt, grid);

    dg::Helmholtz<dg::CartesianGrid2d, dg::HMatrix, dg::HVec> A( grid, alpha, dg::centered); //not_normed
    double epsCG, epsTimerel, epsTimeabs;
//     std::cout << "Type epsilon for CG (1e-5), and eps_rel (1e-5) and eps_abs (1e-10) for TimeStepper\n";
//     std::cin >> epsCG >> epsTimerel >> epsTimeabs;
    epsCG=1e-8;
    epsTimerel=1e-8;
    epsTimeabs=1e-12;
    int counter =0;
    double erel=0;
    
    //////////////////////////Direct sqrt ODE solve
    std::cout << "Solving  via Direct sqrt ODE\n";
    DirectSqrtSolve<dg::CartesianGrid2d, dg::HMatrix, dg::HVec> directsqrtsolve(A, grid, epsCG, epsTimerel, epsTimeabs);
    t.tic();
    counter = directsqrtsolve(x, b); //overwrites b
    t.toc();

    dg::blas1::axpby(1.0, b, -1.0, b_exac, error);
    erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, b_exac));   
    std::cout << "Time steps: "<<std::setw(6)<<counter  << "   Time: "<<t.diff()<<"s  Relative error: "<<erel <<"\n";    

    ////////////////////////Krylov solve via Lanczos method and ODE sqrt solve
    std::cout << "Solving  via Krylov method and sqrt ODE\n";
    unsigned iter;
    std::cout << "# of Lanczos iterations?\n";
    std::cin >> iter;
  
    KrylovSqrtSolve<dg::CartesianGrid2d, dg::HMatrix, DiaMatrix, CooMatrix, dg::HVec> krylovsqrtsolve(A, grid, x,  epsCG, epsTimerel, epsTimeabs, iter);
    t.tic();
    counter = krylovsqrtsolve(x, b); //overwrites b
    t.toc();
    
    dg::blas1::axpby(1.0, b, -1.0, b_exac, error);
    erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, b_exac));   
    std::cout << "Time steps: "<<std::setw(6)<<counter  << " Time: "<<t.diff()<<"s  Relative error: "<<erel <<"\n";
   
    return 0;
}
