#include "hip/hip_runtime.h"
#define __STDCPP_WANT_MATH_SPEC_FUNCS__ 1
#define SILENT
#include <boost/math/special_functions/jacobi_elliptic.hpp>

#include <iostream>
#include <iomanip>

#include "blas.h"
#include "backend/typedefs.h"
#include "topology/evaluation.h"
#include "adaptive.h"
#include "helmholtz.h"
#include "backend/timer.h"

// #include "cg.h"
#include "lanczos.h"
#include "sqrt_cauchy.h"
#include "matrixsqrt.h"
#include "eve.h"

const double lx = 2.*M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;
const double alpha = -0.5;

double lhs( double x, double y){ return sin(x)*sin(y);}
double rhsHelmholtz( double x, double y){ return (1.-2.*alpha)*sin(x)*sin(y);}
double rhsHelmholtzsqrt( double x, double y){ return sqrt(1.-2.*alpha)*sin(x)*sin(y);}

using DiaMatrix =  cusp::dia_matrix<int, double, cusp::host_memory>;
using CooMatrix =  cusp::coo_matrix<int, double, cusp::host_memory>;
int main()
{
    
    dg::Timer t;

    std::cout << "Test program for A^(1/2) x computation \n";
    
    unsigned n, Nx, Ny;
    std::cout << "Type n, Nx and Ny! \n";
    std::cin >> n >> Nx >> Ny;
    std::cout << "Computing on the Grid " <<n<<" x "<<Nx<<" x "<<Ny <<std::endl;
    dg::Grid2d grid( 0, lx, 0, ly,n, Nx, Ny, bcx, bcy);
   //start and end vectors
    dg::HVec x = dg::evaluate(lhs, grid);
    dg::HVec x_exac = dg::evaluate(lhs, grid);
    dg::HVec b = dg::evaluate(rhsHelmholtzsqrt, grid), b_exac(b), error(b_exac);
    dg::HVec bs = dg::evaluate(rhsHelmholtz, grid), bs_exac(bs);

    const dg::HVec w2d = dg::create::weights( grid);
    const dg::HVec v2d = dg::create::inv_weights( grid);

    dg::Helmholtz<dg::CartesianGrid2d, dg::HMatrix, dg::HVec> A( grid, alpha, dg::centered); //not_normed
    double epsCG, epsTimerel, epsTimeabs;
    

//     std::cout << "Type epsilon for CG (1e-5), and eps_rel (1e-5) and eps_abs (1e-10) for TimeStepper\n";
//     std::cin >> epsCG >> epsTimerel >> epsTimeabs;
    epsCG=1e-14;
    epsTimerel=1e-4;
    epsTimeabs=1e-14;
    int counter = 0;
    double erel = 0;
    unsigned iter = 1;
    unsigned iterCauchy = 1;
    
    dg::Invert<dg::HVec> invert( x, grid.size(), epsCG);


    //////////////////////////Direct Cauchy integral solve
    std::cout << "Solving  via Cauchy integral\n";
    CauchySqrtInt<dg::CartesianGrid2d, dg::HMatrix, dg::HVec> cauchysqrtint(A, grid, epsCG);
    dg::EVE<dg::HVec> eve(b, 100);
    std::cout << "# of Cauchy terms?\n";
    std::cin >> iter;
//     double lambda_min = 1; //Exact estimate missing, However as long as chi in helmholtz is 1 it is correct
    double lambda_max;
    t.tic();
    eve(A, bs, bs, A.inv_weights(),lambda_max);
    std::cout << "Maximum EV from EVE is: "<< lambda_max << "\n";
    
    //analyitcal estimate
    double lmin = 1+1, lmax = n*n*Nx*Nx + n*n*Ny*Ny; //Eigenvalues of Laplace
    double hxhy = lx*ly/(n*n*Nx*Ny);
    lmin *= hxhy, lmax *= hxhy; //we multiplied the matrix by w2d
    std::cout << "Min and Maximum EV is: "<< -lmin*alpha+1 << "  "<<-lmax*alpha+1<< "\n";
   
    cauchysqrtint(b, bs,-lmin*alpha+1 ,-lmax*alpha+1, iter);
    t.toc();
    dg::blas1::axpby(1.0, bs, -1.0, bs_exac, error);
    erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, bs_exac));   
    std::cout << "   Time: "<<t.diff()<<"s  Relative error: "<<erel <<"\n";    //error should be much smaller after a few iterations with correct EVs, reason is most likely that the EVs are not exactly estimated, error is also very sensible to min and max EVs
    
    //solve for x=\sqrt{A}^{-1} b'
    t.tic();
    invert(A,x,bs);
    t.toc();
    dg::blas1::axpby(1.0, x, -1.0, x_exac, error);
    erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, x_exac));   
    std::cout << "   Time: "<<t.diff()<<"s  Relative error: "<<erel <<"\n";    //error should be much 
    
    //////////////////////////Direct sqrt ODE solve
    std::cout << "Solving  via Direct sqrt ODE\n";
    DirectSqrtODESolve<dg::CartesianGrid2d, dg::HMatrix, dg::HVec> directsqrtodesolve(A, grid, epsCG, epsTimerel, epsTimeabs);
    t.tic();
    counter = directsqrtodesolve(b, bs); //overwrites b
    t.toc();

    dg::blas1::axpby(1.0, bs, -1.0, bs_exac, error);
    erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, bs_exac));   
    std::cout  << "   Time: "<<t.diff()<<"s  Relative error: "<<erel <<"  Time steps: "<<std::setw(3)<<counter << "\n"; 
    //solve for x=\sqrt{A}^{-1} b'
    t.tic();
    invert(A,x,bs);
    t.toc();
    dg::blas1::axpby(1.0, x, -1.0, x_exac, error);
    erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, x_exac));   
    std::cout << "   Time: "<<t.diff()<<"s  Relative error: "<<erel <<"\n";    //error should be much 


        ////////////////////////Krylov solve via Lanczos method and ODE sqrt solve
    std::cout << "Solving  via Krylov method and Cauchy ODE\n";
    std::cout << "# of Lanczos iterations and Cauchy terms?\n";
    std::cin >> iter >> iterCauchy;
  
    KrylovSqrtCauchySolve<dg::CartesianGrid2d, dg::HMatrix, DiaMatrix, CooMatrix, dg::HVec> krylovsqrtcauchysolve(A, grid, x,  epsCG, iter);
    t.tic();
    krylovsqrtcauchysolve(b, bs, 5); //overwrites b
    t.toc();
    
    dg::blas1::axpby(1.0, bs, -1.0, bs_exac, error);
    erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, bs_exac));   
    std::cout << " Time: "<<t.diff()<<"s  Relative error: "<<erel <<"\n";
    
    //solve for x=\sqrt{A}^{-1} b'
    t.tic();
    invert(A,x,bs);
    t.toc();
    dg::blas1::axpby(1.0, x, -1.0, x_exac, error);
    erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, x_exac));   
    std::cout << "   Time: "<<t.diff()<<"s  Relative error: "<<erel <<"\n";    //error should be much 
    
    ////////////////////////Krylov solve via Lanczos method and ODE sqrt solve
    std::cout << "Solving  via Krylov method and sqrt ODE\n";
    std::cout << "# of Lanczos iterations?\n";
    std::cin >> iter;
  
    KrylovSqrtODESolve<dg::CartesianGrid2d, dg::HMatrix, DiaMatrix, CooMatrix, dg::HVec> krylovsqrtodesolve(A, grid, x,  epsCG, epsTimerel, epsTimeabs, iter);
    t.tic();
    counter = krylovsqrtodesolve(b, bs); //overwrites b
    t.toc();
    
    dg::blas1::axpby(1.0, bs, -1.0, bs_exac, error);
    erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, bs_exac));   
    std::cout  << "   Time: "<<t.diff()<<"s  Relative error: "<<erel <<"  Time steps: "<<std::setw(3)<<counter << "\n"; 
   
    //solve for x=\sqrt{A}^{-1} b'
    t.tic();
    invert(A,x,bs);
    t.toc();
    dg::blas1::axpby(1.0, x, -1.0, x_exac, error);
    erel = sqrt(dg::blas2::dot( w2d, error) / dg::blas2::dot( w2d, x_exac));   
    std::cout << "   Time: "<<t.diff()<<"s  Relative error: "<<erel <<"\n";    //error should be much 
    
    
    //Direct CG sqrt solve TODO
    
    return 0;
}
