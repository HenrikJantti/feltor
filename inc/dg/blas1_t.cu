#include "hip/hip_runtime.h"
//#define CUSP_DEVICE_BLAS_SYSTEM CUSP_DEVICE_BLAS_CUBLAS
#include <iostream>
#include <vector>
#include <array>

#include "blas1.h"

struct EXP{ __host__ __device__ double operator()(double x){return exp(x);}};


//test program that (should ) call every blas1 function for every specialization

//using Vector = std::array<double,2>;
using Vector = thrust::host_vector<double>;
//using Vector = thrust::device_vector<double>;
//using Vector = cusp::array1d<double, cusp::device_memory>;
int main()
{
    Vector v1( 5, 2), v2( 5, 3), v3(5,5), v4(5,4);
    //Vector v1( {2,2}), v2({3,3}), v3({5,5}), v4({4,4}); //std::array
    //thrust::device_vector<double> v1p( 500, 2), v2p( 500, 3), v3p(500,5), v4p(500,4);
    //Vector v1(v1p), v2(v2p), v3(v3p), v4(v4p);
    double temp = dg::blas1::dot(v1,v2);
    std::cout << "5*(2*3) = "<<temp << " (30)\n"; 
    dg::blas1::axpby( 2., v1, 3., v2, v3);
    std::cout << "2*2+ 3*3 = " << v3[0] <<" (13)\n";
    dg::blas1::axpby( 0., v1, 3., v2, v3);
    std::cout << "0*2+ 3*3 = " << v3[0] <<" (9)\n";
    dg::blas1::axpby( 2., v1, 0., v2, v3);
    std::cout << "2*2+ 0*3 = " << v3[0] <<" (4)\n";
    dg::blas1::pointwiseDot( v1, v2, v3);
    std::cout << "2*3 = "<<v3[0]<<" (6)\n";
    dg::blas1::pointwiseDot( 2., v1, v2, -4., v3);
    std::cout << "2*2*3 -4*6 = "<<v3[0]<<" (-12)\n";
    dg::blas1::pointwiseDot( 2., v1, v2,v4, -4., v3);
    std::cout << "2*2*3*4 -4*(-12) = "<<v3[0]<<" (96)\n";
    dg::blas1::axpby( 2., v1, 3., v2);
    std::cout << "2*2+ 3*3 = " << v2[0] <<" (13)\n";
    dg::blas1::axpby( 2.5, v1, 0., v2);
    std::cout << "2.5*2+ 0 = " << v2[0] <<" (5)\n";
    dg::blas1::axpbypgz( 2.5, v1, 2., v2, -0.125, v3);
    std::cout << "2.5*2+ 2.*5-0.125*96 = " << v3[0] <<" (3)\n";
    dg::blas1::pointwiseDivide( 5.,v1,v2,-1,v3);
    std::cout << "5*2/5-1*3 = " << v3[0] <<" (-1)\n";
    dg::blas1::copy( v2, v1);
    std::cout << "5 = " << v1[0] <<" (5)"<< std::endl;
    dg::blas1::scal( v1, 0.4);
    std::cout << "5*0.4 = " << v1[0] <<" (2)"<< std::endl;
    dg::blas1::transform( v1, v3, EXP());
    std::cout << "e^2 = " << v3[0] <<" (7.389056...)"<< std::endl;
    dg::blas1::scal( v2, 0.6);
    dg::blas1::plus( v3, -7.0);
    std::cout << "e^2-7 = " << v3[0] <<" (0.389056...)"<< std::endl;

    //v1 = 2, v2 = 3

    std::cout << "Test std::array \n";
    std::array<Vector, 2> w1( dg::transfer<std::array<Vector,2>>(v1)), w2({v2,v2}), w3({v3,v3}), w4({v4,v4});
    temp = dg::blas1::dot( w1, w2);
    std::cout << "2*5*(2*3) = "<<temp << " (60)\n"; 
    dg::blas1::axpby( 2., w1, 3., w2, w3);
    std::cout << "2*2+ 3*3 = " << w3[0][0] <<" (13)\n";
    dg::blas1::axpby( 0., w1, 3., w2, w3);
    std::cout << "0*2+ 3*3 = " << w3[0][0] <<" (9)\n";
    dg::blas1::axpby( 2., w1, 0., w2, w3);
    std::cout << "2*2+ 0*3 = " << w3[0][0] <<" (4)\n";
    dg::blas1::pointwiseDot( w1, w2, w3);
    std::cout << "2*3 = "<<w3[0][0]<<" (6)\n";
    dg::blas1::pointwiseDot( 2., w1, w2, -4., w3);
    std::cout << "2*2*3 -4*6 = "<<w3[0][0]<<" (-12)\n";
    dg::blas1::pointwiseDot( 2., w1, w2,w4, -4., w3);
    std::cout << "2*2*3*4 -4*(-12) = "<<w3[0][0]<<" (96)\n";
    dg::blas1::pointwiseDot( 2., w1, w2, -4., w1, w2, 0., w2);
    std::cout << "2*2*3 -4*2*3 = "<<w2[0][0]<<" (-12)\n";
    dg::blas1::axpby( 2., w1, 3., w2);
    std::cout << "2*2+ 3*3 = " << w2[0][0] <<" (13)\n";
    dg::blas1::axpby( 2.5, w1, 0., w2);
    std::cout << "2.5*2+ 0 = " << w2[0][0] <<" (5)\n";
    dg::blas1::axpbypgz( 2.5, w1, 2., w2, -0.125, w3);
    std::cout << "2.5*2+ 2.*5-0.125*96 = " << w3[0][0] <<" (3)\n";
    dg::blas1::pointwiseDivide( 5.,w1,w2,-1,w3);
    std::cout << "5*2/5-1*3 = " << w3[0][0] <<" (-1)\n";
    dg::blas1::copy( w2, w1);
    std::cout << "5 = " << w1[0][0] <<" (5)"<< std::endl;
    dg::blas1::scal( w1, 0.4);
    std::cout << "5*0.5 = " << w1[0][0] <<" (2)"<< std::endl;
    dg::blas1::transform( w1, w3, EXP());
    std::cout << "e^2 = " << w3[0][0] <<" (7.389056...)"<< std::endl;
    dg::blas1::scal( w2, 0.6);
    dg::blas1::plus( w3, -7.0);
    std::cout << "e^2-7 = " << w3[0][0] <<" (0.389056...)"<< std::endl;
    std::cout << "FINISHED\n\n";


    return 0;

}

