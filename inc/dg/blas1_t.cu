#include "hip/hip_runtime.h"
//#define CUSP_DEVICE_BLAS_SYSTEM CUSP_DEVICE_BLAS_CUBLAS
#include <iostream>
#include <vector>
#include <array>

#include "blas1.h"
#include "backend/blas1_array.h"

struct EXP{ __host__ __device__ double operator()(double x){return exp(x);}};


//test program that (should ) call every blas1 function for every specialization

//typedef thrust::device_vector<double>  Vector;
typedef cusp::array1d<double, cusp::device_memory>  Vector;
int main()
{
    Vector v1( 5, 2), v2( 5, 3), v3(5), v4(5,4);
    double temp = dg::blas1::dot(v1,v2);
    std::cout << "5*(2*3) = "<<temp << " (30)\n"; 
    dg::blas1::axpby( 2., v1, 3., v2, v3);
    std::cout << "2*2+ 3*3 = " << v3[0] <<" (13)\n";
    dg::blas1::axpby( 0., v1, 3., v2, v3);
    std::cout << "0*2+ 3*3 = " << v3[0] <<" (9)\n";
    dg::blas1::axpby( 2., v1, 0., v2, v3);
    std::cout << "2*2+ 0*3 = " << v3[0] <<" (4)\n";
    dg::blas1::pointwiseDot( v1, v2, v3);
    std::cout << "2*3 = "<<v3[0]<<" (6)\n";
    dg::blas1::pointwiseDot( 2., v1, v2, -4., v3);
    std::cout << "2*2*3 -4*6 = "<<v3[0]<<" (-12)\n";
    dg::blas1::pointwiseDot( 2., v1, v2,v4, -4., v3);
    std::cout << "2*2*3*4 -4*(-12) = "<<v3[0]<<" (96)\n";
    dg::blas1::axpby( 2., v1, 3., v2);
    std::cout << "2*2+ 3*3 = " << v2[0] <<" (13)\n";
    dg::blas1::axpby( 2.5, v1, 0., v2);
    std::cout << "2.5*2+ 0 = " << v2[0] <<" (5)\n";
    dg::blas1::axpbypgz( 2.5, v1, 2., v2, 3., v3);
    std::cout << "2.5*2+ 2.*5-3*12 = " << v3[0] <<" (-21)\n";
    dg::blas1::pointwiseDivide( 5.,v1,v2,-1,v3);
    std::cout << "5*2/5-1*21 = " << v3[0] <<" (-19)\n";
    dg::blas1::copy( v2, v1);
    std::cout << "5 = " << v1[0] <<" (5)"<< std::endl;
    dg::blas1::scal( v1, 0.4);
    std::cout << "5*0.4 = " << v1[0] <<" (2)"<< std::endl;
    dg::blas1::transform( v1, v3, EXP());
    std::cout << "e^2 = " << v3[0] <<" (7.389056...)"<< std::endl;
    dg::blas1::scal( v2, 0.6);
    dg::blas1::plus( v3, -7.0);
    std::cout << "e^2-7 = " << v3[0] <<" (0.389056...)"<< std::endl;

    //v1 = 2, v2 = 3

    std::cout << "Test std::array \n";
    std::array<Vector, 2> w1( {v1,v1}), w2({v2,v2}), w3({v3,v3}), w4({v4,v4});
    temp = dg::blas1::dot( w1, w2);
    std::cout << "2*5*(2*3) = "<<temp << " (60)\n"; 
    dg::blas1::axpby( 2., w1, 3., w2, w3);
    std::cout << "2*2+ 3*3 = " << w3[0][0] <<" (13)\n";
    dg::blas1::axpby( 0., w1, 3., w2, w3);
    std::cout << "0*2+ 3*3 = " << w3[0][0] <<" (9)\n";
    dg::blas1::axpby( 2., w1, 0., w2, w3);
    std::cout << "2*2+ 0*3 = " << w3[0][0] <<" (4)\n";
    dg::blas1::pointwiseDot( w1, w2, w3);
    std::cout << "2*3 = "<<w3[0][0]<<" (6)\n";
    dg::blas1::pointwiseDot( 2., w1, w2, -4., w3);
    std::cout << "2*2*3 -4*6 = "<<w3[0][0]<<" (-12)\n";
    dg::blas1::pointwiseDot( 2., w1, w2,w4, -4., w3);
    std::cout << "2*2*3*4 -4*(-12) = "<<w3[0][0]<<" (96)\n";
    dg::blas1::pointwiseDot( 2., w1[0], w2[0], -4., v1, v2, 0., v2);
    std::cout << "2*2*3 -4*2*3 = "<<v2[0]<<" (-12)\n";
    dg::blas1::axpby( 2., w1, 3., w2);
    std::cout << "2*2+ 3*3 = " << w2[0][0] <<" (13)\n";
    dg::blas1::axpby( 2.5, w1, 0., w2);
    std::cout << "2.5*2+ 0 = " << w2[0][0] <<" (5)\n";
    dg::blas1::copy( w2, w1);
    std::cout << "5 = " << w1[0][0] <<" (5)"<< std::endl;
    dg::blas1::scal( w1, 0.4);
    std::cout << "5*0.5 = " << w1[0][0] <<" (2)"<< std::endl;
    dg::blas1::transform( w1, w3, EXP());
    std::cout << "e^2 = " << w3[0][0] <<" (7.389056...)"<< std::endl;
    dg::blas1::scal( w2, 0.6);
    dg::blas1::plus( w3, -7.0);
    std::cout << "e^2-7 = " << w3[0][0] <<" (0.389056...)"<< std::endl;
    std::cout << "FINISHED\n\n";


    return 0;

}
