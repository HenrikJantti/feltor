#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "dg/blas.h"
#include "derivatives.h"
#include "evaluation.cuh"

double zero( double x, double y) { return 0;}
double sine( double x, double y) { return sin(x)*sin(y);}
double cosx( double x, double y) { return cos(x)*sin(y);}
double cosy( double x, double y) { return cos(y)*sin(x);}
double zero( double x, double y, double z) { return 0;}
double sine( double x, double y, double z) { return sin(x)*sin(y)*sin(z);}
double cosx( double x, double y, double z) { return cos(x)*sin(y)*sin(z);}
double cosy( double x, double y, double z) { return cos(y)*sin(x)*sin(z);}
double cosz( double x, double y, double z) { return cos(z)*sin(x)*sin(y);}

typedef dg::DMatrix Matrix;
typedef dg::DVec Vector;

int main()
{
    unsigned n, Nx, Ny, Nz;
    std::cout << "Type in n, Nx and Ny and Nz!\n";
    std::cin >> n >> Nx >> Ny >> Nz;
    dg::bc bcx=dg::DIR, bcy=dg::PER, bcz=dg::NEU_DIR;
    dg::Grid2d g2d( 0, M_PI, 0.1, 2*M_PI+0.1, n, Nx, Ny, bcx, bcy);
    const Vector w2d = dg::create::weights( g2d);

    Matrix dx2 = dg::create::dx( g2d, dg::forward);
    Matrix dy2 = dg::create::dy( g2d, dg::centered);
    Matrix jx2 = dg::create::jumpX( g2d);
    Matrix jy2 = dg::create::jumpY( g2d);
    Matrix m2[] = {dx2, dy2, jx2, jy2};
    const Vector f2d = dg::evaluate( sine, g2d);
    const Vector dx2d = dg::evaluate( cosx, g2d);
    const Vector dy2d = dg::evaluate( cosy, g2d);
    const Vector null2 = dg::evaluate( zero, g2d);
    Vector sol2[] = {dx2d, dy2d, null2, null2};

    exblas::udouble res;
    std::cout << "WE EXPECT CONVERGENCE IN ALL QUANTITIES!!!\n";
    std::cout << "TEST 2D: DX, DY, JX, JY\n";
    for( unsigned i=0; i<4; i++)
    {
        Vector error = sol2[i];
        dg::blas2::symv( -1., m2[i], f2d, 1., error);
        dg::blas1::pointwiseDot( error, error, error);
        double norm = sqrt(dg::blas1::dot( w2d, error)); res.d = norm;
        std::cout << "Distance to true solution: "<<norm<<"\t"<<res.i<<"\n";
    }
    dg::Grid3d g3d( 0,M_PI, 0.1, 2.*M_PI+0.1, M_PI/2.,M_PI, n, Nx, Ny, Nz, bcx, bcy, bcz);
    const Vector w3d = dg::create::weights( g3d);
    Matrix dx3 = dg::create::dx( g3d, dg::forward);
    Matrix dy3 = dg::create::dy( g3d, dg::centered);
    Matrix dz3 = dg::create::dz( g3d, dg::backward);
    Matrix jx3 = dg::create::jumpX( g3d);
    Matrix jy3 = dg::create::jumpY( g3d);
    Matrix jz3 = dg::create::jumpZ( g3d);
    Matrix m3[] = {dx3, dy3, dz3, jx3, jy3, jz3};
    const Vector f3d = dg::evaluate( sine, g3d);
    const Vector dx3d = dg::evaluate( cosx, g3d);
    const Vector dy3d = dg::evaluate( cosy, g3d);
    const Vector dz3d = dg::evaluate( cosz, g3d);
    const Vector null3 = dg::evaluate( zero, g3d);
    Vector sol3[] = {dx3d, dy3d, dz3d, null3, null3, null3};

    std::cout << "TEST 3D: DX, DY, DZ, JX, JY, JZ\n";
    for( unsigned i=0; i<6; i++)
    {
        Vector error = sol3[i];
        dg::blas2::symv( -1., m3[i], f3d, 1., error);
        double norm = sqrt(dg::blas2::dot( error, w3d, error)); res.d = norm;
        std::cout << "Distance to true solution: "<<norm<<"\t"<<res.i<<"\n";
    }
    //for periodic bc | dirichlet bc
    //n = 1 -> p = 2      2
    //n = 2 -> p = 1      1
    //n = 3 -> p = 3      3
    //n = 4 -> p = 3      3
    //n = 5 -> p = 5      5

    return 0;
}
