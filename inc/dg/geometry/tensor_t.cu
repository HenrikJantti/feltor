#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include "tensor.h"
#include "multiply.h"

void print( const dg::SparseTensor<thrust::host_vector<double> >& t)
{
    for( unsigned i=0; i<3; i++)
    {
        for( unsigned j=0; j<3; j++)
        {
            if(t.isSet(i,j)) std::cout << t.value(i,j)[0]<<" ";
            else std::cout <<"xx ";
        }
        std::cout << "\n";
    }

}
std::ostream& operator<<(std::ostream& os, const dg::SparseElement<thrust::host_vector<double> >& t)
{
    if(t.isSet()) os << t.value()[0]<<" ";
    else os <<"XX ";
    return os;
}

int main()
{
    thrust::host_vector<double> one(1,10), two(1,20), three(1,30), four(1,40), five(1,50), six(1,60), seven(1,70), eight(1,80), nine(1,90);

    dg::SparseTensor<thrust::host_vector<double> > dense2d(3);
    dense2d.idx(0,0) = 0, dense2d.idx(0,1) = 1;
    dense2d.idx(1,0) = 1, dense2d.idx(1,1) = 2;
    dense2d.value(0) = eight; dense2d.value(1) = two; dense2d.value(2) = nine; 
    dg::SparseTensor<thrust::host_vector<double> > sparse3d(4);
    sparse3d.idx(0,0) = 0, sparse3d.idx(0,1) = 1                       ;
    sparse3d.idx(1,0) = 1                       , sparse3d.idx(1,2) = 3;
    sparse3d.idx(2,0) = 1                       , sparse3d.idx(2,2) = 3;
    sparse3d.value(0) = seven; sparse3d.value(1) = three; sparse3d.value(2) = nine, sparse3d.value(3) = one; 

    dg::SparseTensor<thrust::host_vector<double> > empty;

    std::cout << "Test dg::Sparse Tensor class \n";
    std::cout << "Dense 2d Tensor \n";
    print( dense2d);
    std::cout << "Sparse 3d Tensor \n";
    print( sparse3d);
    std::cout << "empty Tensor \n";
    print( empty);


    std::cout<< "Test dg::SparseElement";
    dg::SparseElement<thrust::host_vector<double> > e(eight);
    dg::SparseElement<thrust::host_vector<double> > ee;
    std::cout<<"\n construct: " <<e<<" "<<ee<<"\n";
    ee = e;
    e.value()=nine;
    std::cout << "Assignment and set : "<<ee<<" (80) "<<e<<"(90)\n";
    dg::SparseElement<thrust::host_vector<double> > sqr = e.sqrt();
    std::cout<<"sqrt(): "<<sqr<<" ("<<std::sqrt(90)<<")\n";
    sqr = sqr.invert();
    std::cout<<"invert(): "<<sqr<<"\n";
    sqr.clear();
    std::cout<<"clear(): "<<sqr<<"\n";
    std::cout <<std::flush;
    return 0;


}
